#include "hip/hip_runtime.h"
// Copyright (C) ABQ-LLM (liusongwei.zju@bytedance.com)
// 
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
// 
//          http://www.apache.org/licenses/LICENSE-2.0
// 
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <fstream>
#include <iomanip>
#include <iostream>
#include <sstream>
#include <string>
#include "common/base.h"
#include "common/pack.h"
#include "common/timer.h"
#include "mma_any/aq_wmma_library.h"
#include "mma_any/aq_wmma_op.h"

/// benchmark func for wmma
inline bool isCudaSuccess(hipError_t status)
{
    hipError_t error = status;
    if (error != hipSuccess) {
        std::cerr << "Got bad cuda status: " << hipGetErrorString(error) << std::endl;
        return false;
    }
    return true;
}

bool check(const int *ref_out, const int *out, int m, int n)
{
    for (int i = 0; i < m * n; ++i) {
        if (ref_out[i] != out[i]) {
            return false;
        }
    }
    return true;
}


void print_matrix(int *matrix, int m, int n, bool hex)
{
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            if (hex)
                printf("%x,", matrix[i * n + j]);
            else
                printf("%d,", matrix[i * n + j]);
        }
        printf("\n");
    }
}

int int_pow(int base, int exp)
{
    int result = 1;
    while (exp) {
        if (exp % 2)
            result *= base;
        exp /= 2;
        base *= base;
    }
    return result;
}

void init_matrix(int *matrix, int m, int n, int bits)
{
    for (int i = 0; i < m * n; ++i) {
        matrix[i] = rand() % int_pow(2, bits);
    }
}

void compute_ref(int *w, int *x, int *ref_c, int M, int N, int K, int W_BIT, int X_BIT, bool SIGNED)
{
    for (int m = 0; m < M; m++) {
        for (int n = 0; n < N; n++) {
            int tmp = 0;
            for (int xb = 0; xb < X_BIT; xb++) {
                int X_Multiplier =
                    SIGNED && (xb == X_BIT - 1) ? -1 * int_pow(2, xb) : int_pow(2, xb);
                for (int wb = 0; wb < W_BIT; wb++) {
                    int W_Multiplier =
                        SIGNED && (wb == W_BIT - 1) ? -1 * int_pow(2, wb) : int_pow(2, wb);
                    for (int k_tile = 0; k_tile < K / 32; k_tile++) {
                        int w_int = w[wb * N * K / 32 + n * K / 32 + k_tile];
                        int x_int = x[xb * M * K / 32 + m * K / 32 + k_tile];
                        for (int k = 0; k < 32; k++) {
                            int mask = 1;
                            int x_val = ((mask << k) & x_int) >> k;
                            int w_val = ((mask << k) & w_int) >> k;
                            tmp += X_Multiplier * W_Multiplier * x_val * w_val;
                        }
                    }
                }
            }
            ref_c[m * N + n] = tmp;
        }
    }
}

#define TEST(X_BITS, W_BITS, SIGNED, BM, BN, BK, WM, WN, WK, MMA_M, MMA_N, MMA_K, NSTAGE)      \
    {                                                                                          \
        std::cout << GPU_ARCH << " " << config_str << " ";                                     \
        printf("%d %d %d %d %d %d %d %d %d %d ", BM, BN, BK, WM, WN, WK, MMA_M, MMA_N, MMA_K,  \
               NSTAGE);                                                                        \
        int ret = benchmark<AQ_INIT_FUN(AqBWMMA), AQ_EXEC_FUN(AqBWMMA), AQ_OP_STATE(AqBWMMA)>( \
            AQ_NAME_FUN(AqBWMMA, Init, X_BITS, W_BITS, SIGNED, BM, BN, BK, WM, WN, WK, MMA_M,  \
                        MMA_N, MMA_K, NSTAGE),                                                 \
            AQ_NAME_FUN(AqBWMMA, Exec, X_BITS, W_BITS, SIGNED, BM, BN, BK, WM, WN, WK, MMA_M,  \
                        MMA_N, MMA_K, NSTAGE),                                                 \
            x_bits, w_bits, d_x, d_w, d_x_pack, d_w_pack, m, n, k, d_out, nullptr, h_out,      \
            h_ref_out, false, SIGNED, exec_dur, pack_dur, stream, warmup, repeat);             \
        if (ret == 0 && gflop_count / exec_dur > max_gflop) {                                  \
            max_gflop = gflop_count / exec_dur;                                                \
            min_latency = exec_dur * 1e3;                                                      \
            best_config.str("");                                                               \
            best_config << BM << ", " << BN << ", " << BK << ", " << WM << ", " << WN << ", "  \
                        << WK << ", " << MMA_M << ", " << MMA_N << ", " << MMA_K << ", "       \
                        << NSTAGE;                                                             \
        }                                                                                      \
        printf("packing %f (us) exec %f (us) %f TOPS | %f B-TOPS | %s\n", pack_dur * 1e3,      \
               exec_dur * 1e3, gflop_count / exec_dur, true_gflop_count / exec_dur,            \
               ret == 0  ? "PASSED" :                                                          \
               ret == -1 ? "ERROR" :                                                           \
                           "FAILED");                                                          \
    }

int main(int argc, char **argv)
{
    if (argc < 7) {
        printf("Usage: ./test_any_wmma M N K X_BITS W_BITS SIGNED\n");
        return -1;
    }

    int m = atoi(argv[1]);
    int n = atoi(argv[2]);
    int k = atoi(argv[3]);
    int x_bits = atoi(argv[4]);
    int w_bits = atoi(argv[5]);
    bool quant_sign = atoi(argv[6]) == 1;
    if (k < 128 || k % 128 != 0) {
        printf("Error, k must >= 128 and k % 128 == 0!");
        return -1;
    }
    int repeat = 1000;
    int warmup = 10;
    float exec_dur = 0;
    float pack_dur = 0;
    hipStream_t stream;
    hipStreamCreate(&stream);
    std::string config_str;
    std::stringstream s;
    s << x_bits << " " << w_bits << " " << m << " " << n << " " << k << " ";
    if (quant_sign) {
        s << "sign ";
    } else {
        s << "unsigned ";
    }
    config_str = s.str();
    float true_gflop_count = (float)m / 1e9 * n * k * 2 * x_bits * w_bits;
    float gflop_count = (float)m / 1e9 * n * k * 2;
    float max_gflop = 0;
    float min_latency = 1e9;
    std::stringstream best_config;

    int *h_x = (int *)malloc(m * k * sizeof(int));
    int *h_w = (int *)malloc(n * k * sizeof(int));
    int *h_x_pack = (int *)malloc(x_bits * m * (k / 32) * sizeof(int));
    int *h_w_pack = (int *)malloc(w_bits * n * (k / 32) * sizeof(int));
    int *h_out = (int *)malloc(m * n * sizeof(int));
    int *h_ref_out = (int *)malloc(m * n * sizeof(int));

    int *d_x;
    int *d_x_pack;
    int *d_w;
    int *d_w_pack;
    int *d_out;
    hipMalloc(&d_x, m * k * sizeof(int));
    hipMalloc(&d_w, n * k * sizeof(int));
    hipMalloc(&d_x_pack, x_bits * m * (k / 32) * sizeof(int));
    hipMalloc(&d_w_pack, w_bits * n * (k / 32) * sizeof(int));
    hipMalloc(&d_out, m * n * sizeof(int));
    // INIT HOST TENSOR
    init_matrix(h_x, m, k, x_bits);
    init_matrix(h_w, n, k, w_bits);
    hipMemcpy(d_x, h_x, sizeof(int) * m * k, hipMemcpyHostToDevice);
    hipMemcpy(d_w, h_w, sizeof(int) * n * k, hipMemcpyHostToDevice);

    if (w_bits <= 32) {
        hipError_t err = launch_pack(d_w, d_w_pack, n, k, w_bits);
        if (err != hipSuccess) {
            printf("Line %d: 'weight launch_pack' failed: %s\n", __LINE__, hipGetErrorString(err));
            return -1;
        }
    } else {
        printf("unsupport w_bits %d: for launch_pack func \n", w_bits);
        return -1;
    }

    if (x_bits <= 32) {
        hipError_t err = launch_pack(d_x, d_x_pack, m, k, x_bits);
        if (err != hipSuccess) {
            printf("Line %d: 'activation launch_pack' failed: %s\n", __LINE__,
                   hipGetErrorString(err));
            return -1;
        }
    } else {
        printf("unsupport x_bits %d: for launch_pack func \n", x_bits);
        return -1;
    }

    hipMemcpy(h_x_pack, d_x_pack, x_bits * m * (k / 32) * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_w_pack, d_w_pack, w_bits * n * (k / 32) * sizeof(int), hipMemcpyDeviceToHost);

    compute_ref(h_w_pack, h_x_pack, h_ref_out, m, n, k, w_bits, x_bits, quant_sign);

    switch (x_bits) {
    case 2:
        switch (w_bits) {
        #ifdef W2A2
        case 2:
            if (quant_sign) {
                ////// W2A2 int
                // cta<4,32,256> warp<8,32,128> mma<8,8,128>   WARPS[1x2]
                TEST(2, 2, true, 4, 32, 256, 8, 32, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 4, 32, 256, 8, 32, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 4, 32, 256, 8, 32, 128, 8, 8, 128, 4);
                // cta<4,48,256> warp<8,48,128> mma<8,8,128>   WARPS[1x2]
                TEST(2, 2, true, 4, 48, 256, 8, 48, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 4, 48, 256, 8, 48, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 4, 48, 256, 8, 48, 128, 8, 8, 128, 4);
                // cta<4,64,256> warp<8,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(2, 2, true, 4, 64, 256, 8, 64, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 4, 64, 256, 8, 64, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 4, 64, 256, 8, 64, 128, 8, 8, 128, 4);
                // cta<4,80,256> warp<8,80,128> mma<8,8,128>   WARPS[1x2]
                TEST(2, 2, true, 4, 80, 256, 8, 80, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 4, 80, 256, 8, 80, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 4, 80, 256, 8, 80, 128, 8, 8, 128, 4);
                // cta<4,96,256> warp<8,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(2, 2, true, 4, 96, 256, 8, 96, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 4, 96, 256, 8, 96, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 4, 96, 256, 8, 96, 128, 8, 8, 128, 4);
                // cta<4,112,256> warp<8,112,128> mma<8,8,128>   WARPS[1x2]
                TEST(2, 2, true, 4, 112, 256, 8, 112, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 4, 112, 256, 8, 112, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 4, 112, 256, 8, 112, 128, 8, 8, 128, 4);
                // cta<4,128,256> warp<8,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(2, 2, true, 4, 128, 256, 8, 128, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 4, 128, 256, 8, 128, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 4, 128, 256, 8, 128, 128, 8, 8, 128, 4);
                // cta<8,32,256> warp<16,32,128> mma<8,8,128>   WARPS[1x2]
                TEST(2, 2, true, 8, 32, 256, 16, 32, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 32, 256, 16, 32, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 32, 256, 16, 32, 128, 8, 8, 128, 4);
                // cta<8,48,256> warp<16,48,128> mma<8,8,128>   WARPS[1x2]
                TEST(2, 2, true, 8, 48, 256, 16, 48, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 48, 256, 16, 48, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 48, 256, 16, 48, 128, 8, 8, 128, 4);
                // cta<8,64,256> warp<16,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(2, 2, true, 8, 64, 256, 16, 64, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 64, 256, 16, 64, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 64, 256, 16, 64, 128, 8, 8, 128, 4);
                // cta<8,80,256> warp<16,80,128> mma<8,8,128>   WARPS[1x2]
                TEST(2, 2, true, 8, 80, 256, 16, 80, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 80, 256, 16, 80, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 80, 256, 16, 80, 128, 8, 8, 128, 4);
                // cta<8,96,256> warp<16,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(2, 2, true, 8, 96, 256, 16, 96, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 96, 256, 16, 96, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 96, 256, 16, 96, 128, 8, 8, 128, 4);
                // cta<8,112,256> warp<16,112,128> mma<8,8,128>   WARPS[1x2]
                TEST(2, 2, true, 8, 112, 256, 16, 112, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 112, 256, 16, 112, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 112, 256, 16, 112, 128, 8, 8, 128, 4);
                // cta<8,128,256> warp<16,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(2, 2, true, 8, 128, 256, 16, 128, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 128, 256, 16, 128, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 128, 256, 16, 128, 128, 8, 8, 128, 4);
                // cta<4,32,256> warp<8,16,128> mma<8,8,128>   WARPS[1x4]
                TEST(2, 2, true, 4, 32, 256, 8, 16, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 4, 32, 256, 8, 16, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 4, 32, 256, 8, 16, 128, 8, 8, 128, 4);
                // cta<4,48,256> warp<8,24,128> mma<8,8,128>   WARPS[1x4]
                TEST(2, 2, true, 4, 48, 256, 8, 24, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 4, 48, 256, 8, 24, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 4, 48, 256, 8, 24, 128, 8, 8, 128, 4);
                // cta<4,64,256> warp<8,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(2, 2, true, 4, 64, 256, 8, 32, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 4, 64, 256, 8, 32, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 4, 64, 256, 8, 32, 128, 8, 8, 128, 4);
                // cta<4,80,256> warp<8,40,128> mma<8,8,128>   WARPS[1x4]
                TEST(2, 2, true, 4, 80, 256, 8, 40, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 4, 80, 256, 8, 40, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 4, 80, 256, 8, 40, 128, 8, 8, 128, 4);
                // cta<4,96,256> warp<8,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(2, 2, true, 4, 96, 256, 8, 48, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 4, 96, 256, 8, 48, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 4, 96, 256, 8, 48, 128, 8, 8, 128, 4);
                // cta<4,112,256> warp<8,56,128> mma<8,8,128>   WARPS[1x4]
                TEST(2, 2, true, 4, 112, 256, 8, 56, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 4, 112, 256, 8, 56, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 4, 112, 256, 8, 56, 128, 8, 8, 128, 4);
                // cta<4,128,256> warp<8,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(2, 2, true, 4, 128, 256, 8, 64, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 4, 128, 256, 8, 64, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 4, 128, 256, 8, 64, 128, 8, 8, 128, 4);
                // cta<4,256,256> warp<8,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(2, 2, true, 4, 256, 256, 8, 128, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 4, 256, 256, 8, 128, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 4, 256, 256, 8, 128, 128, 8, 8, 128, 4);
                // cta<8,32,256> warp<16,16,128> mma<8,8,128>   WARPS[1x4]
                TEST(2, 2, true, 8, 32, 256, 16, 16, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 32, 256, 16, 16, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 32, 256, 16, 16, 128, 8, 8, 128, 4);
                // cta<8,48,256> warp<16,24,128> mma<8,8,128>   WARPS[1x4]
                TEST(2, 2, true, 8, 48, 256, 16, 24, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 48, 256, 16, 24, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 48, 256, 16, 24, 128, 8, 8, 128, 4);
                // cta<8,64,256> warp<16,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(2, 2, true, 8, 64, 256, 16, 32, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 64, 256, 16, 32, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 64, 256, 16, 32, 128, 8, 8, 128, 4);
                // cta<8,80,256> warp<16,40,128> mma<8,8,128>   WARPS[1x4]
                TEST(2, 2, true, 8, 80, 256, 16, 40, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 80, 256, 16, 40, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 80, 256, 16, 40, 128, 8, 8, 128, 4);
                // cta<8,96,256> warp<16,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(2, 2, true, 8, 96, 256, 16, 48, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 96, 256, 16, 48, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 96, 256, 16, 48, 128, 8, 8, 128, 4);
                // cta<8,112,256> warp<16,56,128> mma<8,8,128>   WARPS[1x4]
                TEST(2, 2, true, 8, 112, 256, 16, 56, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 112, 256, 16, 56, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 112, 256, 16, 56, 128, 8, 8, 128, 4);
                // cta<8,128,256> warp<16,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(2, 2, true, 8, 128, 256, 16, 64, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 128, 256, 16, 64, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 128, 256, 16, 64, 128, 8, 8, 128, 4);
                // cta<8,256,256> warp<16,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(2, 2, true, 8, 256, 256, 16, 128, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 256, 256, 16, 128, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 256, 256, 16, 128, 128, 8, 8, 128, 4);
                // cta<8,32,256> warp<8,64,128> mma<8,8,128>   WARPS[2x1]
                TEST(2, 2, true, 8, 32, 256, 8, 64, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 32, 256, 8, 64, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 32, 256, 8, 64, 128, 8, 8, 128, 4);
                // cta<8,48,256> warp<8,96,128> mma<8,8,128>   WARPS[2x1]
                TEST(2, 2, true, 8, 48, 256, 8, 96, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 48, 256, 8, 96, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 48, 256, 8, 96, 128, 8, 8, 128, 4);
                // cta<8,64,256> warp<8,128,128> mma<8,8,128>   WARPS[2x1]
                TEST(2, 2, true, 8, 64, 256, 8, 128, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 64, 256, 8, 128, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 64, 256, 8, 128, 128, 8, 8, 128, 4);
                // cta<8,32,256> warp<8,32,128> mma<8,8,128>   WARPS[2x2]
                TEST(2, 2, true, 8, 32, 256, 8, 32, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 32, 256, 8, 32, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 32, 256, 8, 32, 128, 8, 8, 128, 4);
                // cta<8,48,256> warp<8,48,128> mma<8,8,128>   WARPS[2x2]
                TEST(2, 2, true, 8, 48, 256, 8, 48, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 48, 256, 8, 48, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 48, 256, 8, 48, 128, 8, 8, 128, 4);
                // cta<8,64,256> warp<8,64,128> mma<8,8,128>   WARPS[2x2]
                TEST(2, 2, true, 8, 64, 256, 8, 64, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 64, 256, 8, 64, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 64, 256, 8, 64, 128, 8, 8, 128, 4);
                // cta<8,80,256> warp<8,80,128> mma<8,8,128>   WARPS[2x2]
                TEST(2, 2, true, 8, 80, 256, 8, 80, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 80, 256, 8, 80, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 80, 256, 8, 80, 128, 8, 8, 128, 4);
                // cta<8,96,256> warp<8,96,128> mma<8,8,128>   WARPS[2x2]
                TEST(2, 2, true, 8, 96, 256, 8, 96, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 96, 256, 8, 96, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 96, 256, 8, 96, 128, 8, 8, 128, 4);
                // cta<8,112,256> warp<8,112,128> mma<8,8,128>   WARPS[2x2]
                TEST(2, 2, true, 8, 112, 256, 8, 112, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 112, 256, 8, 112, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 112, 256, 8, 112, 128, 8, 8, 128, 4);
                // cta<8,128,256> warp<8,128,128> mma<8,8,128>   WARPS[2x2]
                TEST(2, 2, true, 8, 128, 256, 8, 128, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 128, 256, 8, 128, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 128, 256, 8, 128, 128, 8, 8, 128, 4);
                // cta<4,32,384> warp<8,32,128> mma<8,8,128>   WARPS[1x2]
                TEST(2, 2, true, 4, 32, 384, 8, 32, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 4, 32, 384, 8, 32, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 4, 32, 384, 8, 32, 128, 8, 8, 128, 4);
                // cta<4,48,384> warp<8,48,128> mma<8,8,128>   WARPS[1x2]
                TEST(2, 2, true, 4, 48, 384, 8, 48, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 4, 48, 384, 8, 48, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 4, 48, 384, 8, 48, 128, 8, 8, 128, 4);
                // cta<4,64,384> warp<8,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(2, 2, true, 4, 64, 384, 8, 64, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 4, 64, 384, 8, 64, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 4, 64, 384, 8, 64, 128, 8, 8, 128, 4);
                // cta<4,80,384> warp<8,80,128> mma<8,8,128>   WARPS[1x2]
                TEST(2, 2, true, 4, 80, 384, 8, 80, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 4, 80, 384, 8, 80, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 4, 80, 384, 8, 80, 128, 8, 8, 128, 4);
                // cta<4,96,384> warp<8,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(2, 2, true, 4, 96, 384, 8, 96, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 4, 96, 384, 8, 96, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 4, 96, 384, 8, 96, 128, 8, 8, 128, 4);
                // cta<4,112,384> warp<8,112,128> mma<8,8,128>   WARPS[1x2]
                TEST(2, 2, true, 4, 112, 384, 8, 112, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 4, 112, 384, 8, 112, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 4, 112, 384, 8, 112, 128, 8, 8, 128, 4);
                // cta<4,128,384> warp<8,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(2, 2, true, 4, 128, 384, 8, 128, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 4, 128, 384, 8, 128, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 4, 128, 384, 8, 128, 128, 8, 8, 128, 4);
                // cta<8,32,384> warp<16,32,128> mma<8,8,128>   WARPS[1x2]
                TEST(2, 2, true, 8, 32, 384, 16, 32, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 32, 384, 16, 32, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 32, 384, 16, 32, 128, 8, 8, 128, 4);
                // cta<8,48,384> warp<16,48,128> mma<8,8,128>   WARPS[1x2]
                TEST(2, 2, true, 8, 48, 384, 16, 48, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 48, 384, 16, 48, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 48, 384, 16, 48, 128, 8, 8, 128, 4);
                // cta<8,64,384> warp<16,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(2, 2, true, 8, 64, 384, 16, 64, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 64, 384, 16, 64, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 64, 384, 16, 64, 128, 8, 8, 128, 4);
                // cta<8,80,384> warp<16,80,128> mma<8,8,128>   WARPS[1x2]
                TEST(2, 2, true, 8, 80, 384, 16, 80, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 80, 384, 16, 80, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 80, 384, 16, 80, 128, 8, 8, 128, 4);
                // cta<8,96,384> warp<16,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(2, 2, true, 8, 96, 384, 16, 96, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 96, 384, 16, 96, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 96, 384, 16, 96, 128, 8, 8, 128, 4);
                // cta<8,112,384> warp<16,112,128> mma<8,8,128>   WARPS[1x2]
                TEST(2, 2, true, 8, 112, 384, 16, 112, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 112, 384, 16, 112, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 112, 384, 16, 112, 128, 8, 8, 128, 4);
                // cta<8,128,384> warp<16,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(2, 2, true, 8, 128, 384, 16, 128, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 128, 384, 16, 128, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 128, 384, 16, 128, 128, 8, 8, 128, 4);
                // cta<4,32,384> warp<8,16,128> mma<8,8,128>   WARPS[1x4]
                TEST(2, 2, true, 4, 32, 384, 8, 16, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 4, 32, 384, 8, 16, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 4, 32, 384, 8, 16, 128, 8, 8, 128, 4);
                // cta<4,48,384> warp<8,24,128> mma<8,8,128>   WARPS[1x4]
                TEST(2, 2, true, 4, 48, 384, 8, 24, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 4, 48, 384, 8, 24, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 4, 48, 384, 8, 24, 128, 8, 8, 128, 4);
                // cta<4,64,384> warp<8,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(2, 2, true, 4, 64, 384, 8, 32, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 4, 64, 384, 8, 32, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 4, 64, 384, 8, 32, 128, 8, 8, 128, 4);
                // cta<4,80,384> warp<8,40,128> mma<8,8,128>   WARPS[1x4]
                TEST(2, 2, true, 4, 80, 384, 8, 40, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 4, 80, 384, 8, 40, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 4, 80, 384, 8, 40, 128, 8, 8, 128, 4);
                // cta<4,96,384> warp<8,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(2, 2, true, 4, 96, 384, 8, 48, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 4, 96, 384, 8, 48, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 4, 96, 384, 8, 48, 128, 8, 8, 128, 4);
                // cta<4,112,384> warp<8,56,128> mma<8,8,128>   WARPS[1x4]
                TEST(2, 2, true, 4, 112, 384, 8, 56, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 4, 112, 384, 8, 56, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 4, 112, 384, 8, 56, 128, 8, 8, 128, 4);
                // cta<4,128,384> warp<8,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(2, 2, true, 4, 128, 384, 8, 64, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 4, 128, 384, 8, 64, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 4, 128, 384, 8, 64, 128, 8, 8, 128, 4);
                // cta<4,256,384> warp<8,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(2, 2, true, 4, 256, 384, 8, 128, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 4, 256, 384, 8, 128, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 4, 256, 384, 8, 128, 128, 8, 8, 128, 4);
                // cta<8,32,384> warp<16,16,128> mma<8,8,128>   WARPS[1x4]
                TEST(2, 2, true, 8, 32, 384, 16, 16, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 32, 384, 16, 16, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 32, 384, 16, 16, 128, 8, 8, 128, 4);
                // cta<8,48,384> warp<16,24,128> mma<8,8,128>   WARPS[1x4]
                TEST(2, 2, true, 8, 48, 384, 16, 24, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 48, 384, 16, 24, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 48, 384, 16, 24, 128, 8, 8, 128, 4);
                // cta<8,64,384> warp<16,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(2, 2, true, 8, 64, 384, 16, 32, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 64, 384, 16, 32, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 64, 384, 16, 32, 128, 8, 8, 128, 4);
                // cta<8,80,384> warp<16,40,128> mma<8,8,128>   WARPS[1x4]
                TEST(2, 2, true, 8, 80, 384, 16, 40, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 80, 384, 16, 40, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 80, 384, 16, 40, 128, 8, 8, 128, 4);
                // cta<8,96,384> warp<16,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(2, 2, true, 8, 96, 384, 16, 48, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 96, 384, 16, 48, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 96, 384, 16, 48, 128, 8, 8, 128, 4);
                // cta<8,112,384> warp<16,56,128> mma<8,8,128>   WARPS[1x4]
                TEST(2, 2, true, 8, 112, 384, 16, 56, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 112, 384, 16, 56, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 112, 384, 16, 56, 128, 8, 8, 128, 4);
                // cta<8,128,384> warp<16,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(2, 2, true, 8, 128, 384, 16, 64, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 128, 384, 16, 64, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 128, 384, 16, 64, 128, 8, 8, 128, 4);
                // cta<8,256,384> warp<16,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(2, 2, true, 8, 256, 384, 16, 128, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 256, 384, 16, 128, 128, 8, 8, 128, 3);
                // cta<8,32,384> warp<8,64,128> mma<8,8,128>   WARPS[2x1]
                TEST(2, 2, true, 8, 32, 384, 8, 64, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 32, 384, 8, 64, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 32, 384, 8, 64, 128, 8, 8, 128, 4);
                // cta<8,48,384> warp<8,96,128> mma<8,8,128>   WARPS[2x1]
                TEST(2, 2, true, 8, 48, 384, 8, 96, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 48, 384, 8, 96, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 48, 384, 8, 96, 128, 8, 8, 128, 4);
                // cta<8,64,384> warp<8,128,128> mma<8,8,128>   WARPS[2x1]
                TEST(2, 2, true, 8, 64, 384, 8, 128, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 64, 384, 8, 128, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 64, 384, 8, 128, 128, 8, 8, 128, 4);
                // cta<8,32,384> warp<8,32,128> mma<8,8,128>   WARPS[2x2]
                TEST(2, 2, true, 8, 32, 384, 8, 32, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 32, 384, 8, 32, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 32, 384, 8, 32, 128, 8, 8, 128, 4);
                // cta<8,48,384> warp<8,48,128> mma<8,8,128>   WARPS[2x2]
                TEST(2, 2, true, 8, 48, 384, 8, 48, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 48, 384, 8, 48, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 48, 384, 8, 48, 128, 8, 8, 128, 4);
                // cta<8,64,384> warp<8,64,128> mma<8,8,128>   WARPS[2x2]
                TEST(2, 2, true, 8, 64, 384, 8, 64, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 64, 384, 8, 64, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 64, 384, 8, 64, 128, 8, 8, 128, 4);
                // cta<8,80,384> warp<8,80,128> mma<8,8,128>   WARPS[2x2]
                TEST(2, 2, true, 8, 80, 384, 8, 80, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 80, 384, 8, 80, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 80, 384, 8, 80, 128, 8, 8, 128, 4);
                // cta<8,96,384> warp<8,96,128> mma<8,8,128>   WARPS[2x2]
                TEST(2, 2, true, 8, 96, 384, 8, 96, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 96, 384, 8, 96, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 96, 384, 8, 96, 128, 8, 8, 128, 4);
                // cta<8,112,384> warp<8,112,128> mma<8,8,128>   WARPS[2x2]
                TEST(2, 2, true, 8, 112, 384, 8, 112, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 112, 384, 8, 112, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 112, 384, 8, 112, 128, 8, 8, 128, 4);
                // cta<8,128,384> warp<8,128,128> mma<8,8,128>   WARPS[2x2]
                TEST(2, 2, true, 8, 128, 384, 8, 128, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 128, 384, 8, 128, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 128, 384, 8, 128, 128, 8, 8, 128, 4);
                // cta<4,32,512> warp<8,32,128> mma<8,8,128>   WARPS[1x2]
                TEST(2, 2, true, 4, 32, 512, 8, 32, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 4, 32, 512, 8, 32, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 4, 32, 512, 8, 32, 128, 8, 8, 128, 4);
                // cta<4,48,512> warp<8,48,128> mma<8,8,128>   WARPS[1x2]
                TEST(2, 2, true, 4, 48, 512, 8, 48, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 4, 48, 512, 8, 48, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 4, 48, 512, 8, 48, 128, 8, 8, 128, 4);
                // cta<4,64,512> warp<8,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(2, 2, true, 4, 64, 512, 8, 64, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 4, 64, 512, 8, 64, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 4, 64, 512, 8, 64, 128, 8, 8, 128, 4);
                // cta<4,80,512> warp<8,80,128> mma<8,8,128>   WARPS[1x2]
                TEST(2, 2, true, 4, 80, 512, 8, 80, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 4, 80, 512, 8, 80, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 4, 80, 512, 8, 80, 128, 8, 8, 128, 4);
                // cta<4,96,512> warp<8,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(2, 2, true, 4, 96, 512, 8, 96, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 4, 96, 512, 8, 96, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 4, 96, 512, 8, 96, 128, 8, 8, 128, 4);
                // cta<4,112,512> warp<8,112,128> mma<8,8,128>   WARPS[1x2]
                TEST(2, 2, true, 4, 112, 512, 8, 112, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 4, 112, 512, 8, 112, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 4, 112, 512, 8, 112, 128, 8, 8, 128, 4);
                // cta<4,128,512> warp<8,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(2, 2, true, 4, 128, 512, 8, 128, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 4, 128, 512, 8, 128, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 4, 128, 512, 8, 128, 128, 8, 8, 128, 4);
                // cta<8,32,512> warp<16,32,128> mma<8,8,128>   WARPS[1x2]
                TEST(2, 2, true, 8, 32, 512, 16, 32, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 32, 512, 16, 32, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 32, 512, 16, 32, 128, 8, 8, 128, 4);
                // cta<8,48,512> warp<16,48,128> mma<8,8,128>   WARPS[1x2]
                TEST(2, 2, true, 8, 48, 512, 16, 48, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 48, 512, 16, 48, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 48, 512, 16, 48, 128, 8, 8, 128, 4);
                // cta<8,64,512> warp<16,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(2, 2, true, 8, 64, 512, 16, 64, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 64, 512, 16, 64, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 64, 512, 16, 64, 128, 8, 8, 128, 4);
                // cta<8,80,512> warp<16,80,128> mma<8,8,128>   WARPS[1x2]
                TEST(2, 2, true, 8, 80, 512, 16, 80, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 80, 512, 16, 80, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 80, 512, 16, 80, 128, 8, 8, 128, 4);
                // cta<8,96,512> warp<16,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(2, 2, true, 8, 96, 512, 16, 96, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 96, 512, 16, 96, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 96, 512, 16, 96, 128, 8, 8, 128, 4);
                // cta<8,112,512> warp<16,112,128> mma<8,8,128>   WARPS[1x2]
                TEST(2, 2, true, 8, 112, 512, 16, 112, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 112, 512, 16, 112, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 112, 512, 16, 112, 128, 8, 8, 128, 4);
                // cta<8,128,512> warp<16,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(2, 2, true, 8, 128, 512, 16, 128, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 128, 512, 16, 128, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 128, 512, 16, 128, 128, 8, 8, 128, 4);
                // cta<4,32,512> warp<8,16,128> mma<8,8,128>   WARPS[1x4]
                TEST(2, 2, true, 4, 32, 512, 8, 16, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 4, 32, 512, 8, 16, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 4, 32, 512, 8, 16, 128, 8, 8, 128, 4);
                // cta<4,48,512> warp<8,24,128> mma<8,8,128>   WARPS[1x4]
                TEST(2, 2, true, 4, 48, 512, 8, 24, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 4, 48, 512, 8, 24, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 4, 48, 512, 8, 24, 128, 8, 8, 128, 4);
                // cta<4,64,512> warp<8,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(2, 2, true, 4, 64, 512, 8, 32, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 4, 64, 512, 8, 32, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 4, 64, 512, 8, 32, 128, 8, 8, 128, 4);
                // cta<4,80,512> warp<8,40,128> mma<8,8,128>   WARPS[1x4]
                TEST(2, 2, true, 4, 80, 512, 8, 40, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 4, 80, 512, 8, 40, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 4, 80, 512, 8, 40, 128, 8, 8, 128, 4);
                // cta<4,96,512> warp<8,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(2, 2, true, 4, 96, 512, 8, 48, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 4, 96, 512, 8, 48, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 4, 96, 512, 8, 48, 128, 8, 8, 128, 4);
                // cta<4,112,512> warp<8,56,128> mma<8,8,128>   WARPS[1x4]
                TEST(2, 2, true, 4, 112, 512, 8, 56, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 4, 112, 512, 8, 56, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 4, 112, 512, 8, 56, 128, 8, 8, 128, 4);
                // cta<4,128,512> warp<8,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(2, 2, true, 4, 128, 512, 8, 64, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 4, 128, 512, 8, 64, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 4, 128, 512, 8, 64, 128, 8, 8, 128, 4);
                // cta<4,256,512> warp<8,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(2, 2, true, 4, 256, 512, 8, 128, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 4, 256, 512, 8, 128, 128, 8, 8, 128, 3);
                // cta<8,32,512> warp<16,16,128> mma<8,8,128>   WARPS[1x4]
                TEST(2, 2, true, 8, 32, 512, 16, 16, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 32, 512, 16, 16, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 32, 512, 16, 16, 128, 8, 8, 128, 4);
                // cta<8,48,512> warp<16,24,128> mma<8,8,128>   WARPS[1x4]
                TEST(2, 2, true, 8, 48, 512, 16, 24, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 48, 512, 16, 24, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 48, 512, 16, 24, 128, 8, 8, 128, 4);
                // cta<8,64,512> warp<16,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(2, 2, true, 8, 64, 512, 16, 32, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 64, 512, 16, 32, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 64, 512, 16, 32, 128, 8, 8, 128, 4);
                // cta<8,80,512> warp<16,40,128> mma<8,8,128>   WARPS[1x4]
                TEST(2, 2, true, 8, 80, 512, 16, 40, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 80, 512, 16, 40, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 80, 512, 16, 40, 128, 8, 8, 128, 4);
                // cta<8,96,512> warp<16,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(2, 2, true, 8, 96, 512, 16, 48, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 96, 512, 16, 48, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 96, 512, 16, 48, 128, 8, 8, 128, 4);
                // cta<8,112,512> warp<16,56,128> mma<8,8,128>   WARPS[1x4]
                TEST(2, 2, true, 8, 112, 512, 16, 56, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 112, 512, 16, 56, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 112, 512, 16, 56, 128, 8, 8, 128, 4);
                // cta<8,128,512> warp<16,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(2, 2, true, 8, 128, 512, 16, 64, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 128, 512, 16, 64, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 128, 512, 16, 64, 128, 8, 8, 128, 4);
                // cta<8,256,512> warp<16,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(2, 2, true, 8, 256, 512, 16, 128, 128, 8, 8, 128, 2);
                // cta<8,32,512> warp<8,64,128> mma<8,8,128>   WARPS[2x1]
                TEST(2, 2, true, 8, 32, 512, 8, 64, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 32, 512, 8, 64, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 32, 512, 8, 64, 128, 8, 8, 128, 4);
                // cta<8,48,512> warp<8,96,128> mma<8,8,128>   WARPS[2x1]
                TEST(2, 2, true, 8, 48, 512, 8, 96, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 48, 512, 8, 96, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 48, 512, 8, 96, 128, 8, 8, 128, 4);
                // cta<8,64,512> warp<8,128,128> mma<8,8,128>   WARPS[2x1]
                TEST(2, 2, true, 8, 64, 512, 8, 128, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 64, 512, 8, 128, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 64, 512, 8, 128, 128, 8, 8, 128, 4);
                // cta<8,32,512> warp<8,32,128> mma<8,8,128>   WARPS[2x2]
                TEST(2, 2, true, 8, 32, 512, 8, 32, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 32, 512, 8, 32, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 32, 512, 8, 32, 128, 8, 8, 128, 4);
                // cta<8,48,512> warp<8,48,128> mma<8,8,128>   WARPS[2x2]
                TEST(2, 2, true, 8, 48, 512, 8, 48, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 48, 512, 8, 48, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 48, 512, 8, 48, 128, 8, 8, 128, 4);
                // cta<8,64,512> warp<8,64,128> mma<8,8,128>   WARPS[2x2]
                TEST(2, 2, true, 8, 64, 512, 8, 64, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 64, 512, 8, 64, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 64, 512, 8, 64, 128, 8, 8, 128, 4);
                // cta<8,80,512> warp<8,80,128> mma<8,8,128>   WARPS[2x2]
                TEST(2, 2, true, 8, 80, 512, 8, 80, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 80, 512, 8, 80, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 80, 512, 8, 80, 128, 8, 8, 128, 4);
                // cta<8,96,512> warp<8,96,128> mma<8,8,128>   WARPS[2x2]
                TEST(2, 2, true, 8, 96, 512, 8, 96, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 96, 512, 8, 96, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 96, 512, 8, 96, 128, 8, 8, 128, 4);
                // cta<8,112,512> warp<8,112,128> mma<8,8,128>   WARPS[2x2]
                TEST(2, 2, true, 8, 112, 512, 8, 112, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 112, 512, 8, 112, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 112, 512, 8, 112, 128, 8, 8, 128, 4);
                // cta<8,128,512> warp<8,128,128> mma<8,8,128>   WARPS[2x2]
                TEST(2, 2, true, 8, 128, 512, 8, 128, 128, 8, 8, 128, 2);
                TEST(2, 2, true, 8, 128, 512, 8, 128, 128, 8, 8, 128, 3);
                TEST(2, 2, true, 8, 128, 512, 8, 128, 128, 8, 8, 128, 4);
            } else {
            }
            break;
        #endif
        default:
            printf("unsupport w%da%d\n", w_bits, x_bits);
        }
        break;
    case 3:
        switch (w_bits) {
        #ifdef W3A3
        case 3:
            if (quant_sign) {
                ////// W3A3 int
                // cta<2,32,256> warp<8,48,128> mma<8,8,128>   WARPS[1x2]
                TEST(3, 3, true, 2, 32, 256, 8, 48, 128, 8, 8, 128, 2);
                TEST(3, 3, true, 2, 32, 256, 8, 48, 128, 8, 8, 128, 3);
                TEST(3, 3, true, 2, 32, 256, 8, 48, 128, 8, 8, 128, 4);
                // cta<2,48,256> warp<8,72,128> mma<8,8,128>   WARPS[1x2]
                TEST(3, 3, true, 2, 48, 256, 8, 72, 128, 8, 8, 128, 2);
                TEST(3, 3, true, 2, 48, 256, 8, 72, 128, 8, 8, 128, 3);
                TEST(3, 3, true, 2, 48, 256, 8, 72, 128, 8, 8, 128, 4);
                // cta<2,64,256> warp<8,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(3, 3, true, 2, 64, 256, 8, 96, 128, 8, 8, 128, 2);
                TEST(3, 3, true, 2, 64, 256, 8, 96, 128, 8, 8, 128, 3);
                TEST(3, 3, true, 2, 64, 256, 8, 96, 128, 8, 8, 128, 4);
                // cta<2,80,256> warp<8,120,128> mma<8,8,128>   WARPS[1x2]
                TEST(3, 3, true, 2, 80, 256, 8, 120, 128, 8, 8, 128, 2);
                TEST(3, 3, true, 2, 80, 256, 8, 120, 128, 8, 8, 128, 3);
                TEST(3, 3, true, 2, 80, 256, 8, 120, 128, 8, 8, 128, 4);
                // cta<8,32,256> warp<24,48,128> mma<8,8,128>   WARPS[1x2]
                TEST(3, 3, true, 8, 32, 256, 24, 48, 128, 8, 8, 128, 2);
                // cta<8,48,256> warp<24,72,128> mma<8,8,128>   WARPS[1x2]
                TEST(3, 3, true, 8, 48, 256, 24, 72, 128, 8, 8, 128, 2);
                // cta<8,64,256> warp<24,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(3, 3, true, 8, 64, 256, 24, 96, 128, 8, 8, 128, 2);
                // cta<8,80,256> warp<24,120,128> mma<8,8,128>   WARPS[1x2]
                TEST(3, 3, true, 8, 80, 256, 24, 120, 128, 8, 8, 128, 2);
                // cta<2,32,256> warp<8,24,128> mma<8,8,128>   WARPS[1x4]
                TEST(3, 3, true, 2, 32, 256, 8, 24, 128, 8, 8, 128, 2);
                TEST(3, 3, true, 2, 32, 256, 8, 24, 128, 8, 8, 128, 3);
                TEST(3, 3, true, 2, 32, 256, 8, 24, 128, 8, 8, 128, 4);
                // cta<2,64,256> warp<8,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(3, 3, true, 2, 64, 256, 8, 48, 128, 8, 8, 128, 2);
                TEST(3, 3, true, 2, 64, 256, 8, 48, 128, 8, 8, 128, 3);
                TEST(3, 3, true, 2, 64, 256, 8, 48, 128, 8, 8, 128, 4);
                // cta<2,96,256> warp<8,72,128> mma<8,8,128>   WARPS[1x4]
                TEST(3, 3, true, 2, 96, 256, 8, 72, 128, 8, 8, 128, 2);
                TEST(3, 3, true, 2, 96, 256, 8, 72, 128, 8, 8, 128, 3);
                TEST(3, 3, true, 2, 96, 256, 8, 72, 128, 8, 8, 128, 4);
                // cta<2,128,256> warp<8,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(3, 3, true, 2, 128, 256, 8, 96, 128, 8, 8, 128, 2);
                TEST(3, 3, true, 2, 128, 256, 8, 96, 128, 8, 8, 128, 3);
                TEST(3, 3, true, 2, 128, 256, 8, 96, 128, 8, 8, 128, 4);
                // cta<8,32,256> warp<24,24,128> mma<8,8,128>   WARPS[1x4]
                TEST(3, 3, true, 8, 32, 256, 24, 24, 128, 8, 8, 128, 2);
                // cta<8,64,256> warp<24,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(3, 3, true, 8, 64, 256, 24, 48, 128, 8, 8, 128, 2);
                // cta<8,96,256> warp<24,72,128> mma<8,8,128>   WARPS[1x4]
                TEST(3, 3, true, 8, 96, 256, 24, 72, 128, 8, 8, 128, 2);
                // cta<8,128,256> warp<24,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(3, 3, true, 8, 128, 256, 24, 96, 128, 8, 8, 128, 2);
                // cta<2,32,384> warp<8,48,128> mma<8,8,128>   WARPS[1x2]
                TEST(3, 3, true, 2, 32, 384, 8, 48, 128, 8, 8, 128, 2);
                TEST(3, 3, true, 2, 32, 384, 8, 48, 128, 8, 8, 128, 3);
                TEST(3, 3, true, 2, 32, 384, 8, 48, 128, 8, 8, 128, 4);
                // cta<2,48,384> warp<8,72,128> mma<8,8,128>   WARPS[1x2]
                TEST(3, 3, true, 2, 48, 384, 8, 72, 128, 8, 8, 128, 2);
                TEST(3, 3, true, 2, 48, 384, 8, 72, 128, 8, 8, 128, 3);
                TEST(3, 3, true, 2, 48, 384, 8, 72, 128, 8, 8, 128, 4);
                // cta<2,64,384> warp<8,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(3, 3, true, 2, 64, 384, 8, 96, 128, 8, 8, 128, 2);
                TEST(3, 3, true, 2, 64, 384, 8, 96, 128, 8, 8, 128, 3);
                TEST(3, 3, true, 2, 64, 384, 8, 96, 128, 8, 8, 128, 4);
                // cta<2,80,384> warp<8,120,128> mma<8,8,128>   WARPS[1x2]
                TEST(3, 3, true, 2, 80, 384, 8, 120, 128, 8, 8, 128, 2);
                TEST(3, 3, true, 2, 80, 384, 8, 120, 128, 8, 8, 128, 3);
                TEST(3, 3, true, 2, 80, 384, 8, 120, 128, 8, 8, 128, 4);
                // cta<8,32,384> warp<24,48,128> mma<8,8,128>   WARPS[1x2]
                TEST(3, 3, true, 8, 32, 384, 24, 48, 128, 8, 8, 128, 2);
                TEST(3, 3, true, 8, 32, 384, 24, 48, 128, 8, 8, 128, 3);
                TEST(3, 3, true, 8, 32, 384, 24, 48, 128, 8, 8, 128, 4);
                // cta<8,48,384> warp<24,72,128> mma<8,8,128>   WARPS[1x2]
                TEST(3, 3, true, 8, 48, 384, 24, 72, 128, 8, 8, 128, 2);
                TEST(3, 3, true, 8, 48, 384, 24, 72, 128, 8, 8, 128, 3);
                TEST(3, 3, true, 8, 48, 384, 24, 72, 128, 8, 8, 128, 4);
                // cta<8,64,384> warp<24,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(3, 3, true, 8, 64, 384, 24, 96, 128, 8, 8, 128, 2);
                TEST(3, 3, true, 8, 64, 384, 24, 96, 128, 8, 8, 128, 3);
                TEST(3, 3, true, 8, 64, 384, 24, 96, 128, 8, 8, 128, 4);
                // cta<8,80,384> warp<24,120,128> mma<8,8,128>   WARPS[1x2]
                TEST(3, 3, true, 8, 80, 384, 24, 120, 128, 8, 8, 128, 2);
                TEST(3, 3, true, 8, 80, 384, 24, 120, 128, 8, 8, 128, 3);
                TEST(3, 3, true, 8, 80, 384, 24, 120, 128, 8, 8, 128, 4);
                // cta<2,32,384> warp<8,24,128> mma<8,8,128>   WARPS[1x4]
                TEST(3, 3, true, 2, 32, 384, 8, 24, 128, 8, 8, 128, 2);
                TEST(3, 3, true, 2, 32, 384, 8, 24, 128, 8, 8, 128, 3);
                TEST(3, 3, true, 2, 32, 384, 8, 24, 128, 8, 8, 128, 4);
                // cta<2,64,384> warp<8,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(3, 3, true, 2, 64, 384, 8, 48, 128, 8, 8, 128, 2);
                TEST(3, 3, true, 2, 64, 384, 8, 48, 128, 8, 8, 128, 3);
                TEST(3, 3, true, 2, 64, 384, 8, 48, 128, 8, 8, 128, 4);
                // cta<2,96,384> warp<8,72,128> mma<8,8,128>   WARPS[1x4]
                TEST(3, 3, true, 2, 96, 384, 8, 72, 128, 8, 8, 128, 2);
                TEST(3, 3, true, 2, 96, 384, 8, 72, 128, 8, 8, 128, 3);
                TEST(3, 3, true, 2, 96, 384, 8, 72, 128, 8, 8, 128, 4);
                // cta<2,128,384> warp<8,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(3, 3, true, 2, 128, 384, 8, 96, 128, 8, 8, 128, 2);
                TEST(3, 3, true, 2, 128, 384, 8, 96, 128, 8, 8, 128, 3);
                TEST(3, 3, true, 2, 128, 384, 8, 96, 128, 8, 8, 128, 4);
                // cta<8,32,384> warp<24,24,128> mma<8,8,128>   WARPS[1x4]
                TEST(3, 3, true, 8, 32, 384, 24, 24, 128, 8, 8, 128, 2);
                TEST(3, 3, true, 8, 32, 384, 24, 24, 128, 8, 8, 128, 3);
                TEST(3, 3, true, 8, 32, 384, 24, 24, 128, 8, 8, 128, 4);
                // cta<8,64,384> warp<24,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(3, 3, true, 8, 64, 384, 24, 48, 128, 8, 8, 128, 2);
                TEST(3, 3, true, 8, 64, 384, 24, 48, 128, 8, 8, 128, 3);
                TEST(3, 3, true, 8, 64, 384, 24, 48, 128, 8, 8, 128, 4);
                // cta<8,96,384> warp<24,72,128> mma<8,8,128>   WARPS[1x4]
                TEST(3, 3, true, 8, 96, 384, 24, 72, 128, 8, 8, 128, 2);
                TEST(3, 3, true, 8, 96, 384, 24, 72, 128, 8, 8, 128, 3);
                TEST(3, 3, true, 8, 96, 384, 24, 72, 128, 8, 8, 128, 4);
                // cta<8,128,384> warp<24,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(3, 3, true, 8, 128, 384, 24, 96, 128, 8, 8, 128, 2);
                TEST(3, 3, true, 8, 128, 384, 24, 96, 128, 8, 8, 128, 3);
                TEST(3, 3, true, 8, 128, 384, 24, 96, 128, 8, 8, 128, 4);
                // cta<2,32,512> warp<8,48,128> mma<8,8,128>   WARPS[1x2]
                TEST(3, 3, true, 2, 32, 512, 8, 48, 128, 8, 8, 128, 2);
                TEST(3, 3, true, 2, 32, 512, 8, 48, 128, 8, 8, 128, 3);
                TEST(3, 3, true, 2, 32, 512, 8, 48, 128, 8, 8, 128, 4);
                // cta<2,48,512> warp<8,72,128> mma<8,8,128>   WARPS[1x2]
                TEST(3, 3, true, 2, 48, 512, 8, 72, 128, 8, 8, 128, 2);
                TEST(3, 3, true, 2, 48, 512, 8, 72, 128, 8, 8, 128, 3);
                TEST(3, 3, true, 2, 48, 512, 8, 72, 128, 8, 8, 128, 4);
                // cta<2,64,512> warp<8,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(3, 3, true, 2, 64, 512, 8, 96, 128, 8, 8, 128, 2);
                TEST(3, 3, true, 2, 64, 512, 8, 96, 128, 8, 8, 128, 3);
                TEST(3, 3, true, 2, 64, 512, 8, 96, 128, 8, 8, 128, 4);
                // cta<2,80,512> warp<8,120,128> mma<8,8,128>   WARPS[1x2]
                TEST(3, 3, true, 2, 80, 512, 8, 120, 128, 8, 8, 128, 2);
                TEST(3, 3, true, 2, 80, 512, 8, 120, 128, 8, 8, 128, 3);
                TEST(3, 3, true, 2, 80, 512, 8, 120, 128, 8, 8, 128, 4);
                // cta<8,32,512> warp<24,48,128> mma<8,8,128>   WARPS[1x2]
                TEST(3, 3, true, 8, 32, 512, 24, 48, 128, 8, 8, 128, 2);
                TEST(3, 3, true, 8, 32, 512, 24, 48, 128, 8, 8, 128, 3);
                TEST(3, 3, true, 8, 32, 512, 24, 48, 128, 8, 8, 128, 4);
                // cta<8,48,512> warp<24,72,128> mma<8,8,128>   WARPS[1x2]
                TEST(3, 3, true, 8, 48, 512, 24, 72, 128, 8, 8, 128, 2);
                TEST(3, 3, true, 8, 48, 512, 24, 72, 128, 8, 8, 128, 3);
                TEST(3, 3, true, 8, 48, 512, 24, 72, 128, 8, 8, 128, 4);
                // cta<8,64,512> warp<24,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(3, 3, true, 8, 64, 512, 24, 96, 128, 8, 8, 128, 2);
                TEST(3, 3, true, 8, 64, 512, 24, 96, 128, 8, 8, 128, 3);
                TEST(3, 3, true, 8, 64, 512, 24, 96, 128, 8, 8, 128, 4);
                // cta<8,80,512> warp<24,120,128> mma<8,8,128>   WARPS[1x2]
                TEST(3, 3, true, 8, 80, 512, 24, 120, 128, 8, 8, 128, 2);
                TEST(3, 3, true, 8, 80, 512, 24, 120, 128, 8, 8, 128, 3);
                TEST(3, 3, true, 8, 80, 512, 24, 120, 128, 8, 8, 128, 4);
                // cta<2,32,512> warp<8,24,128> mma<8,8,128>   WARPS[1x4]
                TEST(3, 3, true, 2, 32, 512, 8, 24, 128, 8, 8, 128, 2);
                TEST(3, 3, true, 2, 32, 512, 8, 24, 128, 8, 8, 128, 3);
                TEST(3, 3, true, 2, 32, 512, 8, 24, 128, 8, 8, 128, 4);
                // cta<2,64,512> warp<8,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(3, 3, true, 2, 64, 512, 8, 48, 128, 8, 8, 128, 2);
                TEST(3, 3, true, 2, 64, 512, 8, 48, 128, 8, 8, 128, 3);
                TEST(3, 3, true, 2, 64, 512, 8, 48, 128, 8, 8, 128, 4);
                // cta<2,96,512> warp<8,72,128> mma<8,8,128>   WARPS[1x4]
                TEST(3, 3, true, 2, 96, 512, 8, 72, 128, 8, 8, 128, 2);
                TEST(3, 3, true, 2, 96, 512, 8, 72, 128, 8, 8, 128, 3);
                TEST(3, 3, true, 2, 96, 512, 8, 72, 128, 8, 8, 128, 4);
                // cta<2,128,512> warp<8,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(3, 3, true, 2, 128, 512, 8, 96, 128, 8, 8, 128, 2);
                TEST(3, 3, true, 2, 128, 512, 8, 96, 128, 8, 8, 128, 3);
                TEST(3, 3, true, 2, 128, 512, 8, 96, 128, 8, 8, 128, 4);
                // cta<8,32,512> warp<24,24,128> mma<8,8,128>   WARPS[1x4]
                TEST(3, 3, true, 8, 32, 512, 24, 24, 128, 8, 8, 128, 2);
                TEST(3, 3, true, 8, 32, 512, 24, 24, 128, 8, 8, 128, 3);
                TEST(3, 3, true, 8, 32, 512, 24, 24, 128, 8, 8, 128, 4);
                // cta<8,64,512> warp<24,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(3, 3, true, 8, 64, 512, 24, 48, 128, 8, 8, 128, 2);
                TEST(3, 3, true, 8, 64, 512, 24, 48, 128, 8, 8, 128, 3);
                TEST(3, 3, true, 8, 64, 512, 24, 48, 128, 8, 8, 128, 4);
                // cta<8,96,512> warp<24,72,128> mma<8,8,128>   WARPS[1x4]
                TEST(3, 3, true, 8, 96, 512, 24, 72, 128, 8, 8, 128, 2);
                TEST(3, 3, true, 8, 96, 512, 24, 72, 128, 8, 8, 128, 3);
                TEST(3, 3, true, 8, 96, 512, 24, 72, 128, 8, 8, 128, 4);
                // cta<8,128,512> warp<24,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(3, 3, true, 8, 128, 512, 24, 96, 128, 8, 8, 128, 2);
                TEST(3, 3, true, 8, 128, 512, 24, 96, 128, 8, 8, 128, 3);
            } else {
            }
            break;
        #endif
        default:
            printf("unsupport w%da%d\n", w_bits, x_bits);
        }
        break;
    case 4:
        switch (w_bits) {
        #ifdef W2A4
        case 2:
            if (quant_sign) {
                ////// W2A4 int
                // cta<2,32,256> warp<8,32,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 2, 32, 256, 8, 32, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 2, 32, 256, 8, 32, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 2, 32, 256, 8, 32, 128, 8, 8, 128, 4);
                // cta<2,48,256> warp<8,48,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 2, 48, 256, 8, 48, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 2, 48, 256, 8, 48, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 2, 48, 256, 8, 48, 128, 8, 8, 128, 4);
                // cta<2,64,256> warp<8,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 2, 64, 256, 8, 64, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 2, 64, 256, 8, 64, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 2, 64, 256, 8, 64, 128, 8, 8, 128, 4);
                // cta<2,80,256> warp<8,80,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 2, 80, 256, 8, 80, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 2, 80, 256, 8, 80, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 2, 80, 256, 8, 80, 128, 8, 8, 128, 4);
                // cta<2,96,256> warp<8,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 2, 96, 256, 8, 96, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 2, 96, 256, 8, 96, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 2, 96, 256, 8, 96, 128, 8, 8, 128, 4);
                // cta<2,112,256> warp<8,112,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 2, 112, 256, 8, 112, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 2, 112, 256, 8, 112, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 2, 112, 256, 8, 112, 128, 8, 8, 128, 4);
                // cta<2,128,256> warp<8,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 2, 128, 256, 8, 128, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 2, 128, 256, 8, 128, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 2, 128, 256, 8, 128, 128, 8, 8, 128, 4);
                // cta<4,32,256> warp<16,32,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 4, 32, 256, 16, 32, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 32, 256, 16, 32, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 32, 256, 16, 32, 128, 8, 8, 128, 4);
                // cta<4,48,256> warp<16,48,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 4, 48, 256, 16, 48, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 48, 256, 16, 48, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 48, 256, 16, 48, 128, 8, 8, 128, 4);
                // cta<4,64,256> warp<16,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 4, 64, 256, 16, 64, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 64, 256, 16, 64, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 64, 256, 16, 64, 128, 8, 8, 128, 4);
                // cta<4,80,256> warp<16,80,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 4, 80, 256, 16, 80, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 80, 256, 16, 80, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 80, 256, 16, 80, 128, 8, 8, 128, 4);
                // cta<4,96,256> warp<16,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 4, 96, 256, 16, 96, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 96, 256, 16, 96, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 96, 256, 16, 96, 128, 8, 8, 128, 4);
                // cta<4,112,256> warp<16,112,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 4, 112, 256, 16, 112, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 112, 256, 16, 112, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 112, 256, 16, 112, 128, 8, 8, 128, 4);
                // cta<4,128,256> warp<16,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 4, 128, 256, 16, 128, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 128, 256, 16, 128, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 128, 256, 16, 128, 128, 8, 8, 128, 4);
                // cta<8,32,256> warp<32,32,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 8, 32, 256, 32, 32, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 32, 256, 32, 32, 128, 8, 8, 128, 3);
                // cta<8,48,256> warp<32,48,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 8, 48, 256, 32, 48, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 48, 256, 32, 48, 128, 8, 8, 128, 3);
                // cta<8,64,256> warp<32,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 8, 64, 256, 32, 64, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 64, 256, 32, 64, 128, 8, 8, 128, 3);
                // cta<8,80,256> warp<32,80,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 8, 80, 256, 32, 80, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 80, 256, 32, 80, 128, 8, 8, 128, 3);
                // cta<8,96,256> warp<32,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 8, 96, 256, 32, 96, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 96, 256, 32, 96, 128, 8, 8, 128, 3);
                // cta<8,112,256> warp<32,112,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 8, 112, 256, 32, 112, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 112, 256, 32, 112, 128, 8, 8, 128, 3);
                // cta<8,128,256> warp<32,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 8, 128, 256, 32, 128, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 128, 256, 32, 128, 128, 8, 8, 128, 3);
                // cta<2,32,256> warp<8,16,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 2, 32, 256, 8, 16, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 2, 32, 256, 8, 16, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 2, 32, 256, 8, 16, 128, 8, 8, 128, 4);
                // cta<2,48,256> warp<8,24,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 2, 48, 256, 8, 24, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 2, 48, 256, 8, 24, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 2, 48, 256, 8, 24, 128, 8, 8, 128, 4);
                // cta<2,64,256> warp<8,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 2, 64, 256, 8, 32, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 2, 64, 256, 8, 32, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 2, 64, 256, 8, 32, 128, 8, 8, 128, 4);
                // cta<2,80,256> warp<8,40,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 2, 80, 256, 8, 40, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 2, 80, 256, 8, 40, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 2, 80, 256, 8, 40, 128, 8, 8, 128, 4);
                // cta<2,96,256> warp<8,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 2, 96, 256, 8, 48, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 2, 96, 256, 8, 48, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 2, 96, 256, 8, 48, 128, 8, 8, 128, 4);
                // cta<2,112,256> warp<8,56,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 2, 112, 256, 8, 56, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 2, 112, 256, 8, 56, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 2, 112, 256, 8, 56, 128, 8, 8, 128, 4);
                // cta<2,128,256> warp<8,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 2, 128, 256, 8, 64, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 2, 128, 256, 8, 64, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 2, 128, 256, 8, 64, 128, 8, 8, 128, 4);
                // cta<2,256,256> warp<8,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 2, 256, 256, 8, 128, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 2, 256, 256, 8, 128, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 2, 256, 256, 8, 128, 128, 8, 8, 128, 4);
                // cta<4,32,256> warp<16,16,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 4, 32, 256, 16, 16, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 32, 256, 16, 16, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 32, 256, 16, 16, 128, 8, 8, 128, 4);
                // cta<4,48,256> warp<16,24,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 4, 48, 256, 16, 24, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 48, 256, 16, 24, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 48, 256, 16, 24, 128, 8, 8, 128, 4);
                // cta<4,64,256> warp<16,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 4, 64, 256, 16, 32, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 64, 256, 16, 32, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 64, 256, 16, 32, 128, 8, 8, 128, 4);
                // cta<4,80,256> warp<16,40,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 4, 80, 256, 16, 40, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 80, 256, 16, 40, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 80, 256, 16, 40, 128, 8, 8, 128, 4);
                // cta<4,96,256> warp<16,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 4, 96, 256, 16, 48, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 96, 256, 16, 48, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 96, 256, 16, 48, 128, 8, 8, 128, 4);
                // cta<4,112,256> warp<16,56,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 4, 112, 256, 16, 56, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 112, 256, 16, 56, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 112, 256, 16, 56, 128, 8, 8, 128, 4);
                // cta<4,128,256> warp<16,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 4, 128, 256, 16, 64, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 128, 256, 16, 64, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 128, 256, 16, 64, 128, 8, 8, 128, 4);
                // cta<4,256,256> warp<16,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 4, 256, 256, 16, 128, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 256, 256, 16, 128, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 256, 256, 16, 128, 128, 8, 8, 128, 4);
                // cta<8,32,256> warp<32,16,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 8, 32, 256, 32, 16, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 32, 256, 32, 16, 128, 8, 8, 128, 3);
                // cta<8,48,256> warp<32,24,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 8, 48, 256, 32, 24, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 48, 256, 32, 24, 128, 8, 8, 128, 3);
                // cta<8,64,256> warp<32,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 8, 64, 256, 32, 32, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 64, 256, 32, 32, 128, 8, 8, 128, 3);
                // cta<8,80,256> warp<32,40,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 8, 80, 256, 32, 40, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 80, 256, 32, 40, 128, 8, 8, 128, 3);
                // cta<8,96,256> warp<32,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 8, 96, 256, 32, 48, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 96, 256, 32, 48, 128, 8, 8, 128, 3);
                // cta<8,112,256> warp<32,56,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 8, 112, 256, 32, 56, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 112, 256, 32, 56, 128, 8, 8, 128, 3);
                // cta<8,128,256> warp<32,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 8, 128, 256, 32, 64, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 128, 256, 32, 64, 128, 8, 8, 128, 3);
                // cta<8,256,256> warp<32,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 8, 256, 256, 32, 128, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 256, 256, 32, 128, 128, 8, 8, 128, 3);
                // cta<4,32,256> warp<8,64,128> mma<8,8,128>   WARPS[2x1]
                TEST(4, 2, true, 4, 32, 256, 8, 64, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 32, 256, 8, 64, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 32, 256, 8, 64, 128, 8, 8, 128, 4);
                // cta<4,48,256> warp<8,96,128> mma<8,8,128>   WARPS[2x1]
                TEST(4, 2, true, 4, 48, 256, 8, 96, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 48, 256, 8, 96, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 48, 256, 8, 96, 128, 8, 8, 128, 4);
                // cta<4,64,256> warp<8,128,128> mma<8,8,128>   WARPS[2x1]
                TEST(4, 2, true, 4, 64, 256, 8, 128, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 64, 256, 8, 128, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 64, 256, 8, 128, 128, 8, 8, 128, 4);
                // cta<8,32,256> warp<16,64,128> mma<8,8,128>   WARPS[2x1]
                TEST(4, 2, true, 8, 32, 256, 16, 64, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 32, 256, 16, 64, 128, 8, 8, 128, 3);
                // cta<8,48,256> warp<16,96,128> mma<8,8,128>   WARPS[2x1]
                TEST(4, 2, true, 8, 48, 256, 16, 96, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 48, 256, 16, 96, 128, 8, 8, 128, 3);
                // cta<8,64,256> warp<16,128,128> mma<8,8,128>   WARPS[2x1]
                TEST(4, 2, true, 8, 64, 256, 16, 128, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 64, 256, 16, 128, 128, 8, 8, 128, 3);
                // cta<4,32,256> warp<8,32,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 2, true, 4, 32, 256, 8, 32, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 32, 256, 8, 32, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 32, 256, 8, 32, 128, 8, 8, 128, 4);
                // cta<4,48,256> warp<8,48,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 2, true, 4, 48, 256, 8, 48, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 48, 256, 8, 48, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 48, 256, 8, 48, 128, 8, 8, 128, 4);
                // cta<4,64,256> warp<8,64,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 2, true, 4, 64, 256, 8, 64, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 64, 256, 8, 64, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 64, 256, 8, 64, 128, 8, 8, 128, 4);
                // cta<4,80,256> warp<8,80,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 2, true, 4, 80, 256, 8, 80, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 80, 256, 8, 80, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 80, 256, 8, 80, 128, 8, 8, 128, 4);
                // cta<4,96,256> warp<8,96,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 2, true, 4, 96, 256, 8, 96, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 96, 256, 8, 96, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 96, 256, 8, 96, 128, 8, 8, 128, 4);
                // cta<4,112,256> warp<8,112,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 2, true, 4, 112, 256, 8, 112, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 112, 256, 8, 112, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 112, 256, 8, 112, 128, 8, 8, 128, 4);
                // cta<4,128,256> warp<8,128,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 2, true, 4, 128, 256, 8, 128, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 128, 256, 8, 128, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 128, 256, 8, 128, 128, 8, 8, 128, 4);
                // cta<8,32,256> warp<16,32,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 2, true, 8, 32, 256, 16, 32, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 32, 256, 16, 32, 128, 8, 8, 128, 3);
                // cta<8,48,256> warp<16,48,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 2, true, 8, 48, 256, 16, 48, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 48, 256, 16, 48, 128, 8, 8, 128, 3);
                // cta<8,64,256> warp<16,64,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 2, true, 8, 64, 256, 16, 64, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 64, 256, 16, 64, 128, 8, 8, 128, 3);
                // cta<8,80,256> warp<16,80,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 2, true, 8, 80, 256, 16, 80, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 80, 256, 16, 80, 128, 8, 8, 128, 3);
                // cta<8,96,256> warp<16,96,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 2, true, 8, 96, 256, 16, 96, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 96, 256, 16, 96, 128, 8, 8, 128, 3);
                // cta<8,112,256> warp<16,112,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 2, true, 8, 112, 256, 16, 112, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 112, 256, 16, 112, 128, 8, 8, 128, 3);
                // cta<8,128,256> warp<16,128,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 2, true, 8, 128, 256, 16, 128, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 128, 256, 16, 128, 128, 8, 8, 128, 3);
                // cta<8,32,256> warp<8,64,128> mma<8,8,128>   WARPS[4x1]
                TEST(4, 2, true, 8, 32, 256, 8, 64, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 32, 256, 8, 64, 128, 8, 8, 128, 3);
                // cta<8,48,256> warp<8,96,128> mma<8,8,128>   WARPS[4x1]
                TEST(4, 2, true, 8, 48, 256, 8, 96, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 48, 256, 8, 96, 128, 8, 8, 128, 3);
                // cta<8,64,256> warp<8,128,128> mma<8,8,128>   WARPS[4x1]
                TEST(4, 2, true, 8, 64, 256, 8, 128, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 64, 256, 8, 128, 128, 8, 8, 128, 3);
                // cta<2,32,384> warp<8,32,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 2, 32, 384, 8, 32, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 2, 32, 384, 8, 32, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 2, 32, 384, 8, 32, 128, 8, 8, 128, 4);
                // cta<2,48,384> warp<8,48,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 2, 48, 384, 8, 48, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 2, 48, 384, 8, 48, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 2, 48, 384, 8, 48, 128, 8, 8, 128, 4);
                // cta<2,64,384> warp<8,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 2, 64, 384, 8, 64, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 2, 64, 384, 8, 64, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 2, 64, 384, 8, 64, 128, 8, 8, 128, 4);
                // cta<2,80,384> warp<8,80,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 2, 80, 384, 8, 80, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 2, 80, 384, 8, 80, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 2, 80, 384, 8, 80, 128, 8, 8, 128, 4);
                // cta<2,96,384> warp<8,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 2, 96, 384, 8, 96, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 2, 96, 384, 8, 96, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 2, 96, 384, 8, 96, 128, 8, 8, 128, 4);
                // cta<2,112,384> warp<8,112,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 2, 112, 384, 8, 112, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 2, 112, 384, 8, 112, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 2, 112, 384, 8, 112, 128, 8, 8, 128, 4);
                // cta<2,128,384> warp<8,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 2, 128, 384, 8, 128, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 2, 128, 384, 8, 128, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 2, 128, 384, 8, 128, 128, 8, 8, 128, 4);
                // cta<4,32,384> warp<16,32,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 4, 32, 384, 16, 32, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 32, 384, 16, 32, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 32, 384, 16, 32, 128, 8, 8, 128, 4);
                // cta<4,48,384> warp<16,48,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 4, 48, 384, 16, 48, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 48, 384, 16, 48, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 48, 384, 16, 48, 128, 8, 8, 128, 4);
                // cta<4,64,384> warp<16,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 4, 64, 384, 16, 64, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 64, 384, 16, 64, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 64, 384, 16, 64, 128, 8, 8, 128, 4);
                // cta<4,80,384> warp<16,80,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 4, 80, 384, 16, 80, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 80, 384, 16, 80, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 80, 384, 16, 80, 128, 8, 8, 128, 4);
                // cta<4,96,384> warp<16,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 4, 96, 384, 16, 96, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 96, 384, 16, 96, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 96, 384, 16, 96, 128, 8, 8, 128, 4);
                // cta<4,112,384> warp<16,112,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 4, 112, 384, 16, 112, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 112, 384, 16, 112, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 112, 384, 16, 112, 128, 8, 8, 128, 4);
                // cta<4,128,384> warp<16,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 4, 128, 384, 16, 128, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 128, 384, 16, 128, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 128, 384, 16, 128, 128, 8, 8, 128, 4);
                // cta<8,32,384> warp<32,32,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 8, 32, 384, 32, 32, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 32, 384, 32, 32, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 8, 32, 384, 32, 32, 128, 8, 8, 128, 4);
                // cta<8,48,384> warp<32,48,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 8, 48, 384, 32, 48, 128, 8, 8, 128, 2);
                // cta<8,64,384> warp<32,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 8, 64, 384, 32, 64, 128, 8, 8, 128, 2);
                // cta<8,80,384> warp<32,80,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 8, 80, 384, 32, 80, 128, 8, 8, 128, 2);
                // cta<8,96,384> warp<32,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 8, 96, 384, 32, 96, 128, 8, 8, 128, 2);
                // cta<8,112,384> warp<32,112,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 8, 112, 384, 32, 112, 128, 8, 8, 128, 2);
                // cta<8,128,384> warp<32,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 8, 128, 384, 32, 128, 128, 8, 8, 128, 2);
                // cta<2,32,384> warp<8,16,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 2, 32, 384, 8, 16, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 2, 32, 384, 8, 16, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 2, 32, 384, 8, 16, 128, 8, 8, 128, 4);
                // cta<2,48,384> warp<8,24,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 2, 48, 384, 8, 24, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 2, 48, 384, 8, 24, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 2, 48, 384, 8, 24, 128, 8, 8, 128, 4);
                // cta<2,64,384> warp<8,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 2, 64, 384, 8, 32, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 2, 64, 384, 8, 32, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 2, 64, 384, 8, 32, 128, 8, 8, 128, 4);
                // cta<2,80,384> warp<8,40,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 2, 80, 384, 8, 40, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 2, 80, 384, 8, 40, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 2, 80, 384, 8, 40, 128, 8, 8, 128, 4);
                // cta<2,96,384> warp<8,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 2, 96, 384, 8, 48, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 2, 96, 384, 8, 48, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 2, 96, 384, 8, 48, 128, 8, 8, 128, 4);
                // cta<2,112,384> warp<8,56,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 2, 112, 384, 8, 56, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 2, 112, 384, 8, 56, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 2, 112, 384, 8, 56, 128, 8, 8, 128, 4);
                // cta<2,128,384> warp<8,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 2, 128, 384, 8, 64, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 2, 128, 384, 8, 64, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 2, 128, 384, 8, 64, 128, 8, 8, 128, 4);
                // cta<2,256,384> warp<8,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 2, 256, 384, 8, 128, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 2, 256, 384, 8, 128, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 2, 256, 384, 8, 128, 128, 8, 8, 128, 4);
                // cta<4,32,384> warp<16,16,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 4, 32, 384, 16, 16, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 32, 384, 16, 16, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 32, 384, 16, 16, 128, 8, 8, 128, 4);
                // cta<4,48,384> warp<16,24,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 4, 48, 384, 16, 24, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 48, 384, 16, 24, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 48, 384, 16, 24, 128, 8, 8, 128, 4);
                // cta<4,64,384> warp<16,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 4, 64, 384, 16, 32, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 64, 384, 16, 32, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 64, 384, 16, 32, 128, 8, 8, 128, 4);
                // cta<4,80,384> warp<16,40,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 4, 80, 384, 16, 40, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 80, 384, 16, 40, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 80, 384, 16, 40, 128, 8, 8, 128, 4);
                // cta<4,96,384> warp<16,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 4, 96, 384, 16, 48, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 96, 384, 16, 48, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 96, 384, 16, 48, 128, 8, 8, 128, 4);
                // cta<4,112,384> warp<16,56,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 4, 112, 384, 16, 56, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 112, 384, 16, 56, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 112, 384, 16, 56, 128, 8, 8, 128, 4);
                // cta<4,128,384> warp<16,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 4, 128, 384, 16, 64, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 128, 384, 16, 64, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 128, 384, 16, 64, 128, 8, 8, 128, 4);
                // cta<4,256,384> warp<16,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 4, 256, 384, 16, 128, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 256, 384, 16, 128, 128, 8, 8, 128, 3);
                // cta<8,32,384> warp<32,16,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 8, 32, 384, 32, 16, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 32, 384, 32, 16, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 8, 32, 384, 32, 16, 128, 8, 8, 128, 4);
                // cta<8,48,384> warp<32,24,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 8, 48, 384, 32, 24, 128, 8, 8, 128, 2);
                // cta<8,64,384> warp<32,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 8, 64, 384, 32, 32, 128, 8, 8, 128, 2);
                // cta<8,80,384> warp<32,40,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 8, 80, 384, 32, 40, 128, 8, 8, 128, 2);
                // cta<8,96,384> warp<32,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 8, 96, 384, 32, 48, 128, 8, 8, 128, 2);
                // cta<8,112,384> warp<32,56,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 8, 112, 384, 32, 56, 128, 8, 8, 128, 2);
                // cta<8,128,384> warp<32,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 8, 128, 384, 32, 64, 128, 8, 8, 128, 2);
                // cta<8,256,384> warp<32,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 8, 256, 384, 32, 128, 128, 8, 8, 128, 2);
                // cta<4,32,384> warp<8,64,128> mma<8,8,128>   WARPS[2x1]
                TEST(4, 2, true, 4, 32, 384, 8, 64, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 32, 384, 8, 64, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 32, 384, 8, 64, 128, 8, 8, 128, 4);
                // cta<4,48,384> warp<8,96,128> mma<8,8,128>   WARPS[2x1]
                TEST(4, 2, true, 4, 48, 384, 8, 96, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 48, 384, 8, 96, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 48, 384, 8, 96, 128, 8, 8, 128, 4);
                // cta<4,64,384> warp<8,128,128> mma<8,8,128>   WARPS[2x1]
                TEST(4, 2, true, 4, 64, 384, 8, 128, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 64, 384, 8, 128, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 64, 384, 8, 128, 128, 8, 8, 128, 4);
                // cta<8,32,384> warp<16,64,128> mma<8,8,128>   WARPS[2x1]
                TEST(4, 2, true, 8, 32, 384, 16, 64, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 32, 384, 16, 64, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 8, 32, 384, 16, 64, 128, 8, 8, 128, 4);
                // cta<8,48,384> warp<16,96,128> mma<8,8,128>   WARPS[2x1]
                TEST(4, 2, true, 8, 48, 384, 16, 96, 128, 8, 8, 128, 2);
                // cta<8,64,384> warp<16,128,128> mma<8,8,128>   WARPS[2x1]
                TEST(4, 2, true, 8, 64, 384, 16, 128, 128, 8, 8, 128, 2);
                // cta<4,32,384> warp<8,32,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 2, true, 4, 32, 384, 8, 32, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 32, 384, 8, 32, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 32, 384, 8, 32, 128, 8, 8, 128, 4);
                // cta<4,48,384> warp<8,48,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 2, true, 4, 48, 384, 8, 48, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 48, 384, 8, 48, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 48, 384, 8, 48, 128, 8, 8, 128, 4);
                // cta<4,64,384> warp<8,64,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 2, true, 4, 64, 384, 8, 64, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 64, 384, 8, 64, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 64, 384, 8, 64, 128, 8, 8, 128, 4);
                // cta<4,80,384> warp<8,80,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 2, true, 4, 80, 384, 8, 80, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 80, 384, 8, 80, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 80, 384, 8, 80, 128, 8, 8, 128, 4);
                // cta<4,96,384> warp<8,96,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 2, true, 4, 96, 384, 8, 96, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 96, 384, 8, 96, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 96, 384, 8, 96, 128, 8, 8, 128, 4);
                // cta<4,112,384> warp<8,112,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 2, true, 4, 112, 384, 8, 112, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 112, 384, 8, 112, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 112, 384, 8, 112, 128, 8, 8, 128, 4);
                // cta<4,128,384> warp<8,128,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 2, true, 4, 128, 384, 8, 128, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 128, 384, 8, 128, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 128, 384, 8, 128, 128, 8, 8, 128, 4);
                // cta<8,32,384> warp<16,32,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 2, true, 8, 32, 384, 16, 32, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 32, 384, 16, 32, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 8, 32, 384, 16, 32, 128, 8, 8, 128, 4);
                // cta<8,48,384> warp<16,48,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 2, true, 8, 48, 384, 16, 48, 128, 8, 8, 128, 2);
                // cta<8,64,384> warp<16,64,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 2, true, 8, 64, 384, 16, 64, 128, 8, 8, 128, 2);
                // cta<8,80,384> warp<16,80,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 2, true, 8, 80, 384, 16, 80, 128, 8, 8, 128, 2);
                // cta<8,96,384> warp<16,96,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 2, true, 8, 96, 384, 16, 96, 128, 8, 8, 128, 2);
                // cta<8,112,384> warp<16,112,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 2, true, 8, 112, 384, 16, 112, 128, 8, 8, 128, 2);
                // cta<8,128,384> warp<16,128,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 2, true, 8, 128, 384, 16, 128, 128, 8, 8, 128, 2);
                // cta<8,32,384> warp<8,64,128> mma<8,8,128>   WARPS[4x1]
                TEST(4, 2, true, 8, 32, 384, 8, 64, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 32, 384, 8, 64, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 8, 32, 384, 8, 64, 128, 8, 8, 128, 4);
                // cta<8,48,384> warp<8,96,128> mma<8,8,128>   WARPS[4x1]
                TEST(4, 2, true, 8, 48, 384, 8, 96, 128, 8, 8, 128, 2);
                // cta<8,64,384> warp<8,128,128> mma<8,8,128>   WARPS[4x1]
                TEST(4, 2, true, 8, 64, 384, 8, 128, 128, 8, 8, 128, 2);
                // cta<2,32,512> warp<8,32,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 2, 32, 512, 8, 32, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 2, 32, 512, 8, 32, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 2, 32, 512, 8, 32, 128, 8, 8, 128, 4);
                // cta<2,48,512> warp<8,48,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 2, 48, 512, 8, 48, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 2, 48, 512, 8, 48, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 2, 48, 512, 8, 48, 128, 8, 8, 128, 4);
                // cta<2,64,512> warp<8,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 2, 64, 512, 8, 64, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 2, 64, 512, 8, 64, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 2, 64, 512, 8, 64, 128, 8, 8, 128, 4);
                // cta<2,80,512> warp<8,80,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 2, 80, 512, 8, 80, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 2, 80, 512, 8, 80, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 2, 80, 512, 8, 80, 128, 8, 8, 128, 4);
                // cta<2,96,512> warp<8,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 2, 96, 512, 8, 96, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 2, 96, 512, 8, 96, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 2, 96, 512, 8, 96, 128, 8, 8, 128, 4);
                // cta<2,112,512> warp<8,112,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 2, 112, 512, 8, 112, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 2, 112, 512, 8, 112, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 2, 112, 512, 8, 112, 128, 8, 8, 128, 4);
                // cta<2,128,512> warp<8,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 2, 128, 512, 8, 128, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 2, 128, 512, 8, 128, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 2, 128, 512, 8, 128, 128, 8, 8, 128, 4);
                // cta<4,32,512> warp<16,32,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 4, 32, 512, 16, 32, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 32, 512, 16, 32, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 32, 512, 16, 32, 128, 8, 8, 128, 4);
                // cta<4,48,512> warp<16,48,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 4, 48, 512, 16, 48, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 48, 512, 16, 48, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 48, 512, 16, 48, 128, 8, 8, 128, 4);
                // cta<4,64,512> warp<16,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 4, 64, 512, 16, 64, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 64, 512, 16, 64, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 64, 512, 16, 64, 128, 8, 8, 128, 4);
                // cta<4,80,512> warp<16,80,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 4, 80, 512, 16, 80, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 80, 512, 16, 80, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 80, 512, 16, 80, 128, 8, 8, 128, 4);
                // cta<4,96,512> warp<16,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 4, 96, 512, 16, 96, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 96, 512, 16, 96, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 96, 512, 16, 96, 128, 8, 8, 128, 4);
                // cta<4,112,512> warp<16,112,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 4, 112, 512, 16, 112, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 112, 512, 16, 112, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 112, 512, 16, 112, 128, 8, 8, 128, 4);
                // cta<4,128,512> warp<16,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 4, 128, 512, 16, 128, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 128, 512, 16, 128, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 128, 512, 16, 128, 128, 8, 8, 128, 4);
                // cta<8,32,512> warp<32,32,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 8, 32, 512, 32, 32, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 32, 512, 32, 32, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 8, 32, 512, 32, 32, 128, 8, 8, 128, 4);
                // cta<8,48,512> warp<32,48,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 8, 48, 512, 32, 48, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 48, 512, 32, 48, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 8, 48, 512, 32, 48, 128, 8, 8, 128, 4);
                // cta<8,64,512> warp<32,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 8, 64, 512, 32, 64, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 64, 512, 32, 64, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 8, 64, 512, 32, 64, 128, 8, 8, 128, 4);
                // cta<8,80,512> warp<32,80,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 8, 80, 512, 32, 80, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 80, 512, 32, 80, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 8, 80, 512, 32, 80, 128, 8, 8, 128, 4);
                // cta<8,96,512> warp<32,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 8, 96, 512, 32, 96, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 96, 512, 32, 96, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 8, 96, 512, 32, 96, 128, 8, 8, 128, 4);
                // cta<8,112,512> warp<32,112,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 8, 112, 512, 32, 112, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 112, 512, 32, 112, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 8, 112, 512, 32, 112, 128, 8, 8, 128, 4);
                // cta<8,128,512> warp<32,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 2, true, 8, 128, 512, 32, 128, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 128, 512, 32, 128, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 8, 128, 512, 32, 128, 128, 8, 8, 128, 4);
                // cta<2,32,512> warp<8,16,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 2, 32, 512, 8, 16, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 2, 32, 512, 8, 16, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 2, 32, 512, 8, 16, 128, 8, 8, 128, 4);
                // cta<2,48,512> warp<8,24,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 2, 48, 512, 8, 24, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 2, 48, 512, 8, 24, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 2, 48, 512, 8, 24, 128, 8, 8, 128, 4);
                // cta<2,64,512> warp<8,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 2, 64, 512, 8, 32, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 2, 64, 512, 8, 32, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 2, 64, 512, 8, 32, 128, 8, 8, 128, 4);
                // cta<2,80,512> warp<8,40,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 2, 80, 512, 8, 40, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 2, 80, 512, 8, 40, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 2, 80, 512, 8, 40, 128, 8, 8, 128, 4);
                // cta<2,96,512> warp<8,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 2, 96, 512, 8, 48, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 2, 96, 512, 8, 48, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 2, 96, 512, 8, 48, 128, 8, 8, 128, 4);
                // cta<2,112,512> warp<8,56,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 2, 112, 512, 8, 56, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 2, 112, 512, 8, 56, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 2, 112, 512, 8, 56, 128, 8, 8, 128, 4);
                // cta<2,128,512> warp<8,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 2, 128, 512, 8, 64, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 2, 128, 512, 8, 64, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 2, 128, 512, 8, 64, 128, 8, 8, 128, 4);
                // cta<2,256,512> warp<8,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 2, 256, 512, 8, 128, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 2, 256, 512, 8, 128, 128, 8, 8, 128, 3);
                // cta<4,32,512> warp<16,16,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 4, 32, 512, 16, 16, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 32, 512, 16, 16, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 32, 512, 16, 16, 128, 8, 8, 128, 4);
                // cta<4,48,512> warp<16,24,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 4, 48, 512, 16, 24, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 48, 512, 16, 24, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 48, 512, 16, 24, 128, 8, 8, 128, 4);
                // cta<4,64,512> warp<16,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 4, 64, 512, 16, 32, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 64, 512, 16, 32, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 64, 512, 16, 32, 128, 8, 8, 128, 4);
                // cta<4,80,512> warp<16,40,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 4, 80, 512, 16, 40, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 80, 512, 16, 40, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 80, 512, 16, 40, 128, 8, 8, 128, 4);
                // cta<4,96,512> warp<16,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 4, 96, 512, 16, 48, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 96, 512, 16, 48, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 96, 512, 16, 48, 128, 8, 8, 128, 4);
                // cta<4,112,512> warp<16,56,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 4, 112, 512, 16, 56, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 112, 512, 16, 56, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 112, 512, 16, 56, 128, 8, 8, 128, 4);
                // cta<4,128,512> warp<16,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 4, 128, 512, 16, 64, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 128, 512, 16, 64, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 128, 512, 16, 64, 128, 8, 8, 128, 4);
                // cta<4,256,512> warp<16,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 4, 256, 512, 16, 128, 128, 8, 8, 128, 2);
                // cta<8,32,512> warp<32,16,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 8, 32, 512, 32, 16, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 32, 512, 32, 16, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 8, 32, 512, 32, 16, 128, 8, 8, 128, 4);
                // cta<8,48,512> warp<32,24,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 8, 48, 512, 32, 24, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 48, 512, 32, 24, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 8, 48, 512, 32, 24, 128, 8, 8, 128, 4);
                // cta<8,64,512> warp<32,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 8, 64, 512, 32, 32, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 64, 512, 32, 32, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 8, 64, 512, 32, 32, 128, 8, 8, 128, 4);
                // cta<8,80,512> warp<32,40,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 8, 80, 512, 32, 40, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 80, 512, 32, 40, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 8, 80, 512, 32, 40, 128, 8, 8, 128, 4);
                // cta<8,96,512> warp<32,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 8, 96, 512, 32, 48, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 96, 512, 32, 48, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 8, 96, 512, 32, 48, 128, 8, 8, 128, 4);
                // cta<8,112,512> warp<32,56,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 8, 112, 512, 32, 56, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 112, 512, 32, 56, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 8, 112, 512, 32, 56, 128, 8, 8, 128, 4);
                // cta<8,128,512> warp<32,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 8, 128, 512, 32, 64, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 128, 512, 32, 64, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 8, 128, 512, 32, 64, 128, 8, 8, 128, 4);
                // cta<8,256,512> warp<32,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 2, true, 8, 256, 512, 32, 128, 128, 8, 8, 128, 2);
                // cta<4,32,512> warp<8,64,128> mma<8,8,128>   WARPS[2x1]
                TEST(4, 2, true, 4, 32, 512, 8, 64, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 32, 512, 8, 64, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 32, 512, 8, 64, 128, 8, 8, 128, 4);
                // cta<4,48,512> warp<8,96,128> mma<8,8,128>   WARPS[2x1]
                TEST(4, 2, true, 4, 48, 512, 8, 96, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 48, 512, 8, 96, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 48, 512, 8, 96, 128, 8, 8, 128, 4);
                // cta<4,64,512> warp<8,128,128> mma<8,8,128>   WARPS[2x1]
                TEST(4, 2, true, 4, 64, 512, 8, 128, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 64, 512, 8, 128, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 64, 512, 8, 128, 128, 8, 8, 128, 4);
                // cta<8,32,512> warp<16,64,128> mma<8,8,128>   WARPS[2x1]
                TEST(4, 2, true, 8, 32, 512, 16, 64, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 32, 512, 16, 64, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 8, 32, 512, 16, 64, 128, 8, 8, 128, 4);
                // cta<8,48,512> warp<16,96,128> mma<8,8,128>   WARPS[2x1]
                TEST(4, 2, true, 8, 48, 512, 16, 96, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 48, 512, 16, 96, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 8, 48, 512, 16, 96, 128, 8, 8, 128, 4);
                // cta<8,64,512> warp<16,128,128> mma<8,8,128>   WARPS[2x1]
                TEST(4, 2, true, 8, 64, 512, 16, 128, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 64, 512, 16, 128, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 8, 64, 512, 16, 128, 128, 8, 8, 128, 4);
                // cta<4,32,512> warp<8,32,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 2, true, 4, 32, 512, 8, 32, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 32, 512, 8, 32, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 32, 512, 8, 32, 128, 8, 8, 128, 4);
                // cta<4,48,512> warp<8,48,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 2, true, 4, 48, 512, 8, 48, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 48, 512, 8, 48, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 48, 512, 8, 48, 128, 8, 8, 128, 4);
                // cta<4,64,512> warp<8,64,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 2, true, 4, 64, 512, 8, 64, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 64, 512, 8, 64, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 64, 512, 8, 64, 128, 8, 8, 128, 4);
                // cta<4,80,512> warp<8,80,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 2, true, 4, 80, 512, 8, 80, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 80, 512, 8, 80, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 80, 512, 8, 80, 128, 8, 8, 128, 4);
                // cta<4,96,512> warp<8,96,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 2, true, 4, 96, 512, 8, 96, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 96, 512, 8, 96, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 96, 512, 8, 96, 128, 8, 8, 128, 4);
                // cta<4,112,512> warp<8,112,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 2, true, 4, 112, 512, 8, 112, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 112, 512, 8, 112, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 112, 512, 8, 112, 128, 8, 8, 128, 4);
                // cta<4,128,512> warp<8,128,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 2, true, 4, 128, 512, 8, 128, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 4, 128, 512, 8, 128, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 4, 128, 512, 8, 128, 128, 8, 8, 128, 4);
                // cta<8,32,512> warp<16,32,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 2, true, 8, 32, 512, 16, 32, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 32, 512, 16, 32, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 8, 32, 512, 16, 32, 128, 8, 8, 128, 4);
                // cta<8,48,512> warp<16,48,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 2, true, 8, 48, 512, 16, 48, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 48, 512, 16, 48, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 8, 48, 512, 16, 48, 128, 8, 8, 128, 4);
                // cta<8,64,512> warp<16,64,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 2, true, 8, 64, 512, 16, 64, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 64, 512, 16, 64, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 8, 64, 512, 16, 64, 128, 8, 8, 128, 4);
                // cta<8,80,512> warp<16,80,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 2, true, 8, 80, 512, 16, 80, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 80, 512, 16, 80, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 8, 80, 512, 16, 80, 128, 8, 8, 128, 4);
                // cta<8,96,512> warp<16,96,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 2, true, 8, 96, 512, 16, 96, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 96, 512, 16, 96, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 8, 96, 512, 16, 96, 128, 8, 8, 128, 4);
                // cta<8,112,512> warp<16,112,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 2, true, 8, 112, 512, 16, 112, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 112, 512, 16, 112, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 8, 112, 512, 16, 112, 128, 8, 8, 128, 4);
                // cta<8,128,512> warp<16,128,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 2, true, 8, 128, 512, 16, 128, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 128, 512, 16, 128, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 8, 128, 512, 16, 128, 128, 8, 8, 128, 4);
                // cta<8,32,512> warp<8,64,128> mma<8,8,128>   WARPS[4x1]
                TEST(4, 2, true, 8, 32, 512, 8, 64, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 32, 512, 8, 64, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 8, 32, 512, 8, 64, 128, 8, 8, 128, 4);
                // cta<8,48,512> warp<8,96,128> mma<8,8,128>   WARPS[4x1]
                TEST(4, 2, true, 8, 48, 512, 8, 96, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 48, 512, 8, 96, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 8, 48, 512, 8, 96, 128, 8, 8, 128, 4);
                // cta<8,64,512> warp<8,128,128> mma<8,8,128>   WARPS[4x1]
                TEST(4, 2, true, 8, 64, 512, 8, 128, 128, 8, 8, 128, 2);
                TEST(4, 2, true, 8, 64, 512, 8, 128, 128, 8, 8, 128, 3);
                TEST(4, 2, true, 8, 64, 512, 8, 128, 128, 8, 8, 128, 4);
            } else {
            }
            break;
        #endif

        #ifdef W4A4
        case 4:
            if (quant_sign) {
                ////// W4A4 int
                // cta<2,32,256> warp<8,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 4, true, 2, 32, 256, 8, 64, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 2, 32, 256, 8, 64, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 2, 32, 256, 8, 64, 128, 8, 8, 128, 4);
                // cta<2,48,256> warp<8,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 4, true, 2, 48, 256, 8, 96, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 2, 48, 256, 8, 96, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 2, 48, 256, 8, 96, 128, 8, 8, 128, 4);
                // cta<2,64,256> warp<8,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 4, true, 2, 64, 256, 8, 128, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 2, 64, 256, 8, 128, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 2, 64, 256, 8, 128, 128, 8, 8, 128, 4);
                // cta<4,32,256> warp<16,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 4, true, 4, 32, 256, 16, 64, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 4, 32, 256, 16, 64, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 4, 32, 256, 16, 64, 128, 8, 8, 128, 4);
                // cta<4,48,256> warp<16,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 4, true, 4, 48, 256, 16, 96, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 4, 48, 256, 16, 96, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 4, 48, 256, 16, 96, 128, 8, 8, 128, 4);
                // cta<4,64,256> warp<16,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 4, true, 4, 64, 256, 16, 128, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 4, 64, 256, 16, 128, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 4, 64, 256, 16, 128, 128, 8, 8, 128, 4);
                // cta<8,32,256> warp<32,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 4, true, 8, 32, 256, 32, 64, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 8, 32, 256, 32, 64, 128, 8, 8, 128, 3);
                // cta<8,48,256> warp<32,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 4, true, 8, 48, 256, 32, 96, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 8, 48, 256, 32, 96, 128, 8, 8, 128, 3);
                // cta<8,64,256> warp<32,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 4, true, 8, 64, 256, 32, 128, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 8, 64, 256, 32, 128, 128, 8, 8, 128, 3);
                // cta<2,32,256> warp<8,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 2, 32, 256, 8, 32, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 2, 32, 256, 8, 32, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 2, 32, 256, 8, 32, 128, 8, 8, 128, 4);
                // cta<2,48,256> warp<8,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 2, 48, 256, 8, 48, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 2, 48, 256, 8, 48, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 2, 48, 256, 8, 48, 128, 8, 8, 128, 4);
                // cta<2,64,256> warp<8,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 2, 64, 256, 8, 64, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 2, 64, 256, 8, 64, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 2, 64, 256, 8, 64, 128, 8, 8, 128, 4);
                // cta<2,80,256> warp<8,80,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 2, 80, 256, 8, 80, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 2, 80, 256, 8, 80, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 2, 80, 256, 8, 80, 128, 8, 8, 128, 4);
                // cta<2,96,256> warp<8,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 2, 96, 256, 8, 96, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 2, 96, 256, 8, 96, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 2, 96, 256, 8, 96, 128, 8, 8, 128, 4);
                // cta<2,112,256> warp<8,112,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 2, 112, 256, 8, 112, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 2, 112, 256, 8, 112, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 2, 112, 256, 8, 112, 128, 8, 8, 128, 4);
                // cta<2,128,256> warp<8,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 2, 128, 256, 8, 128, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 2, 128, 256, 8, 128, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 2, 128, 256, 8, 128, 128, 8, 8, 128, 4);
                // cta<4,32,256> warp<16,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 4, 32, 256, 16, 32, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 4, 32, 256, 16, 32, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 4, 32, 256, 16, 32, 128, 8, 8, 128, 4);
                // cta<4,48,256> warp<16,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 4, 48, 256, 16, 48, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 4, 48, 256, 16, 48, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 4, 48, 256, 16, 48, 128, 8, 8, 128, 4);
                // cta<4,64,256> warp<16,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 4, 64, 256, 16, 64, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 4, 64, 256, 16, 64, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 4, 64, 256, 16, 64, 128, 8, 8, 128, 4);
                // cta<4,80,256> warp<16,80,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 4, 80, 256, 16, 80, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 4, 80, 256, 16, 80, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 4, 80, 256, 16, 80, 128, 8, 8, 128, 4);
                // cta<4,96,256> warp<16,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 4, 96, 256, 16, 96, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 4, 96, 256, 16, 96, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 4, 96, 256, 16, 96, 128, 8, 8, 128, 4);
                // cta<4,112,256> warp<16,112,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 4, 112, 256, 16, 112, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 4, 112, 256, 16, 112, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 4, 112, 256, 16, 112, 128, 8, 8, 128, 4);
                // cta<4,128,256> warp<16,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 4, 128, 256, 16, 128, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 4, 128, 256, 16, 128, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 4, 128, 256, 16, 128, 128, 8, 8, 128, 4);
                // cta<8,32,256> warp<32,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 8, 32, 256, 32, 32, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 8, 32, 256, 32, 32, 128, 8, 8, 128, 3);
                // cta<8,48,256> warp<32,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 8, 48, 256, 32, 48, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 8, 48, 256, 32, 48, 128, 8, 8, 128, 3);
                // cta<8,64,256> warp<32,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 8, 64, 256, 32, 64, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 8, 64, 256, 32, 64, 128, 8, 8, 128, 3);
                // cta<8,80,256> warp<32,80,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 8, 80, 256, 32, 80, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 8, 80, 256, 32, 80, 128, 8, 8, 128, 3);
                // cta<8,96,256> warp<32,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 8, 96, 256, 32, 96, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 8, 96, 256, 32, 96, 128, 8, 8, 128, 3);
                // cta<8,112,256> warp<32,112,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 8, 112, 256, 32, 112, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 8, 112, 256, 32, 112, 128, 8, 8, 128, 3);
                // cta<8,128,256> warp<32,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 8, 128, 256, 32, 128, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 8, 128, 256, 32, 128, 128, 8, 8, 128, 3);
                // cta<4,32,256> warp<8,128,128> mma<8,8,128>   WARPS[2x1]
                TEST(4, 4, true, 4, 32, 256, 8, 128, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 4, 32, 256, 8, 128, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 4, 32, 256, 8, 128, 128, 8, 8, 128, 4);
                // cta<8,32,256> warp<16,128,128> mma<8,8,128>   WARPS[2x1]
                TEST(4, 4, true, 8, 32, 256, 16, 128, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 8, 32, 256, 16, 128, 128, 8, 8, 128, 3);
                // cta<4,32,256> warp<8,64,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 4, true, 4, 32, 256, 8, 64, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 4, 32, 256, 8, 64, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 4, 32, 256, 8, 64, 128, 8, 8, 128, 4);
                // cta<4,48,256> warp<8,96,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 4, true, 4, 48, 256, 8, 96, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 4, 48, 256, 8, 96, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 4, 48, 256, 8, 96, 128, 8, 8, 128, 4);
                // cta<4,64,256> warp<8,128,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 4, true, 4, 64, 256, 8, 128, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 4, 64, 256, 8, 128, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 4, 64, 256, 8, 128, 128, 8, 8, 128, 4);
                // cta<8,32,256> warp<16,64,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 4, true, 8, 32, 256, 16, 64, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 8, 32, 256, 16, 64, 128, 8, 8, 128, 3);
                // cta<8,48,256> warp<16,96,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 4, true, 8, 48, 256, 16, 96, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 8, 48, 256, 16, 96, 128, 8, 8, 128, 3);
                // cta<8,64,256> warp<16,128,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 4, true, 8, 64, 256, 16, 128, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 8, 64, 256, 16, 128, 128, 8, 8, 128, 3);
                // cta<8,32,256> warp<8,128,128> mma<8,8,128>   WARPS[4x1]
                TEST(4, 4, true, 8, 32, 256, 8, 128, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 8, 32, 256, 8, 128, 128, 8, 8, 128, 3);
                // cta<2,32,384> warp<8,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 4, true, 2, 32, 384, 8, 64, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 2, 32, 384, 8, 64, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 2, 32, 384, 8, 64, 128, 8, 8, 128, 4);
                // cta<2,48,384> warp<8,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 4, true, 2, 48, 384, 8, 96, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 2, 48, 384, 8, 96, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 2, 48, 384, 8, 96, 128, 8, 8, 128, 4);
                // cta<2,64,384> warp<8,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 4, true, 2, 64, 384, 8, 128, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 2, 64, 384, 8, 128, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 2, 64, 384, 8, 128, 128, 8, 8, 128, 4);
                // cta<4,32,384> warp<16,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 4, true, 4, 32, 384, 16, 64, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 4, 32, 384, 16, 64, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 4, 32, 384, 16, 64, 128, 8, 8, 128, 4);
                // cta<4,48,384> warp<16,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 4, true, 4, 48, 384, 16, 96, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 4, 48, 384, 16, 96, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 4, 48, 384, 16, 96, 128, 8, 8, 128, 4);
                // cta<4,64,384> warp<16,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 4, true, 4, 64, 384, 16, 128, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 4, 64, 384, 16, 128, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 4, 64, 384, 16, 128, 128, 8, 8, 128, 4);
                // cta<8,32,384> warp<32,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 4, true, 8, 32, 384, 32, 64, 128, 8, 8, 128, 2);
                // cta<8,48,384> warp<32,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 4, true, 8, 48, 384, 32, 96, 128, 8, 8, 128, 2);
                // cta<8,64,384> warp<32,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 4, true, 8, 64, 384, 32, 128, 128, 8, 8, 128, 2);
                // cta<2,32,384> warp<8,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 2, 32, 384, 8, 32, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 2, 32, 384, 8, 32, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 2, 32, 384, 8, 32, 128, 8, 8, 128, 4);
                // cta<2,48,384> warp<8,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 2, 48, 384, 8, 48, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 2, 48, 384, 8, 48, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 2, 48, 384, 8, 48, 128, 8, 8, 128, 4);
                // cta<2,64,384> warp<8,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 2, 64, 384, 8, 64, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 2, 64, 384, 8, 64, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 2, 64, 384, 8, 64, 128, 8, 8, 128, 4);
                // cta<2,80,384> warp<8,80,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 2, 80, 384, 8, 80, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 2, 80, 384, 8, 80, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 2, 80, 384, 8, 80, 128, 8, 8, 128, 4);
                // cta<2,96,384> warp<8,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 2, 96, 384, 8, 96, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 2, 96, 384, 8, 96, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 2, 96, 384, 8, 96, 128, 8, 8, 128, 4);
                // cta<2,112,384> warp<8,112,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 2, 112, 384, 8, 112, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 2, 112, 384, 8, 112, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 2, 112, 384, 8, 112, 128, 8, 8, 128, 4);
                // cta<2,128,384> warp<8,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 2, 128, 384, 8, 128, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 2, 128, 384, 8, 128, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 2, 128, 384, 8, 128, 128, 8, 8, 128, 4);
                // cta<4,32,384> warp<16,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 4, 32, 384, 16, 32, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 4, 32, 384, 16, 32, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 4, 32, 384, 16, 32, 128, 8, 8, 128, 4);
                // cta<4,48,384> warp<16,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 4, 48, 384, 16, 48, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 4, 48, 384, 16, 48, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 4, 48, 384, 16, 48, 128, 8, 8, 128, 4);
                // cta<4,64,384> warp<16,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 4, 64, 384, 16, 64, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 4, 64, 384, 16, 64, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 4, 64, 384, 16, 64, 128, 8, 8, 128, 4);
                // cta<4,80,384> warp<16,80,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 4, 80, 384, 16, 80, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 4, 80, 384, 16, 80, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 4, 80, 384, 16, 80, 128, 8, 8, 128, 4);
                // cta<4,96,384> warp<16,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 4, 96, 384, 16, 96, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 4, 96, 384, 16, 96, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 4, 96, 384, 16, 96, 128, 8, 8, 128, 4);
                // cta<4,112,384> warp<16,112,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 4, 112, 384, 16, 112, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 4, 112, 384, 16, 112, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 4, 112, 384, 16, 112, 128, 8, 8, 128, 4);
                // cta<4,128,384> warp<16,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 4, 128, 384, 16, 128, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 4, 128, 384, 16, 128, 128, 8, 8, 128, 3);
                // cta<8,32,384> warp<32,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 8, 32, 384, 32, 32, 128, 8, 8, 128, 2);
                // cta<8,48,384> warp<32,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 8, 48, 384, 32, 48, 128, 8, 8, 128, 2);
                // cta<8,64,384> warp<32,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 8, 64, 384, 32, 64, 128, 8, 8, 128, 2);
                // cta<8,80,384> warp<32,80,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 8, 80, 384, 32, 80, 128, 8, 8, 128, 2);
                // cta<8,96,384> warp<32,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 8, 96, 384, 32, 96, 128, 8, 8, 128, 2);
                // cta<8,112,384> warp<32,112,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 8, 112, 384, 32, 112, 128, 8, 8, 128, 2);
                // cta<8,128,384> warp<32,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 8, 128, 384, 32, 128, 128, 8, 8, 128, 2);
                // cta<4,32,384> warp<8,128,128> mma<8,8,128>   WARPS[2x1]
                TEST(4, 4, true, 4, 32, 384, 8, 128, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 4, 32, 384, 8, 128, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 4, 32, 384, 8, 128, 128, 8, 8, 128, 4);
                // cta<8,32,384> warp<16,128,128> mma<8,8,128>   WARPS[2x1]
                TEST(4, 4, true, 8, 32, 384, 16, 128, 128, 8, 8, 128, 2);
                // cta<4,32,384> warp<8,64,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 4, true, 4, 32, 384, 8, 64, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 4, 32, 384, 8, 64, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 4, 32, 384, 8, 64, 128, 8, 8, 128, 4);
                // cta<4,48,384> warp<8,96,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 4, true, 4, 48, 384, 8, 96, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 4, 48, 384, 8, 96, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 4, 48, 384, 8, 96, 128, 8, 8, 128, 4);
                // cta<4,64,384> warp<8,128,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 4, true, 4, 64, 384, 8, 128, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 4, 64, 384, 8, 128, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 4, 64, 384, 8, 128, 128, 8, 8, 128, 4);
                // cta<8,32,384> warp<16,64,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 4, true, 8, 32, 384, 16, 64, 128, 8, 8, 128, 2);
                // cta<8,48,384> warp<16,96,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 4, true, 8, 48, 384, 16, 96, 128, 8, 8, 128, 2);
                // cta<8,64,384> warp<16,128,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 4, true, 8, 64, 384, 16, 128, 128, 8, 8, 128, 2);
                // cta<8,32,384> warp<8,128,128> mma<8,8,128>   WARPS[4x1]
                TEST(4, 4, true, 8, 32, 384, 8, 128, 128, 8, 8, 128, 2);
                // cta<2,32,512> warp<8,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 4, true, 2, 32, 512, 8, 64, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 2, 32, 512, 8, 64, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 2, 32, 512, 8, 64, 128, 8, 8, 128, 4);
                // cta<2,48,512> warp<8,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 4, true, 2, 48, 512, 8, 96, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 2, 48, 512, 8, 96, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 2, 48, 512, 8, 96, 128, 8, 8, 128, 4);
                // cta<2,64,512> warp<8,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 4, true, 2, 64, 512, 8, 128, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 2, 64, 512, 8, 128, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 2, 64, 512, 8, 128, 128, 8, 8, 128, 4);
                // cta<4,32,512> warp<16,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 4, true, 4, 32, 512, 16, 64, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 4, 32, 512, 16, 64, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 4, 32, 512, 16, 64, 128, 8, 8, 128, 4);
                // cta<4,48,512> warp<16,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 4, true, 4, 48, 512, 16, 96, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 4, 48, 512, 16, 96, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 4, 48, 512, 16, 96, 128, 8, 8, 128, 4);
                // cta<4,64,512> warp<16,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 4, true, 4, 64, 512, 16, 128, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 4, 64, 512, 16, 128, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 4, 64, 512, 16, 128, 128, 8, 8, 128, 4);
                // cta<8,32,512> warp<32,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 4, true, 8, 32, 512, 32, 64, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 8, 32, 512, 32, 64, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 8, 32, 512, 32, 64, 128, 8, 8, 128, 4);
                // cta<8,48,512> warp<32,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 4, true, 8, 48, 512, 32, 96, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 8, 48, 512, 32, 96, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 8, 48, 512, 32, 96, 128, 8, 8, 128, 4);
                // cta<8,64,512> warp<32,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(4, 4, true, 8, 64, 512, 32, 128, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 8, 64, 512, 32, 128, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 8, 64, 512, 32, 128, 128, 8, 8, 128, 4);
                // cta<2,32,512> warp<8,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 2, 32, 512, 8, 32, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 2, 32, 512, 8, 32, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 2, 32, 512, 8, 32, 128, 8, 8, 128, 4);
                // cta<2,48,512> warp<8,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 2, 48, 512, 8, 48, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 2, 48, 512, 8, 48, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 2, 48, 512, 8, 48, 128, 8, 8, 128, 4);
                // cta<2,64,512> warp<8,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 2, 64, 512, 8, 64, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 2, 64, 512, 8, 64, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 2, 64, 512, 8, 64, 128, 8, 8, 128, 4);
                // cta<2,80,512> warp<8,80,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 2, 80, 512, 8, 80, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 2, 80, 512, 8, 80, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 2, 80, 512, 8, 80, 128, 8, 8, 128, 4);
                // cta<2,96,512> warp<8,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 2, 96, 512, 8, 96, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 2, 96, 512, 8, 96, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 2, 96, 512, 8, 96, 128, 8, 8, 128, 4);
                // cta<2,112,512> warp<8,112,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 2, 112, 512, 8, 112, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 2, 112, 512, 8, 112, 128, 8, 8, 128, 3);
                // cta<2,128,512> warp<8,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 2, 128, 512, 8, 128, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 2, 128, 512, 8, 128, 128, 8, 8, 128, 3);
                // cta<4,32,512> warp<16,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 4, 32, 512, 16, 32, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 4, 32, 512, 16, 32, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 4, 32, 512, 16, 32, 128, 8, 8, 128, 4);
                // cta<4,48,512> warp<16,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 4, 48, 512, 16, 48, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 4, 48, 512, 16, 48, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 4, 48, 512, 16, 48, 128, 8, 8, 128, 4);
                // cta<4,64,512> warp<16,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 4, 64, 512, 16, 64, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 4, 64, 512, 16, 64, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 4, 64, 512, 16, 64, 128, 8, 8, 128, 4);
                // cta<4,80,512> warp<16,80,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 4, 80, 512, 16, 80, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 4, 80, 512, 16, 80, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 4, 80, 512, 16, 80, 128, 8, 8, 128, 4);
                // cta<4,96,512> warp<16,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 4, 96, 512, 16, 96, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 4, 96, 512, 16, 96, 128, 8, 8, 128, 3);
                // cta<4,112,512> warp<16,112,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 4, 112, 512, 16, 112, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 4, 112, 512, 16, 112, 128, 8, 8, 128, 3);
                // cta<4,128,512> warp<16,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 4, 128, 512, 16, 128, 128, 8, 8, 128, 2);
                // cta<8,32,512> warp<32,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 8, 32, 512, 32, 32, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 8, 32, 512, 32, 32, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 8, 32, 512, 32, 32, 128, 8, 8, 128, 4);
                // cta<8,48,512> warp<32,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 8, 48, 512, 32, 48, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 8, 48, 512, 32, 48, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 8, 48, 512, 32, 48, 128, 8, 8, 128, 4);
                // cta<8,64,512> warp<32,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 8, 64, 512, 32, 64, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 8, 64, 512, 32, 64, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 8, 64, 512, 32, 64, 128, 8, 8, 128, 4);
                // cta<8,80,512> warp<32,80,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 8, 80, 512, 32, 80, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 8, 80, 512, 32, 80, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 8, 80, 512, 32, 80, 128, 8, 8, 128, 4);
                // cta<8,96,512> warp<32,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 8, 96, 512, 32, 96, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 8, 96, 512, 32, 96, 128, 8, 8, 128, 3);
                // cta<8,112,512> warp<32,112,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 8, 112, 512, 32, 112, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 8, 112, 512, 32, 112, 128, 8, 8, 128, 3);
                // cta<8,128,512> warp<32,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(4, 4, true, 8, 128, 512, 32, 128, 128, 8, 8, 128, 2);
                // cta<4,32,512> warp<8,128,128> mma<8,8,128>   WARPS[2x1]
                TEST(4, 4, true, 4, 32, 512, 8, 128, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 4, 32, 512, 8, 128, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 4, 32, 512, 8, 128, 128, 8, 8, 128, 4);
                // cta<8,32,512> warp<16,128,128> mma<8,8,128>   WARPS[2x1]
                TEST(4, 4, true, 8, 32, 512, 16, 128, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 8, 32, 512, 16, 128, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 8, 32, 512, 16, 128, 128, 8, 8, 128, 4);
                // cta<4,32,512> warp<8,64,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 4, true, 4, 32, 512, 8, 64, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 4, 32, 512, 8, 64, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 4, 32, 512, 8, 64, 128, 8, 8, 128, 4);
                // cta<4,48,512> warp<8,96,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 4, true, 4, 48, 512, 8, 96, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 4, 48, 512, 8, 96, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 4, 48, 512, 8, 96, 128, 8, 8, 128, 4);
                // cta<4,64,512> warp<8,128,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 4, true, 4, 64, 512, 8, 128, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 4, 64, 512, 8, 128, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 4, 64, 512, 8, 128, 128, 8, 8, 128, 4);
                // cta<8,32,512> warp<16,64,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 4, true, 8, 32, 512, 16, 64, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 8, 32, 512, 16, 64, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 8, 32, 512, 16, 64, 128, 8, 8, 128, 4);
                // cta<8,48,512> warp<16,96,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 4, true, 8, 48, 512, 16, 96, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 8, 48, 512, 16, 96, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 8, 48, 512, 16, 96, 128, 8, 8, 128, 4);
                // cta<8,64,512> warp<16,128,128> mma<8,8,128>   WARPS[2x2]
                TEST(4, 4, true, 8, 64, 512, 16, 128, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 8, 64, 512, 16, 128, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 8, 64, 512, 16, 128, 128, 8, 8, 128, 4);
                // cta<8,32,512> warp<8,128,128> mma<8,8,128>   WARPS[4x1]
                TEST(4, 4, true, 8, 32, 512, 8, 128, 128, 8, 8, 128, 2);
                TEST(4, 4, true, 8, 32, 512, 8, 128, 128, 8, 8, 128, 3);
                TEST(4, 4, true, 8, 32, 512, 8, 128, 128, 8, 8, 128, 4);
            } else {
            }
            break;
        #endif
        default:
            printf("unsupport w%da%d\n", w_bits, x_bits);
        }
        break;
    case 5:
        switch (w_bits) {
        #ifdef W5A5
        case 5:
            if (quant_sign) {
                ////// W5A5 int
                // cta<1,32,256> warp<8,80,128> mma<8,8,128>   WARPS[1x2]
                TEST(5, 5, true, 1, 32, 256, 8, 80, 128, 8, 8, 128, 2);
                TEST(5, 5, true, 1, 32, 256, 8, 80, 128, 8, 8, 128, 3);
                TEST(5, 5, true, 1, 32, 256, 8, 80, 128, 8, 8, 128, 4);
                // cta<1,48,256> warp<8,120,128> mma<8,8,128>   WARPS[1x2]
                TEST(5, 5, true, 1, 48, 256, 8, 120, 128, 8, 8, 128, 2);
                TEST(5, 5, true, 1, 48, 256, 8, 120, 128, 8, 8, 128, 3);
                TEST(5, 5, true, 1, 48, 256, 8, 120, 128, 8, 8, 128, 4);
                // cta<8,32,256> warp<40,80,128> mma<8,8,128>   WARPS[1x2]
                TEST(5, 5, true, 8, 32, 256, 40, 80, 128, 8, 8, 128, 2);
                TEST(5, 5, true, 8, 32, 256, 40, 80, 128, 8, 8, 128, 3);
                TEST(5, 5, true, 8, 32, 256, 40, 80, 128, 8, 8, 128, 4);
                // cta<8,48,256> warp<40,120,128> mma<8,8,128>   WARPS[1x2]
                TEST(5, 5, true, 8, 48, 256, 40, 120, 128, 8, 8, 128, 2);
                TEST(5, 5, true, 8, 48, 256, 40, 120, 128, 8, 8, 128, 3);
                TEST(5, 5, true, 8, 48, 256, 40, 120, 128, 8, 8, 128, 4);
                // cta<1,32,256> warp<8,40,128> mma<8,8,128>   WARPS[1x4]
                TEST(5, 5, true, 1, 32, 256, 8, 40, 128, 8, 8, 128, 2);
                TEST(5, 5, true, 1, 32, 256, 8, 40, 128, 8, 8, 128, 3);
                TEST(5, 5, true, 1, 32, 256, 8, 40, 128, 8, 8, 128, 4);
                // cta<1,64,256> warp<8,80,128> mma<8,8,128>   WARPS[1x4]
                TEST(5, 5, true, 1, 64, 256, 8, 80, 128, 8, 8, 128, 2);
                TEST(5, 5, true, 1, 64, 256, 8, 80, 128, 8, 8, 128, 3);
                TEST(5, 5, true, 1, 64, 256, 8, 80, 128, 8, 8, 128, 4);
                // cta<1,96,256> warp<8,120,128> mma<8,8,128>   WARPS[1x4]
                TEST(5, 5, true, 1, 96, 256, 8, 120, 128, 8, 8, 128, 2);
                TEST(5, 5, true, 1, 96, 256, 8, 120, 128, 8, 8, 128, 3);
                TEST(5, 5, true, 1, 96, 256, 8, 120, 128, 8, 8, 128, 4);
                // cta<8,32,256> warp<40,40,128> mma<8,8,128>   WARPS[1x4]
                TEST(5, 5, true, 8, 32, 256, 40, 40, 128, 8, 8, 128, 2);
                TEST(5, 5, true, 8, 32, 256, 40, 40, 128, 8, 8, 128, 3);
                TEST(5, 5, true, 8, 32, 256, 40, 40, 128, 8, 8, 128, 4);
                // cta<8,64,256> warp<40,80,128> mma<8,8,128>   WARPS[1x4]
                TEST(5, 5, true, 8, 64, 256, 40, 80, 128, 8, 8, 128, 2);
                TEST(5, 5, true, 8, 64, 256, 40, 80, 128, 8, 8, 128, 3);
                TEST(5, 5, true, 8, 64, 256, 40, 80, 128, 8, 8, 128, 4);
                // cta<8,96,256> warp<40,120,128> mma<8,8,128>   WARPS[1x4]
                TEST(5, 5, true, 8, 96, 256, 40, 120, 128, 8, 8, 128, 2);
                TEST(5, 5, true, 8, 96, 256, 40, 120, 128, 8, 8, 128, 3);
                TEST(5, 5, true, 8, 96, 256, 40, 120, 128, 8, 8, 128, 4);
                // cta<1,32,384> warp<8,80,128> mma<8,8,128>   WARPS[1x2]
                TEST(5, 5, true, 1, 32, 384, 8, 80, 128, 8, 8, 128, 2);
                TEST(5, 5, true, 1, 32, 384, 8, 80, 128, 8, 8, 128, 3);
                TEST(5, 5, true, 1, 32, 384, 8, 80, 128, 8, 8, 128, 4);
                // cta<1,48,384> warp<8,120,128> mma<8,8,128>   WARPS[1x2]
                TEST(5, 5, true, 1, 48, 384, 8, 120, 128, 8, 8, 128, 2);
                TEST(5, 5, true, 1, 48, 384, 8, 120, 128, 8, 8, 128, 3);
                TEST(5, 5, true, 1, 48, 384, 8, 120, 128, 8, 8, 128, 4);
                // cta<8,32,384> warp<40,80,128> mma<8,8,128>   WARPS[1x2]
                TEST(5, 5, true, 8, 32, 384, 40, 80, 128, 8, 8, 128, 2);
                // cta<8,48,384> warp<40,120,128> mma<8,8,128>   WARPS[1x2]
                TEST(5, 5, true, 8, 48, 384, 40, 120, 128, 8, 8, 128, 2);
                // cta<1,32,384> warp<8,40,128> mma<8,8,128>   WARPS[1x4]
                TEST(5, 5, true, 1, 32, 384, 8, 40, 128, 8, 8, 128, 2);
                TEST(5, 5, true, 1, 32, 384, 8, 40, 128, 8, 8, 128, 3);
                TEST(5, 5, true, 1, 32, 384, 8, 40, 128, 8, 8, 128, 4);
                // cta<1,64,384> warp<8,80,128> mma<8,8,128>   WARPS[1x4]
                TEST(5, 5, true, 1, 64, 384, 8, 80, 128, 8, 8, 128, 2);
                TEST(5, 5, true, 1, 64, 384, 8, 80, 128, 8, 8, 128, 3);
                TEST(5, 5, true, 1, 64, 384, 8, 80, 128, 8, 8, 128, 4);
                // cta<1,96,384> warp<8,120,128> mma<8,8,128>   WARPS[1x4]
                TEST(5, 5, true, 1, 96, 384, 8, 120, 128, 8, 8, 128, 2);
                TEST(5, 5, true, 1, 96, 384, 8, 120, 128, 8, 8, 128, 3);
                TEST(5, 5, true, 1, 96, 384, 8, 120, 128, 8, 8, 128, 4);
                // cta<8,32,384> warp<40,40,128> mma<8,8,128>   WARPS[1x4]
                TEST(5, 5, true, 8, 32, 384, 40, 40, 128, 8, 8, 128, 2);
                // cta<8,64,384> warp<40,80,128> mma<8,8,128>   WARPS[1x4]
                TEST(5, 5, true, 8, 64, 384, 40, 80, 128, 8, 8, 128, 2);
                TEST(5, 5, true, 8, 64, 384, 40, 80, 128, 8, 8, 128, 3);
                // cta<8,96,384> warp<40,120,128> mma<8,8,128>   WARPS[1x4]
                TEST(5, 5, true, 8, 96, 384, 40, 120, 128, 8, 8, 128, 2);
                TEST(5, 5, true, 8, 96, 384, 40, 120, 128, 8, 8, 128, 3);
                // cta<1,32,512> warp<8,80,128> mma<8,8,128>   WARPS[1x2]
                TEST(5, 5, true, 1, 32, 512, 8, 80, 128, 8, 8, 128, 2);
                TEST(5, 5, true, 1, 32, 512, 8, 80, 128, 8, 8, 128, 3);
                TEST(5, 5, true, 1, 32, 512, 8, 80, 128, 8, 8, 128, 4);
                // cta<1,48,512> warp<8,120,128> mma<8,8,128>   WARPS[1x2]
                TEST(5, 5, true, 1, 48, 512, 8, 120, 128, 8, 8, 128, 2);
                TEST(5, 5, true, 1, 48, 512, 8, 120, 128, 8, 8, 128, 3);
                TEST(5, 5, true, 1, 48, 512, 8, 120, 128, 8, 8, 128, 4);
                // cta<8,32,512> warp<40,80,128> mma<8,8,128>   WARPS[1x2]
                TEST(5, 5, true, 8, 32, 512, 40, 80, 128, 8, 8, 128, 2);
                // cta<8,48,512> warp<40,120,128> mma<8,8,128>   WARPS[1x2]
                TEST(5, 5, true, 8, 48, 512, 40, 120, 128, 8, 8, 128, 2);
                // cta<1,32,512> warp<8,40,128> mma<8,8,128>   WARPS[1x4]
                TEST(5, 5, true, 1, 32, 512, 8, 40, 128, 8, 8, 128, 2);
                TEST(5, 5, true, 1, 32, 512, 8, 40, 128, 8, 8, 128, 3);
                TEST(5, 5, true, 1, 32, 512, 8, 40, 128, 8, 8, 128, 4);
                // cta<1,64,512> warp<8,80,128> mma<8,8,128>   WARPS[1x4]
                TEST(5, 5, true, 1, 64, 512, 8, 80, 128, 8, 8, 128, 2);
                TEST(5, 5, true, 1, 64, 512, 8, 80, 128, 8, 8, 128, 3);
                TEST(5, 5, true, 1, 64, 512, 8, 80, 128, 8, 8, 128, 4);
                // cta<1,96,512> warp<8,120,128> mma<8,8,128>   WARPS[1x4]
                TEST(5, 5, true, 1, 96, 512, 8, 120, 128, 8, 8, 128, 2);
                TEST(5, 5, true, 1, 96, 512, 8, 120, 128, 8, 8, 128, 3);
                // cta<8,32,512> warp<40,40,128> mma<8,8,128>   WARPS[1x4]
                TEST(5, 5, true, 8, 32, 512, 40, 40, 128, 8, 8, 128, 2);
                // cta<8,64,512> warp<40,80,128> mma<8,8,128>   WARPS[1x4]
                TEST(5, 5, true, 8, 64, 512, 40, 80, 128, 8, 8, 128, 2);
                // cta<8,96,512> warp<40,120,128> mma<8,8,128>   WARPS[1x4]
                TEST(5, 5, true, 8, 96, 512, 40, 120, 128, 8, 8, 128, 2);
            } else {
            }
            break;
        #endif
        default:
            printf("unsupport w%da%d\n", w_bits, x_bits);
        }
        break;
    case 6:
        switch (w_bits) {
        #ifdef W2A6
        case 2:
            if(quant_sign){
                ////// W2A6 int
                // cta<1,32,256> warp<8,32,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 1, 32, 256, 8, 32, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 1, 32, 256, 8, 32, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 1, 32, 256, 8, 32, 128, 8, 8, 128, 4);
                // cta<1,48,256> warp<8,48,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 1, 48, 256, 8, 48, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 1, 48, 256, 8, 48, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 1, 48, 256, 8, 48, 128, 8, 8, 128, 4);
                // cta<1,64,256> warp<8,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 1, 64, 256, 8, 64, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 1, 64, 256, 8, 64, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 1, 64, 256, 8, 64, 128, 8, 8, 128, 4);
                // cta<1,80,256> warp<8,80,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 1, 80, 256, 8, 80, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 1, 80, 256, 8, 80, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 1, 80, 256, 8, 80, 128, 8, 8, 128, 4);
                // cta<1,96,256> warp<8,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 1, 96, 256, 8, 96, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 1, 96, 256, 8, 96, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 1, 96, 256, 8, 96, 128, 8, 8, 128, 4);
                // cta<1,112,256> warp<8,112,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 1, 112, 256, 8, 112, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 1, 112, 256, 8, 112, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 1, 112, 256, 8, 112, 128, 8, 8, 128, 4);
                // cta<1,128,256> warp<8,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 1, 128, 256, 8, 128, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 1, 128, 256, 8, 128, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 1, 128, 256, 8, 128, 128, 8, 8, 128, 4);
                // cta<4,32,256> warp<24,32,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 4, 32, 256, 24, 32, 128, 8, 8, 128, 2);
                // cta<4,48,256> warp<24,48,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 4, 48, 256, 24, 48, 128, 8, 8, 128, 2);
                // cta<4,64,256> warp<24,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 4, 64, 256, 24, 64, 128, 8, 8, 128, 2);
                // cta<4,80,256> warp<24,80,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 4, 80, 256, 24, 80, 128, 8, 8, 128, 2);
                // cta<4,96,256> warp<24,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 4, 96, 256, 24, 96, 128, 8, 8, 128, 2);
                // cta<4,112,256> warp<24,112,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 4, 112, 256, 24, 112, 128, 8, 8, 128, 2);
                // cta<4,128,256> warp<24,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 4, 128, 256, 24, 128, 128, 8, 8, 128, 2);
                // cta<8,32,256> warp<48,32,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 8, 32, 256, 48, 32, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 8, 32, 256, 48, 32, 128, 8, 8, 128, 3);
                // cta<8,48,256> warp<48,48,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 8, 48, 256, 48, 48, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 8, 48, 256, 48, 48, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 8, 48, 256, 48, 48, 128, 8, 8, 128, 4);
                // cta<8,64,256> warp<48,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 8, 64, 256, 48, 64, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 8, 64, 256, 48, 64, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 8, 64, 256, 48, 64, 128, 8, 8, 128, 4);
                // cta<8,80,256> warp<48,80,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 8, 80, 256, 48, 80, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 8, 80, 256, 48, 80, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 8, 80, 256, 48, 80, 128, 8, 8, 128, 4);
                // cta<8,96,256> warp<48,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 8, 96, 256, 48, 96, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 8, 96, 256, 48, 96, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 8, 96, 256, 48, 96, 128, 8, 8, 128, 4);
                TEST(6, 2, true, 8, 96, 256, 48, 96, 128, 8, 8, 128, 5);
                // cta<8,112,256> warp<48,112,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 8, 112, 256, 48, 112, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 8, 112, 256, 48, 112, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 8, 112, 256, 48, 112, 128, 8, 8, 128, 4);
                TEST(6, 2, true, 8, 112, 256, 48, 112, 128, 8, 8, 128, 5);
                // cta<8,128,256> warp<48,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 8, 128, 256, 48, 128, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 8, 128, 256, 48, 128, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 8, 128, 256, 48, 128, 128, 8, 8, 128, 4);
                TEST(6, 2, true, 8, 128, 256, 48, 128, 128, 8, 8, 128, 5);
                // cta<1,32,256> warp<8,16,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 1, 32, 256, 8, 16, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 1, 32, 256, 8, 16, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 1, 32, 256, 8, 16, 128, 8, 8, 128, 4);
                // cta<1,48,256> warp<8,24,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 1, 48, 256, 8, 24, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 1, 48, 256, 8, 24, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 1, 48, 256, 8, 24, 128, 8, 8, 128, 4);
                // cta<1,64,256> warp<8,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 1, 64, 256, 8, 32, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 1, 64, 256, 8, 32, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 1, 64, 256, 8, 32, 128, 8, 8, 128, 4);
                // cta<1,80,256> warp<8,40,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 1, 80, 256, 8, 40, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 1, 80, 256, 8, 40, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 1, 80, 256, 8, 40, 128, 8, 8, 128, 4);
                // cta<1,96,256> warp<8,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 1, 96, 256, 8, 48, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 1, 96, 256, 8, 48, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 1, 96, 256, 8, 48, 128, 8, 8, 128, 4);
                // cta<1,112,256> warp<8,56,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 1, 112, 256, 8, 56, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 1, 112, 256, 8, 56, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 1, 112, 256, 8, 56, 128, 8, 8, 128, 4);
                // cta<1,128,256> warp<8,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 1, 128, 256, 8, 64, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 1, 128, 256, 8, 64, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 1, 128, 256, 8, 64, 128, 8, 8, 128, 4);
                // cta<1,256,256> warp<8,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 1, 256, 256, 8, 128, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 1, 256, 256, 8, 128, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 1, 256, 256, 8, 128, 128, 8, 8, 128, 4);
                // cta<4,32,256> warp<24,16,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 4, 32, 256, 24, 16, 128, 8, 8, 128, 2);
                // cta<4,48,256> warp<24,24,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 4, 48, 256, 24, 24, 128, 8, 8, 128, 2);
                // cta<4,64,256> warp<24,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 4, 64, 256, 24, 32, 128, 8, 8, 128, 2);
                // cta<4,80,256> warp<24,40,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 4, 80, 256, 24, 40, 128, 8, 8, 128, 2);
                // cta<4,96,256> warp<24,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 4, 96, 256, 24, 48, 128, 8, 8, 128, 2);
                // cta<4,112,256> warp<24,56,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 4, 112, 256, 24, 56, 128, 8, 8, 128, 2);
                // cta<4,128,256> warp<24,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 4, 128, 256, 24, 64, 128, 8, 8, 128, 2);
                // cta<4,256,256> warp<24,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 4, 256, 256, 24, 128, 128, 8, 8, 128, 2);
                // cta<8,32,256> warp<48,16,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 8, 32, 256, 48, 16, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 8, 32, 256, 48, 16, 128, 8, 8, 128, 3);
                // cta<8,48,256> warp<48,24,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 8, 48, 256, 48, 24, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 8, 48, 256, 48, 24, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 8, 48, 256, 48, 24, 128, 8, 8, 128, 4);
                // cta<8,64,256> warp<48,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 8, 64, 256, 48, 32, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 8, 64, 256, 48, 32, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 8, 64, 256, 48, 32, 128, 8, 8, 128, 4);
                // cta<8,80,256> warp<48,40,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 8, 80, 256, 48, 40, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 8, 80, 256, 48, 40, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 8, 80, 256, 48, 40, 128, 8, 8, 128, 4);
                // cta<8,96,256> warp<48,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 8, 96, 256, 48, 48, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 8, 96, 256, 48, 48, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 8, 96, 256, 48, 48, 128, 8, 8, 128, 4);
                TEST(6, 2, true, 8, 96, 256, 48, 48, 128, 8, 8, 128, 5);
                // cta<8,112,256> warp<48,56,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 8, 112, 256, 48, 56, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 8, 112, 256, 48, 56, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 8, 112, 256, 48, 56, 128, 8, 8, 128, 4);
                TEST(6, 2, true, 8, 112, 256, 48, 56, 128, 8, 8, 128, 5);
                // cta<8,128,256> warp<48,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 8, 128, 256, 48, 64, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 8, 128, 256, 48, 64, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 8, 128, 256, 48, 64, 128, 8, 8, 128, 4);
                TEST(6, 2, true, 8, 128, 256, 48, 64, 128, 8, 8, 128, 5);
                // cta<8,256,256> warp<48,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 8, 256, 256, 48, 128, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 8, 256, 256, 48, 128, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 8, 256, 256, 48, 128, 128, 8, 8, 128, 4);
                TEST(6, 2, true, 8, 256, 256, 48, 128, 128, 8, 8, 128, 5);
                // cta<8,32,256> warp<24,64,128> mma<8,8,128>   WARPS[2x1]
                TEST(6, 2, true, 8, 32, 256, 24, 64, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 8, 32, 256, 24, 64, 128, 8, 8, 128, 3);
                // cta<8,48,256> warp<24,96,128> mma<8,8,128>   WARPS[2x1]
                TEST(6, 2, true, 8, 48, 256, 24, 96, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 8, 48, 256, 24, 96, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 8, 48, 256, 24, 96, 128, 8, 8, 128, 4);
                // cta<8,64,256> warp<24,128,128> mma<8,8,128>   WARPS[2x1]
                TEST(6, 2, true, 8, 64, 256, 24, 128, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 8, 64, 256, 24, 128, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 8, 64, 256, 24, 128, 128, 8, 8, 128, 4);
                // cta<8,32,256> warp<24,32,128> mma<8,8,128>   WARPS[2x2]
                TEST(6, 2, true, 8, 32, 256, 24, 32, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 8, 32, 256, 24, 32, 128, 8, 8, 128, 3);
                // cta<8,48,256> warp<24,48,128> mma<8,8,128>   WARPS[2x2]
                TEST(6, 2, true, 8, 48, 256, 24, 48, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 8, 48, 256, 24, 48, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 8, 48, 256, 24, 48, 128, 8, 8, 128, 4);
                // cta<8,64,256> warp<24,64,128> mma<8,8,128>   WARPS[2x2]
                TEST(6, 2, true, 8, 64, 256, 24, 64, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 8, 64, 256, 24, 64, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 8, 64, 256, 24, 64, 128, 8, 8, 128, 4);
                // cta<8,80,256> warp<24,80,128> mma<8,8,128>   WARPS[2x2]
                TEST(6, 2, true, 8, 80, 256, 24, 80, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 8, 80, 256, 24, 80, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 8, 80, 256, 24, 80, 128, 8, 8, 128, 4);
                // cta<8,96,256> warp<24,96,128> mma<8,8,128>   WARPS[2x2]
                TEST(6, 2, true, 8, 96, 256, 24, 96, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 8, 96, 256, 24, 96, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 8, 96, 256, 24, 96, 128, 8, 8, 128, 4);
                TEST(6, 2, true, 8, 96, 256, 24, 96, 128, 8, 8, 128, 5);
                // cta<8,112,256> warp<24,112,128> mma<8,8,128>   WARPS[2x2]
                TEST(6, 2, true, 8, 112, 256, 24, 112, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 8, 112, 256, 24, 112, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 8, 112, 256, 24, 112, 128, 8, 8, 128, 4);
                TEST(6, 2, true, 8, 112, 256, 24, 112, 128, 8, 8, 128, 5);
                // cta<8,128,256> warp<24,128,128> mma<8,8,128>   WARPS[2x2]
                TEST(6, 2, true, 8, 128, 256, 24, 128, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 8, 128, 256, 24, 128, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 8, 128, 256, 24, 128, 128, 8, 8, 128, 4);
                TEST(6, 2, true, 8, 128, 256, 24, 128, 128, 8, 8, 128, 5);
                // cta<1,32,384> warp<8,32,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 1, 32, 384, 8, 32, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 1, 32, 384, 8, 32, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 1, 32, 384, 8, 32, 128, 8, 8, 128, 4);
                // cta<1,48,384> warp<8,48,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 1, 48, 384, 8, 48, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 1, 48, 384, 8, 48, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 1, 48, 384, 8, 48, 128, 8, 8, 128, 4);
                // cta<1,64,384> warp<8,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 1, 64, 384, 8, 64, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 1, 64, 384, 8, 64, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 1, 64, 384, 8, 64, 128, 8, 8, 128, 4);
                // cta<1,80,384> warp<8,80,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 1, 80, 384, 8, 80, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 1, 80, 384, 8, 80, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 1, 80, 384, 8, 80, 128, 8, 8, 128, 4);
                // cta<1,96,384> warp<8,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 1, 96, 384, 8, 96, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 1, 96, 384, 8, 96, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 1, 96, 384, 8, 96, 128, 8, 8, 128, 4);
                // cta<1,112,384> warp<8,112,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 1, 112, 384, 8, 112, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 1, 112, 384, 8, 112, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 1, 112, 384, 8, 112, 128, 8, 8, 128, 4);
                // cta<1,128,384> warp<8,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 1, 128, 384, 8, 128, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 1, 128, 384, 8, 128, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 1, 128, 384, 8, 128, 128, 8, 8, 128, 4);
                // cta<4,32,384> warp<24,32,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 4, 32, 384, 24, 32, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 4, 32, 384, 24, 32, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 4, 32, 384, 24, 32, 128, 8, 8, 128, 4);
                // cta<4,48,384> warp<24,48,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 4, 48, 384, 24, 48, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 4, 48, 384, 24, 48, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 4, 48, 384, 24, 48, 128, 8, 8, 128, 4);
                // cta<4,64,384> warp<24,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 4, 64, 384, 24, 64, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 4, 64, 384, 24, 64, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 4, 64, 384, 24, 64, 128, 8, 8, 128, 4);
                // cta<4,80,384> warp<24,80,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 4, 80, 384, 24, 80, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 4, 80, 384, 24, 80, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 4, 80, 384, 24, 80, 128, 8, 8, 128, 4);
                // cta<4,96,384> warp<24,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 4, 96, 384, 24, 96, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 4, 96, 384, 24, 96, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 4, 96, 384, 24, 96, 128, 8, 8, 128, 4);
                // cta<4,112,384> warp<24,112,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 4, 112, 384, 24, 112, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 4, 112, 384, 24, 112, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 4, 112, 384, 24, 112, 128, 8, 8, 128, 4);
                // cta<4,128,384> warp<24,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 4, 128, 384, 24, 128, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 4, 128, 384, 24, 128, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 4, 128, 384, 24, 128, 128, 8, 8, 128, 4);
                // cta<8,32,384> warp<48,32,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 8, 32, 384, 48, 32, 128, 8, 8, 128, 2);
                // cta<8,48,384> warp<48,48,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 8, 48, 384, 48, 48, 128, 8, 8, 128, 2);
                // cta<8,64,384> warp<48,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 8, 64, 384, 48, 64, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 8, 64, 384, 48, 64, 128, 8, 8, 128, 3);
                // cta<8,80,384> warp<48,80,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 8, 80, 384, 48, 80, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 8, 80, 384, 48, 80, 128, 8, 8, 128, 3);
                // cta<8,96,384> warp<48,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 8, 96, 384, 48, 96, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 8, 96, 384, 48, 96, 128, 8, 8, 128, 3);
                // cta<8,112,384> warp<48,112,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 8, 112, 384, 48, 112, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 8, 112, 384, 48, 112, 128, 8, 8, 128, 3);
                // cta<8,128,384> warp<48,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 8, 128, 384, 48, 128, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 8, 128, 384, 48, 128, 128, 8, 8, 128, 3);
                // cta<1,32,384> warp<8,16,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 1, 32, 384, 8, 16, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 1, 32, 384, 8, 16, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 1, 32, 384, 8, 16, 128, 8, 8, 128, 4);
                // cta<1,48,384> warp<8,24,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 1, 48, 384, 8, 24, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 1, 48, 384, 8, 24, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 1, 48, 384, 8, 24, 128, 8, 8, 128, 4);
                // cta<1,64,384> warp<8,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 1, 64, 384, 8, 32, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 1, 64, 384, 8, 32, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 1, 64, 384, 8, 32, 128, 8, 8, 128, 4);
                // cta<1,80,384> warp<8,40,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 1, 80, 384, 8, 40, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 1, 80, 384, 8, 40, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 1, 80, 384, 8, 40, 128, 8, 8, 128, 4);
                // cta<1,96,384> warp<8,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 1, 96, 384, 8, 48, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 1, 96, 384, 8, 48, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 1, 96, 384, 8, 48, 128, 8, 8, 128, 4);
                // cta<1,112,384> warp<8,56,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 1, 112, 384, 8, 56, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 1, 112, 384, 8, 56, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 1, 112, 384, 8, 56, 128, 8, 8, 128, 4);
                // cta<1,128,384> warp<8,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 1, 128, 384, 8, 64, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 1, 128, 384, 8, 64, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 1, 128, 384, 8, 64, 128, 8, 8, 128, 4);
                // cta<1,256,384> warp<8,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 1, 256, 384, 8, 128, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 1, 256, 384, 8, 128, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 1, 256, 384, 8, 128, 128, 8, 8, 128, 4);
                // cta<4,32,384> warp<24,16,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 4, 32, 384, 24, 16, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 4, 32, 384, 24, 16, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 4, 32, 384, 24, 16, 128, 8, 8, 128, 4);
                // cta<4,48,384> warp<24,24,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 4, 48, 384, 24, 24, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 4, 48, 384, 24, 24, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 4, 48, 384, 24, 24, 128, 8, 8, 128, 4);
                // cta<4,64,384> warp<24,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 4, 64, 384, 24, 32, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 4, 64, 384, 24, 32, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 4, 64, 384, 24, 32, 128, 8, 8, 128, 4);
                // cta<4,80,384> warp<24,40,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 4, 80, 384, 24, 40, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 4, 80, 384, 24, 40, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 4, 80, 384, 24, 40, 128, 8, 8, 128, 4);
                // cta<4,96,384> warp<24,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 4, 96, 384, 24, 48, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 4, 96, 384, 24, 48, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 4, 96, 384, 24, 48, 128, 8, 8, 128, 4);
                // cta<4,112,384> warp<24,56,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 4, 112, 384, 24, 56, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 4, 112, 384, 24, 56, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 4, 112, 384, 24, 56, 128, 8, 8, 128, 4);
                // cta<4,128,384> warp<24,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 4, 128, 384, 24, 64, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 4, 128, 384, 24, 64, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 4, 128, 384, 24, 64, 128, 8, 8, 128, 4);
                // cta<4,256,384> warp<24,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 4, 256, 384, 24, 128, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 4, 256, 384, 24, 128, 128, 8, 8, 128, 3);
                // cta<8,32,384> warp<48,16,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 8, 32, 384, 48, 16, 128, 8, 8, 128, 2);
                // cta<8,48,384> warp<48,24,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 8, 48, 384, 48, 24, 128, 8, 8, 128, 2);
                // cta<8,64,384> warp<48,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 8, 64, 384, 48, 32, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 8, 64, 384, 48, 32, 128, 8, 8, 128, 3);
                // cta<8,80,384> warp<48,40,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 8, 80, 384, 48, 40, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 8, 80, 384, 48, 40, 128, 8, 8, 128, 3);
                // cta<8,96,384> warp<48,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 8, 96, 384, 48, 48, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 8, 96, 384, 48, 48, 128, 8, 8, 128, 3);
                // cta<8,112,384> warp<48,56,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 8, 112, 384, 48, 56, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 8, 112, 384, 48, 56, 128, 8, 8, 128, 3);
                // cta<8,128,384> warp<48,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 8, 128, 384, 48, 64, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 8, 128, 384, 48, 64, 128, 8, 8, 128, 3);
                // cta<8,256,384> warp<48,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 8, 256, 384, 48, 128, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 8, 256, 384, 48, 128, 128, 8, 8, 128, 3);
                // cta<8,32,384> warp<24,64,128> mma<8,8,128>   WARPS[2x1]
                TEST(6, 2, true, 8, 32, 384, 24, 64, 128, 8, 8, 128, 2);
                // cta<8,48,384> warp<24,96,128> mma<8,8,128>   WARPS[2x1]
                TEST(6, 2, true, 8, 48, 384, 24, 96, 128, 8, 8, 128, 2);
                // cta<8,64,384> warp<24,128,128> mma<8,8,128>   WARPS[2x1]
                TEST(6, 2, true, 8, 64, 384, 24, 128, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 8, 64, 384, 24, 128, 128, 8, 8, 128, 3);
                // cta<8,32,384> warp<24,32,128> mma<8,8,128>   WARPS[2x2]
                TEST(6, 2, true, 8, 32, 384, 24, 32, 128, 8, 8, 128, 2);
                // cta<8,48,384> warp<24,48,128> mma<8,8,128>   WARPS[2x2]
                TEST(6, 2, true, 8, 48, 384, 24, 48, 128, 8, 8, 128, 2);
                // cta<8,64,384> warp<24,64,128> mma<8,8,128>   WARPS[2x2]
                TEST(6, 2, true, 8, 64, 384, 24, 64, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 8, 64, 384, 24, 64, 128, 8, 8, 128, 3);
                // cta<8,80,384> warp<24,80,128> mma<8,8,128>   WARPS[2x2]
                TEST(6, 2, true, 8, 80, 384, 24, 80, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 8, 80, 384, 24, 80, 128, 8, 8, 128, 3);
                // cta<8,96,384> warp<24,96,128> mma<8,8,128>   WARPS[2x2]
                TEST(6, 2, true, 8, 96, 384, 24, 96, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 8, 96, 384, 24, 96, 128, 8, 8, 128, 3);
                // cta<8,112,384> warp<24,112,128> mma<8,8,128>   WARPS[2x2]
                TEST(6, 2, true, 8, 112, 384, 24, 112, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 8, 112, 384, 24, 112, 128, 8, 8, 128, 3);
                // cta<8,128,384> warp<24,128,128> mma<8,8,128>   WARPS[2x2]
                TEST(6, 2, true, 8, 128, 384, 24, 128, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 8, 128, 384, 24, 128, 128, 8, 8, 128, 3);
                // cta<1,32,512> warp<8,32,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 1, 32, 512, 8, 32, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 1, 32, 512, 8, 32, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 1, 32, 512, 8, 32, 128, 8, 8, 128, 4);
                // cta<1,48,512> warp<8,48,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 1, 48, 512, 8, 48, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 1, 48, 512, 8, 48, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 1, 48, 512, 8, 48, 128, 8, 8, 128, 4);
                // cta<1,64,512> warp<8,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 1, 64, 512, 8, 64, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 1, 64, 512, 8, 64, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 1, 64, 512, 8, 64, 128, 8, 8, 128, 4);
                // cta<1,80,512> warp<8,80,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 1, 80, 512, 8, 80, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 1, 80, 512, 8, 80, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 1, 80, 512, 8, 80, 128, 8, 8, 128, 4);
                // cta<1,96,512> warp<8,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 1, 96, 512, 8, 96, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 1, 96, 512, 8, 96, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 1, 96, 512, 8, 96, 128, 8, 8, 128, 4);
                // cta<1,112,512> warp<8,112,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 1, 112, 512, 8, 112, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 1, 112, 512, 8, 112, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 1, 112, 512, 8, 112, 128, 8, 8, 128, 4);
                // cta<1,128,512> warp<8,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 1, 128, 512, 8, 128, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 1, 128, 512, 8, 128, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 1, 128, 512, 8, 128, 128, 8, 8, 128, 4);
                // cta<4,32,512> warp<24,32,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 4, 32, 512, 24, 32, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 4, 32, 512, 24, 32, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 4, 32, 512, 24, 32, 128, 8, 8, 128, 4);
                // cta<4,48,512> warp<24,48,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 4, 48, 512, 24, 48, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 4, 48, 512, 24, 48, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 4, 48, 512, 24, 48, 128, 8, 8, 128, 4);
                // cta<4,64,512> warp<24,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 4, 64, 512, 24, 64, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 4, 64, 512, 24, 64, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 4, 64, 512, 24, 64, 128, 8, 8, 128, 4);
                // cta<4,80,512> warp<24,80,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 4, 80, 512, 24, 80, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 4, 80, 512, 24, 80, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 4, 80, 512, 24, 80, 128, 8, 8, 128, 4);
                // cta<4,96,512> warp<24,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 4, 96, 512, 24, 96, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 4, 96, 512, 24, 96, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 4, 96, 512, 24, 96, 128, 8, 8, 128, 4);
                // cta<4,112,512> warp<24,112,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 4, 112, 512, 24, 112, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 4, 112, 512, 24, 112, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 4, 112, 512, 24, 112, 128, 8, 8, 128, 4);
                // cta<4,128,512> warp<24,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 4, 128, 512, 24, 128, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 4, 128, 512, 24, 128, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 4, 128, 512, 24, 128, 128, 8, 8, 128, 4);
                // cta<8,32,512> warp<48,32,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 8, 32, 512, 48, 32, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 8, 32, 512, 48, 32, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 8, 32, 512, 48, 32, 128, 8, 8, 128, 4);
                // cta<8,48,512> warp<48,48,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 8, 48, 512, 48, 48, 128, 8, 8, 128, 2);
                // cta<8,64,512> warp<48,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 8, 64, 512, 48, 64, 128, 8, 8, 128, 2);
                // cta<8,80,512> warp<48,80,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 8, 80, 512, 48, 80, 128, 8, 8, 128, 2);
                // cta<8,96,512> warp<48,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 8, 96, 512, 48, 96, 128, 8, 8, 128, 2);
                // cta<8,112,512> warp<48,112,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 8, 112, 512, 48, 112, 128, 8, 8, 128, 2);
                // cta<8,128,512> warp<48,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 2, true, 8, 128, 512, 48, 128, 128, 8, 8, 128, 2);
                // cta<1,32,512> warp<8,16,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 1, 32, 512, 8, 16, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 1, 32, 512, 8, 16, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 1, 32, 512, 8, 16, 128, 8, 8, 128, 4);
                // cta<1,48,512> warp<8,24,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 1, 48, 512, 8, 24, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 1, 48, 512, 8, 24, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 1, 48, 512, 8, 24, 128, 8, 8, 128, 4);
                // cta<1,64,512> warp<8,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 1, 64, 512, 8, 32, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 1, 64, 512, 8, 32, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 1, 64, 512, 8, 32, 128, 8, 8, 128, 4);
                // cta<1,80,512> warp<8,40,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 1, 80, 512, 8, 40, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 1, 80, 512, 8, 40, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 1, 80, 512, 8, 40, 128, 8, 8, 128, 4);
                // cta<1,96,512> warp<8,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 1, 96, 512, 8, 48, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 1, 96, 512, 8, 48, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 1, 96, 512, 8, 48, 128, 8, 8, 128, 4);
                // cta<1,112,512> warp<8,56,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 1, 112, 512, 8, 56, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 1, 112, 512, 8, 56, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 1, 112, 512, 8, 56, 128, 8, 8, 128, 4);
                // cta<1,128,512> warp<8,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 1, 128, 512, 8, 64, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 1, 128, 512, 8, 64, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 1, 128, 512, 8, 64, 128, 8, 8, 128, 4);
                // cta<1,256,512> warp<8,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 1, 256, 512, 8, 128, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 1, 256, 512, 8, 128, 128, 8, 8, 128, 3);
                // cta<4,32,512> warp<24,16,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 4, 32, 512, 24, 16, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 4, 32, 512, 24, 16, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 4, 32, 512, 24, 16, 128, 8, 8, 128, 4);
                // cta<4,48,512> warp<24,24,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 4, 48, 512, 24, 24, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 4, 48, 512, 24, 24, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 4, 48, 512, 24, 24, 128, 8, 8, 128, 4);
                // cta<4,64,512> warp<24,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 4, 64, 512, 24, 32, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 4, 64, 512, 24, 32, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 4, 64, 512, 24, 32, 128, 8, 8, 128, 4);
                // cta<4,80,512> warp<24,40,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 4, 80, 512, 24, 40, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 4, 80, 512, 24, 40, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 4, 80, 512, 24, 40, 128, 8, 8, 128, 4);
                // cta<4,96,512> warp<24,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 4, 96, 512, 24, 48, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 4, 96, 512, 24, 48, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 4, 96, 512, 24, 48, 128, 8, 8, 128, 4);
                // cta<4,112,512> warp<24,56,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 4, 112, 512, 24, 56, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 4, 112, 512, 24, 56, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 4, 112, 512, 24, 56, 128, 8, 8, 128, 4);
                // cta<4,128,512> warp<24,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 4, 128, 512, 24, 64, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 4, 128, 512, 24, 64, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 4, 128, 512, 24, 64, 128, 8, 8, 128, 4);
                // cta<4,256,512> warp<24,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 4, 256, 512, 24, 128, 128, 8, 8, 128, 2);
                // cta<8,32,512> warp<48,16,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 8, 32, 512, 48, 16, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 8, 32, 512, 48, 16, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 8, 32, 512, 48, 16, 128, 8, 8, 128, 4);
                // cta<8,48,512> warp<48,24,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 8, 48, 512, 48, 24, 128, 8, 8, 128, 2);
                // cta<8,64,512> warp<48,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 8, 64, 512, 48, 32, 128, 8, 8, 128, 2);
                // cta<8,80,512> warp<48,40,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 8, 80, 512, 48, 40, 128, 8, 8, 128, 2);
                // cta<8,96,512> warp<48,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 8, 96, 512, 48, 48, 128, 8, 8, 128, 2);
                // cta<8,112,512> warp<48,56,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 8, 112, 512, 48, 56, 128, 8, 8, 128, 2);
                // cta<8,128,512> warp<48,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 8, 128, 512, 48, 64, 128, 8, 8, 128, 2);
                // cta<8,256,512> warp<48,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 2, true, 8, 256, 512, 48, 128, 128, 8, 8, 128, 2);
                // cta<8,32,512> warp<24,64,128> mma<8,8,128>   WARPS[2x1]
                TEST(6, 2, true, 8, 32, 512, 24, 64, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 8, 32, 512, 24, 64, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 8, 32, 512, 24, 64, 128, 8, 8, 128, 4);
                // cta<8,48,512> warp<24,96,128> mma<8,8,128>   WARPS[2x1]
                TEST(6, 2, true, 8, 48, 512, 24, 96, 128, 8, 8, 128, 2);
                // cta<8,64,512> warp<24,128,128> mma<8,8,128>   WARPS[2x1]
                TEST(6, 2, true, 8, 64, 512, 24, 128, 128, 8, 8, 128, 2);
                // cta<8,32,512> warp<24,32,128> mma<8,8,128>   WARPS[2x2]
                TEST(6, 2, true, 8, 32, 512, 24, 32, 128, 8, 8, 128, 2);
                TEST(6, 2, true, 8, 32, 512, 24, 32, 128, 8, 8, 128, 3);
                TEST(6, 2, true, 8, 32, 512, 24, 32, 128, 8, 8, 128, 4);
                // cta<8,48,512> warp<24,48,128> mma<8,8,128>   WARPS[2x2]
                TEST(6, 2, true, 8, 48, 512, 24, 48, 128, 8, 8, 128, 2);
                // cta<8,64,512> warp<24,64,128> mma<8,8,128>   WARPS[2x2]
                TEST(6, 2, true, 8, 64, 512, 24, 64, 128, 8, 8, 128, 2);
                // cta<8,80,512> warp<24,80,128> mma<8,8,128>   WARPS[2x2]
                TEST(6, 2, true, 8, 80, 512, 24, 80, 128, 8, 8, 128, 2);
                // cta<8,96,512> warp<24,96,128> mma<8,8,128>   WARPS[2x2]
                TEST(6, 2, true, 8, 96, 512, 24, 96, 128, 8, 8, 128, 2);
                // cta<8,112,512> warp<24,112,128> mma<8,8,128>   WARPS[2x2]
                TEST(6, 2, true, 8, 112, 512, 24, 112, 128, 8, 8, 128, 2);
                // cta<8,128,512> warp<24,128,128> mma<8,8,128>   WARPS[2x2]
                TEST(6, 2, true, 8, 128, 512, 24, 128, 128, 8, 8, 128, 2);
            }else{}
            break;
        #endif
        #ifdef W6A6
        case 6:
            if (quant_sign) {
                ////// W6A6 int
                // cta<1,32,256> warp<8,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 6, true, 1, 32, 256, 8, 96, 128, 8, 8, 128, 2);
                TEST(6, 6, true, 1, 32, 256, 8, 96, 128, 8, 8, 128, 3);
                TEST(6, 6, true, 1, 32, 256, 8, 96, 128, 8, 8, 128, 4);
                // cta<4,32,256> warp<24,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 6, true, 4, 32, 256, 24, 96, 128, 8, 8, 128, 2);
                // cta<8,32,256> warp<48,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 6, true, 8, 32, 256, 48, 96, 128, 8, 8, 128, 2);
                TEST(6, 6, true, 8, 32, 256, 48, 96, 128, 8, 8, 128, 3);
                TEST(6, 6, true, 8, 32, 256, 48, 96, 128, 8, 8, 128, 4);
                TEST(6, 6, true, 8, 32, 256, 48, 96, 128, 8, 8, 128, 5);
                // cta<1,32,256> warp<8,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 6, true, 1, 32, 256, 8, 48, 128, 8, 8, 128, 2);
                TEST(6, 6, true, 1, 32, 256, 8, 48, 128, 8, 8, 128, 3);
                TEST(6, 6, true, 1, 32, 256, 8, 48, 128, 8, 8, 128, 4);
                // cta<1,48,256> warp<8,72,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 6, true, 1, 48, 256, 8, 72, 128, 8, 8, 128, 2);
                TEST(6, 6, true, 1, 48, 256, 8, 72, 128, 8, 8, 128, 3);
                TEST(6, 6, true, 1, 48, 256, 8, 72, 128, 8, 8, 128, 4);
                // cta<1,64,256> warp<8,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 6, true, 1, 64, 256, 8, 96, 128, 8, 8, 128, 2);
                TEST(6, 6, true, 1, 64, 256, 8, 96, 128, 8, 8, 128, 3);
                TEST(6, 6, true, 1, 64, 256, 8, 96, 128, 8, 8, 128, 4);
                // cta<1,80,256> warp<8,120,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 6, true, 1, 80, 256, 8, 120, 128, 8, 8, 128, 2);
                TEST(6, 6, true, 1, 80, 256, 8, 120, 128, 8, 8, 128, 3);
                TEST(6, 6, true, 1, 80, 256, 8, 120, 128, 8, 8, 128, 4);
                // cta<4,32,256> warp<24,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 6, true, 4, 32, 256, 24, 48, 128, 8, 8, 128, 2);
                // cta<4,48,256> warp<24,72,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 6, true, 4, 48, 256, 24, 72, 128, 8, 8, 128, 2);
                // cta<4,64,256> warp<24,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 6, true, 4, 64, 256, 24, 96, 128, 8, 8, 128, 2);
                // cta<4,80,256> warp<24,120,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 6, true, 4, 80, 256, 24, 120, 128, 8, 8, 128, 2);
                // cta<8,32,256> warp<48,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 6, true, 8, 32, 256, 48, 48, 128, 8, 8, 128, 2);
                TEST(6, 6, true, 8, 32, 256, 48, 48, 128, 8, 8, 128, 3);
                TEST(6, 6, true, 8, 32, 256, 48, 48, 128, 8, 8, 128, 4);
                TEST(6, 6, true, 8, 32, 256, 48, 48, 128, 8, 8, 128, 5);
                // cta<8,48,256> warp<48,72,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 6, true, 8, 48, 256, 48, 72, 128, 8, 8, 128, 2);
                TEST(6, 6, true, 8, 48, 256, 48, 72, 128, 8, 8, 128, 3);
                TEST(6, 6, true, 8, 48, 256, 48, 72, 128, 8, 8, 128, 4);
                TEST(6, 6, true, 8, 48, 256, 48, 72, 128, 8, 8, 128, 5);
                // cta<8,64,256> warp<48,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 6, true, 8, 64, 256, 48, 96, 128, 8, 8, 128, 2);
                TEST(6, 6, true, 8, 64, 256, 48, 96, 128, 8, 8, 128, 3);
                TEST(6, 6, true, 8, 64, 256, 48, 96, 128, 8, 8, 128, 4);
                TEST(6, 6, true, 8, 64, 256, 48, 96, 128, 8, 8, 128, 5);
                // cta<8,80,256> warp<48,120,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 6, true, 8, 80, 256, 48, 120, 128, 8, 8, 128, 2);
                TEST(6, 6, true, 8, 80, 256, 48, 120, 128, 8, 8, 128, 3);
                TEST(6, 6, true, 8, 80, 256, 48, 120, 128, 8, 8, 128, 4);
                TEST(6, 6, true, 8, 80, 256, 48, 120, 128, 8, 8, 128, 5);
                // cta<8,32,256> warp<24,96,128> mma<8,8,128>   WARPS[2x2]
                TEST(6, 6, true, 8, 32, 256, 24, 96, 128, 8, 8, 128, 2);
                TEST(6, 6, true, 8, 32, 256, 24, 96, 128, 8, 8, 128, 3);
                TEST(6, 6, true, 8, 32, 256, 24, 96, 128, 8, 8, 128, 4);
                TEST(6, 6, true, 8, 32, 256, 24, 96, 128, 8, 8, 128, 5);
                // cta<1,32,384> warp<8,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 6, true, 1, 32, 384, 8, 96, 128, 8, 8, 128, 2);
                TEST(6, 6, true, 1, 32, 384, 8, 96, 128, 8, 8, 128, 3);
                TEST(6, 6, true, 1, 32, 384, 8, 96, 128, 8, 8, 128, 4);
                // cta<4,32,384> warp<24,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 6, true, 4, 32, 384, 24, 96, 128, 8, 8, 128, 2);
                TEST(6, 6, true, 4, 32, 384, 24, 96, 128, 8, 8, 128, 3);
                TEST(6, 6, true, 4, 32, 384, 24, 96, 128, 8, 8, 128, 4);
                // cta<8,32,384> warp<48,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 6, true, 8, 32, 384, 48, 96, 128, 8, 8, 128, 2);
                TEST(6, 6, true, 8, 32, 384, 48, 96, 128, 8, 8, 128, 3);
                // cta<1,32,384> warp<8,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 6, true, 1, 32, 384, 8, 48, 128, 8, 8, 128, 2);
                TEST(6, 6, true, 1, 32, 384, 8, 48, 128, 8, 8, 128, 3);
                TEST(6, 6, true, 1, 32, 384, 8, 48, 128, 8, 8, 128, 4);
                // cta<1,48,384> warp<8,72,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 6, true, 1, 48, 384, 8, 72, 128, 8, 8, 128, 2);
                TEST(6, 6, true, 1, 48, 384, 8, 72, 128, 8, 8, 128, 3);
                TEST(6, 6, true, 1, 48, 384, 8, 72, 128, 8, 8, 128, 4);
                // cta<1,64,384> warp<8,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 6, true, 1, 64, 384, 8, 96, 128, 8, 8, 128, 2);
                TEST(6, 6, true, 1, 64, 384, 8, 96, 128, 8, 8, 128, 3);
                TEST(6, 6, true, 1, 64, 384, 8, 96, 128, 8, 8, 128, 4);
                // cta<1,80,384> warp<8,120,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 6, true, 1, 80, 384, 8, 120, 128, 8, 8, 128, 2);
                TEST(6, 6, true, 1, 80, 384, 8, 120, 128, 8, 8, 128, 3);
                TEST(6, 6, true, 1, 80, 384, 8, 120, 128, 8, 8, 128, 4);
                // cta<4,32,384> warp<24,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 6, true, 4, 32, 384, 24, 48, 128, 8, 8, 128, 2);
                TEST(6, 6, true, 4, 32, 384, 24, 48, 128, 8, 8, 128, 3);
                TEST(6, 6, true, 4, 32, 384, 24, 48, 128, 8, 8, 128, 4);
                // cta<4,48,384> warp<24,72,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 6, true, 4, 48, 384, 24, 72, 128, 8, 8, 128, 2);
                TEST(6, 6, true, 4, 48, 384, 24, 72, 128, 8, 8, 128, 3);
                TEST(6, 6, true, 4, 48, 384, 24, 72, 128, 8, 8, 128, 4);
                // cta<4,64,384> warp<24,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 6, true, 4, 64, 384, 24, 96, 128, 8, 8, 128, 2);
                TEST(6, 6, true, 4, 64, 384, 24, 96, 128, 8, 8, 128, 3);
                TEST(6, 6, true, 4, 64, 384, 24, 96, 128, 8, 8, 128, 4);
                // cta<4,80,384> warp<24,120,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 6, true, 4, 80, 384, 24, 120, 128, 8, 8, 128, 2);
                TEST(6, 6, true, 4, 80, 384, 24, 120, 128, 8, 8, 128, 3);
                TEST(6, 6, true, 4, 80, 384, 24, 120, 128, 8, 8, 128, 4);
                // cta<8,32,384> warp<48,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 6, true, 8, 32, 384, 48, 48, 128, 8, 8, 128, 2);
                TEST(6, 6, true, 8, 32, 384, 48, 48, 128, 8, 8, 128, 3);
                // cta<8,48,384> warp<48,72,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 6, true, 8, 48, 384, 48, 72, 128, 8, 8, 128, 2);
                TEST(6, 6, true, 8, 48, 384, 48, 72, 128, 8, 8, 128, 3);
                // cta<8,64,384> warp<48,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 6, true, 8, 64, 384, 48, 96, 128, 8, 8, 128, 2);
                TEST(6, 6, true, 8, 64, 384, 48, 96, 128, 8, 8, 128, 3);
                // cta<8,80,384> warp<48,120,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 6, true, 8, 80, 384, 48, 120, 128, 8, 8, 128, 2);
                TEST(6, 6, true, 8, 80, 384, 48, 120, 128, 8, 8, 128, 3);
                // cta<8,32,384> warp<24,96,128> mma<8,8,128>   WARPS[2x2]
                TEST(6, 6, true, 8, 32, 384, 24, 96, 128, 8, 8, 128, 2);
                TEST(6, 6, true, 8, 32, 384, 24, 96, 128, 8, 8, 128, 3);
                // cta<1,32,512> warp<8,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 6, true, 1, 32, 512, 8, 96, 128, 8, 8, 128, 2);
                TEST(6, 6, true, 1, 32, 512, 8, 96, 128, 8, 8, 128, 3);
                TEST(6, 6, true, 1, 32, 512, 8, 96, 128, 8, 8, 128, 4);
                // cta<4,32,512> warp<24,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 6, true, 4, 32, 512, 24, 96, 128, 8, 8, 128, 2);
                TEST(6, 6, true, 4, 32, 512, 24, 96, 128, 8, 8, 128, 3);
                TEST(6, 6, true, 4, 32, 512, 24, 96, 128, 8, 8, 128, 4);
                // cta<8,32,512> warp<48,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(6, 6, true, 8, 32, 512, 48, 96, 128, 8, 8, 128, 2);
                // cta<1,32,512> warp<8,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 6, true, 1, 32, 512, 8, 48, 128, 8, 8, 128, 2);
                TEST(6, 6, true, 1, 32, 512, 8, 48, 128, 8, 8, 128, 3);
                TEST(6, 6, true, 1, 32, 512, 8, 48, 128, 8, 8, 128, 4);
                // cta<1,48,512> warp<8,72,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 6, true, 1, 48, 512, 8, 72, 128, 8, 8, 128, 2);
                TEST(6, 6, true, 1, 48, 512, 8, 72, 128, 8, 8, 128, 3);
                TEST(6, 6, true, 1, 48, 512, 8, 72, 128, 8, 8, 128, 4);
                // cta<1,64,512> warp<8,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 6, true, 1, 64, 512, 8, 96, 128, 8, 8, 128, 2);
                TEST(6, 6, true, 1, 64, 512, 8, 96, 128, 8, 8, 128, 3);
                TEST(6, 6, true, 1, 64, 512, 8, 96, 128, 8, 8, 128, 4);
                // cta<1,80,512> warp<8,120,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 6, true, 1, 80, 512, 8, 120, 128, 8, 8, 128, 2);
                TEST(6, 6, true, 1, 80, 512, 8, 120, 128, 8, 8, 128, 3);
                // cta<4,32,512> warp<24,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 6, true, 4, 32, 512, 24, 48, 128, 8, 8, 128, 2);
                TEST(6, 6, true, 4, 32, 512, 24, 48, 128, 8, 8, 128, 3);
                TEST(6, 6, true, 4, 32, 512, 24, 48, 128, 8, 8, 128, 4);
                // cta<4,48,512> warp<24,72,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 6, true, 4, 48, 512, 24, 72, 128, 8, 8, 128, 2);
                TEST(6, 6, true, 4, 48, 512, 24, 72, 128, 8, 8, 128, 3);
                TEST(6, 6, true, 4, 48, 512, 24, 72, 128, 8, 8, 128, 4);
                // cta<4,64,512> warp<24,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 6, true, 4, 64, 512, 24, 96, 128, 8, 8, 128, 2);
                TEST(6, 6, true, 4, 64, 512, 24, 96, 128, 8, 8, 128, 3);
                // cta<4,80,512> warp<24,120,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 6, true, 4, 80, 512, 24, 120, 128, 8, 8, 128, 2);
                TEST(6, 6, true, 4, 80, 512, 24, 120, 128, 8, 8, 128, 3);
                // cta<8,32,512> warp<48,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 6, true, 8, 32, 512, 48, 48, 128, 8, 8, 128, 2);
                // cta<8,48,512> warp<48,72,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 6, true, 8, 48, 512, 48, 72, 128, 8, 8, 128, 2);
                // cta<8,64,512> warp<48,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 6, true, 8, 64, 512, 48, 96, 128, 8, 8, 128, 2);
                // cta<8,80,512> warp<48,120,128> mma<8,8,128>   WARPS[1x4]
                TEST(6, 6, true, 8, 80, 512, 48, 120, 128, 8, 8, 128, 2);
                // cta<8,32,512> warp<24,96,128> mma<8,8,128>   WARPS[2x2]
                TEST(6, 6, true, 8, 32, 512, 24, 96, 128, 8, 8, 128, 2);
            } else {
            }
            break;
        #endif
        default:
            printf("unsupport w%da%d\n", w_bits, x_bits);
        }
        break;
    case 7:
        switch (w_bits) {
        #ifdef W7A7
        case 7:
            if (quant_sign) {
                ////// W7A7 int
                // cta<1,32,256> warp<8,112,128> mma<8,8,128>   WARPS[1x2]
                TEST(7, 7, true, 1, 32, 256, 8, 112, 128, 8, 8, 128, 2);
                TEST(7, 7, true, 1, 32, 256, 8, 112, 128, 8, 8, 128, 3);
                TEST(7, 7, true, 1, 32, 256, 8, 112, 128, 8, 8, 128, 4);
                // cta<8,32,256> warp<56,112,128> mma<8,8,128>   WARPS[1x2]
                TEST(7, 7, true, 8, 32, 256, 56, 112, 128, 8, 8, 128, 2);
                TEST(7, 7, true, 8, 32, 256, 56, 112, 128, 8, 8, 128, 3);
                TEST(7, 7, true, 8, 32, 256, 56, 112, 128, 8, 8, 128, 4);
                TEST(7, 7, true, 8, 32, 256, 56, 112, 128, 8, 8, 128, 5);
                // cta<1,32,256> warp<8,56,128> mma<8,8,128>   WARPS[1x4]
                TEST(7, 7, true, 1, 32, 256, 8, 56, 128, 8, 8, 128, 2);
                TEST(7, 7, true, 1, 32, 256, 8, 56, 128, 8, 8, 128, 3);
                TEST(7, 7, true, 1, 32, 256, 8, 56, 128, 8, 8, 128, 4);
                // cta<1,64,256> warp<8,112,128> mma<8,8,128>   WARPS[1x4]
                TEST(7, 7, true, 1, 64, 256, 8, 112, 128, 8, 8, 128, 2);
                TEST(7, 7, true, 1, 64, 256, 8, 112, 128, 8, 8, 128, 3);
                TEST(7, 7, true, 1, 64, 256, 8, 112, 128, 8, 8, 128, 4);
                // cta<8,32,256> warp<56,56,128> mma<8,8,128>   WARPS[1x4]
                TEST(7, 7, true, 8, 32, 256, 56, 56, 128, 8, 8, 128, 2);
                TEST(7, 7, true, 8, 32, 256, 56, 56, 128, 8, 8, 128, 3);
                TEST(7, 7, true, 8, 32, 256, 56, 56, 128, 8, 8, 128, 4);
                TEST(7, 7, true, 8, 32, 256, 56, 56, 128, 8, 8, 128, 5);
                // cta<1,32,384> warp<8,112,128> mma<8,8,128>   WARPS[1x2]
                TEST(7, 7, true, 1, 32, 384, 8, 112, 128, 8, 8, 128, 2);
                TEST(7, 7, true, 1, 32, 384, 8, 112, 128, 8, 8, 128, 3);
                TEST(7, 7, true, 1, 32, 384, 8, 112, 128, 8, 8, 128, 4);
                // cta<8,32,384> warp<56,112,128> mma<8,8,128>   WARPS[1x2]
                TEST(7, 7, true, 8, 32, 384, 56, 112, 128, 8, 8, 128, 2);
                TEST(7, 7, true, 8, 32, 384, 56, 112, 128, 8, 8, 128, 3);
                // cta<1,32,384> warp<8,56,128> mma<8,8,128>   WARPS[1x4]
                TEST(7, 7, true, 1, 32, 384, 8, 56, 128, 8, 8, 128, 2);
                TEST(7, 7, true, 1, 32, 384, 8, 56, 128, 8, 8, 128, 3);
                TEST(7, 7, true, 1, 32, 384, 8, 56, 128, 8, 8, 128, 4);
                // cta<1,64,384> warp<8,112,128> mma<8,8,128>   WARPS[1x4]
                TEST(7, 7, true, 1, 64, 384, 8, 112, 128, 8, 8, 128, 2);
                TEST(7, 7, true, 1, 64, 384, 8, 112, 128, 8, 8, 128, 3);
                TEST(7, 7, true, 1, 64, 384, 8, 112, 128, 8, 8, 128, 4);
                // cta<8,32,384> warp<56,56,128> mma<8,8,128>   WARPS[1x4]
                TEST(7, 7, true, 8, 32, 384, 56, 56, 128, 8, 8, 128, 2);
                TEST(7, 7, true, 8, 32, 384, 56, 56, 128, 8, 8, 128, 3);
                // cta<1,32,512> warp<8,112,128> mma<8,8,128>   WARPS[1x2]
                TEST(7, 7, true, 1, 32, 512, 8, 112, 128, 8, 8, 128, 2);
                TEST(7, 7, true, 1, 32, 512, 8, 112, 128, 8, 8, 128, 3);
                TEST(7, 7, true, 1, 32, 512, 8, 112, 128, 8, 8, 128, 4);
                // cta<8,32,512> warp<56,112,128> mma<8,8,128>   WARPS[1x2]
                TEST(7, 7, true, 8, 32, 512, 56, 112, 128, 8, 8, 128, 2);
                // cta<1,32,512> warp<8,56,128> mma<8,8,128>   WARPS[1x4]
                TEST(7, 7, true, 1, 32, 512, 8, 56, 128, 8, 8, 128, 2);
                TEST(7, 7, true, 1, 32, 512, 8, 56, 128, 8, 8, 128, 3);
                TEST(7, 7, true, 1, 32, 512, 8, 56, 128, 8, 8, 128, 4);
                // cta<1,64,512> warp<8,112,128> mma<8,8,128>   WARPS[1x4]
                TEST(7, 7, true, 1, 64, 512, 8, 112, 128, 8, 8, 128, 2);
                TEST(7, 7, true, 1, 64, 512, 8, 112, 128, 8, 8, 128, 3);
                // cta<8,32,512> warp<56,56,128> mma<8,8,128>   WARPS[1x4]
                TEST(7, 7, true, 8, 32, 512, 56, 56, 128, 8, 8, 128, 2);
            } else {
            }
            break;
        #endif
        default:
            printf("unsupport w%da%d\n", w_bits, x_bits);
        }
        break;
    case 8:
        switch (w_bits) {
        #ifdef W2A8
        case 2:
            if (quant_sign) {
                ////// W2A8 int
                // cta<1,32,256> warp<8,32,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 1, 32, 256, 8, 32, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 1, 32, 256, 8, 32, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 1, 32, 256, 8, 32, 128, 8, 8, 128, 4);
                // cta<1,48,256> warp<8,48,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 1, 48, 256, 8, 48, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 1, 48, 256, 8, 48, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 1, 48, 256, 8, 48, 128, 8, 8, 128, 4);
                // cta<1,64,256> warp<8,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 1, 64, 256, 8, 64, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 1, 64, 256, 8, 64, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 1, 64, 256, 8, 64, 128, 8, 8, 128, 4);
                // cta<1,80,256> warp<8,80,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 1, 80, 256, 8, 80, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 1, 80, 256, 8, 80, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 1, 80, 256, 8, 80, 128, 8, 8, 128, 4);
                // cta<1,96,256> warp<8,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 1, 96, 256, 8, 96, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 1, 96, 256, 8, 96, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 1, 96, 256, 8, 96, 128, 8, 8, 128, 4);
                // cta<1,112,256> warp<8,112,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 1, 112, 256, 8, 112, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 1, 112, 256, 8, 112, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 1, 112, 256, 8, 112, 128, 8, 8, 128, 4);
                // cta<1,128,256> warp<8,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 1, 128, 256, 8, 128, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 1, 128, 256, 8, 128, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 1, 128, 256, 8, 128, 128, 8, 8, 128, 4);
                // cta<4,32,256> warp<32,32,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 4, 32, 256, 32, 32, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 32, 256, 32, 32, 128, 8, 8, 128, 3);
                // cta<4,48,256> warp<32,48,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 4, 48, 256, 32, 48, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 48, 256, 32, 48, 128, 8, 8, 128, 3);
                // cta<4,64,256> warp<32,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 4, 64, 256, 32, 64, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 64, 256, 32, 64, 128, 8, 8, 128, 3);
                // cta<4,80,256> warp<32,80,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 4, 80, 256, 32, 80, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 80, 256, 32, 80, 128, 8, 8, 128, 3);
                // cta<4,96,256> warp<32,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 4, 96, 256, 32, 96, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 96, 256, 32, 96, 128, 8, 8, 128, 3);
                // cta<4,112,256> warp<32,112,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 4, 112, 256, 32, 112, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 112, 256, 32, 112, 128, 8, 8, 128, 3);
                // cta<4,128,256> warp<32,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 4, 128, 256, 32, 128, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 128, 256, 32, 128, 128, 8, 8, 128, 3);
                // cta<8,32,256> warp<64,32,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 8, 32, 256, 64, 32, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 32, 256, 64, 32, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 8, 32, 256, 64, 32, 128, 8, 8, 128, 4);
                // cta<8,48,256> warp<64,48,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 8, 48, 256, 64, 48, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 48, 256, 64, 48, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 8, 48, 256, 64, 48, 128, 8, 8, 128, 4);
                TEST(8, 2, true, 8, 48, 256, 64, 48, 128, 8, 8, 128, 5);
                // cta<8,64,256> warp<64,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 8, 64, 256, 64, 64, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 64, 256, 64, 64, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 8, 64, 256, 64, 64, 128, 8, 8, 128, 4);
                TEST(8, 2, true, 8, 64, 256, 64, 64, 128, 8, 8, 128, 5);
                // cta<8,80,256> warp<64,80,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 8, 80, 256, 64, 80, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 80, 256, 64, 80, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 8, 80, 256, 64, 80, 128, 8, 8, 128, 4);
                TEST(8, 2, true, 8, 80, 256, 64, 80, 128, 8, 8, 128, 5);
                TEST(8, 2, true, 8, 80, 256, 64, 80, 128, 8, 8, 128, 6);
                // cta<8,96,256> warp<64,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 8, 96, 256, 64, 96, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 96, 256, 64, 96, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 8, 96, 256, 64, 96, 128, 8, 8, 128, 4);
                TEST(8, 2, true, 8, 96, 256, 64, 96, 128, 8, 8, 128, 5);
                TEST(8, 2, true, 8, 96, 256, 64, 96, 128, 8, 8, 128, 6);
                // cta<8,112,256> warp<64,112,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 8, 112, 256, 64, 112, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 112, 256, 64, 112, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 8, 112, 256, 64, 112, 128, 8, 8, 128, 4);
                TEST(8, 2, true, 8, 112, 256, 64, 112, 128, 8, 8, 128, 5);
                TEST(8, 2, true, 8, 112, 256, 64, 112, 128, 8, 8, 128, 6);
                // cta<1,32,256> warp<8,16,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 1, 32, 256, 8, 16, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 1, 32, 256, 8, 16, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 1, 32, 256, 8, 16, 128, 8, 8, 128, 4);
                // cta<1,48,256> warp<8,24,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 1, 48, 256, 8, 24, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 1, 48, 256, 8, 24, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 1, 48, 256, 8, 24, 128, 8, 8, 128, 4);
                // cta<1,64,256> warp<8,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 1, 64, 256, 8, 32, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 1, 64, 256, 8, 32, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 1, 64, 256, 8, 32, 128, 8, 8, 128, 4);
                // cta<1,80,256> warp<8,40,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 1, 80, 256, 8, 40, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 1, 80, 256, 8, 40, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 1, 80, 256, 8, 40, 128, 8, 8, 128, 4);
                // cta<1,96,256> warp<8,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 1, 96, 256, 8, 48, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 1, 96, 256, 8, 48, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 1, 96, 256, 8, 48, 128, 8, 8, 128, 4);
                // cta<1,112,256> warp<8,56,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 1, 112, 256, 8, 56, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 1, 112, 256, 8, 56, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 1, 112, 256, 8, 56, 128, 8, 8, 128, 4);
                // cta<1,128,256> warp<8,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 1, 128, 256, 8, 64, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 1, 128, 256, 8, 64, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 1, 128, 256, 8, 64, 128, 8, 8, 128, 4);
                // cta<1,256,256> warp<8,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 1, 256, 256, 8, 128, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 1, 256, 256, 8, 128, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 1, 256, 256, 8, 128, 128, 8, 8, 128, 4);
                // cta<4,32,256> warp<32,16,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 4, 32, 256, 32, 16, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 32, 256, 32, 16, 128, 8, 8, 128, 3);
                // cta<4,48,256> warp<32,24,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 4, 48, 256, 32, 24, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 48, 256, 32, 24, 128, 8, 8, 128, 3);
                // cta<4,64,256> warp<32,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 4, 64, 256, 32, 32, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 64, 256, 32, 32, 128, 8, 8, 128, 3);
                // cta<4,80,256> warp<32,40,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 4, 80, 256, 32, 40, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 80, 256, 32, 40, 128, 8, 8, 128, 3);
                // cta<4,96,256> warp<32,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 4, 96, 256, 32, 48, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 96, 256, 32, 48, 128, 8, 8, 128, 3);
                // cta<4,112,256> warp<32,56,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 4, 112, 256, 32, 56, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 112, 256, 32, 56, 128, 8, 8, 128, 3);
                // cta<4,128,256> warp<32,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 4, 128, 256, 32, 64, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 128, 256, 32, 64, 128, 8, 8, 128, 3);
                // cta<4,256,256> warp<32,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 4, 256, 256, 32, 128, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 256, 256, 32, 128, 128, 8, 8, 128, 3);
                // cta<8,32,256> warp<64,16,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 8, 32, 256, 64, 16, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 32, 256, 64, 16, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 8, 32, 256, 64, 16, 128, 8, 8, 128, 4);
                // cta<8,48,256> warp<64,24,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 8, 48, 256, 64, 24, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 48, 256, 64, 24, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 8, 48, 256, 64, 24, 128, 8, 8, 128, 4);
                TEST(8, 2, true, 8, 48, 256, 64, 24, 128, 8, 8, 128, 5);
                // cta<8,64,256> warp<64,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 8, 64, 256, 64, 32, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 64, 256, 64, 32, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 8, 64, 256, 64, 32, 128, 8, 8, 128, 4);
                TEST(8, 2, true, 8, 64, 256, 64, 32, 128, 8, 8, 128, 5);
                // cta<8,80,256> warp<64,40,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 8, 80, 256, 64, 40, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 80, 256, 64, 40, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 8, 80, 256, 64, 40, 128, 8, 8, 128, 4);
                TEST(8, 2, true, 8, 80, 256, 64, 40, 128, 8, 8, 128, 5);
                TEST(8, 2, true, 8, 80, 256, 64, 40, 128, 8, 8, 128, 6);
                // cta<8,96,256> warp<64,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 8, 96, 256, 64, 48, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 96, 256, 64, 48, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 8, 96, 256, 64, 48, 128, 8, 8, 128, 4);
                TEST(8, 2, true, 8, 96, 256, 64, 48, 128, 8, 8, 128, 5);
                TEST(8, 2, true, 8, 96, 256, 64, 48, 128, 8, 8, 128, 6);
                // cta<8,112,256> warp<64,56,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 8, 112, 256, 64, 56, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 112, 256, 64, 56, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 8, 112, 256, 64, 56, 128, 8, 8, 128, 4);
                TEST(8, 2, true, 8, 112, 256, 64, 56, 128, 8, 8, 128, 5);
                TEST(8, 2, true, 8, 112, 256, 64, 56, 128, 8, 8, 128, 6);
                // cta<8,128,256> warp<64,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 8, 128, 256, 64, 64, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 128, 256, 64, 64, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 8, 128, 256, 64, 64, 128, 8, 8, 128, 4);
                TEST(8, 2, true, 8, 128, 256, 64, 64, 128, 8, 8, 128, 5);
                TEST(8, 2, true, 8, 128, 256, 64, 64, 128, 8, 8, 128, 6);
                // cta<4,32,256> warp<16,64,128> mma<8,8,128>   WARPS[2x1]
                TEST(8, 2, true, 4, 32, 256, 16, 64, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 32, 256, 16, 64, 128, 8, 8, 128, 3);
                // cta<4,48,256> warp<16,96,128> mma<8,8,128>   WARPS[2x1]
                TEST(8, 2, true, 4, 48, 256, 16, 96, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 48, 256, 16, 96, 128, 8, 8, 128, 3);
                // cta<4,64,256> warp<16,128,128> mma<8,8,128>   WARPS[2x1]
                TEST(8, 2, true, 4, 64, 256, 16, 128, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 64, 256, 16, 128, 128, 8, 8, 128, 3);
                // cta<8,32,256> warp<32,64,128> mma<8,8,128>   WARPS[2x1]
                TEST(8, 2, true, 8, 32, 256, 32, 64, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 32, 256, 32, 64, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 8, 32, 256, 32, 64, 128, 8, 8, 128, 4);
                // cta<8,48,256> warp<32,96,128> mma<8,8,128>   WARPS[2x1]
                TEST(8, 2, true, 8, 48, 256, 32, 96, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 48, 256, 32, 96, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 8, 48, 256, 32, 96, 128, 8, 8, 128, 4);
                TEST(8, 2, true, 8, 48, 256, 32, 96, 128, 8, 8, 128, 5);
                // cta<8,64,256> warp<32,128,128> mma<8,8,128>   WARPS[2x1]
                TEST(8, 2, true, 8, 64, 256, 32, 128, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 64, 256, 32, 128, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 8, 64, 256, 32, 128, 128, 8, 8, 128, 4);
                TEST(8, 2, true, 8, 64, 256, 32, 128, 128, 8, 8, 128, 5);
                // cta<4,32,256> warp<16,32,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 2, true, 4, 32, 256, 16, 32, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 32, 256, 16, 32, 128, 8, 8, 128, 3);
                // cta<4,48,256> warp<16,48,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 2, true, 4, 48, 256, 16, 48, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 48, 256, 16, 48, 128, 8, 8, 128, 3);
                // cta<4,64,256> warp<16,64,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 2, true, 4, 64, 256, 16, 64, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 64, 256, 16, 64, 128, 8, 8, 128, 3);
                // cta<4,80,256> warp<16,80,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 2, true, 4, 80, 256, 16, 80, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 80, 256, 16, 80, 128, 8, 8, 128, 3);
                // cta<4,96,256> warp<16,96,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 2, true, 4, 96, 256, 16, 96, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 96, 256, 16, 96, 128, 8, 8, 128, 3);
                // cta<4,112,256> warp<16,112,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 2, true, 4, 112, 256, 16, 112, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 112, 256, 16, 112, 128, 8, 8, 128, 3);
                // cta<4,128,256> warp<16,128,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 2, true, 4, 128, 256, 16, 128, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 128, 256, 16, 128, 128, 8, 8, 128, 3);
                // cta<8,32,256> warp<32,32,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 2, true, 8, 32, 256, 32, 32, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 32, 256, 32, 32, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 8, 32, 256, 32, 32, 128, 8, 8, 128, 4);
                // cta<8,48,256> warp<32,48,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 2, true, 8, 48, 256, 32, 48, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 48, 256, 32, 48, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 8, 48, 256, 32, 48, 128, 8, 8, 128, 4);
                TEST(8, 2, true, 8, 48, 256, 32, 48, 128, 8, 8, 128, 5);
                // cta<8,64,256> warp<32,64,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 2, true, 8, 64, 256, 32, 64, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 64, 256, 32, 64, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 8, 64, 256, 32, 64, 128, 8, 8, 128, 4);
                TEST(8, 2, true, 8, 64, 256, 32, 64, 128, 8, 8, 128, 5);
                // cta<8,80,256> warp<32,80,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 2, true, 8, 80, 256, 32, 80, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 80, 256, 32, 80, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 8, 80, 256, 32, 80, 128, 8, 8, 128, 4);
                TEST(8, 2, true, 8, 80, 256, 32, 80, 128, 8, 8, 128, 5);
                TEST(8, 2, true, 8, 80, 256, 32, 80, 128, 8, 8, 128, 6);
                // cta<8,96,256> warp<32,96,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 2, true, 8, 96, 256, 32, 96, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 96, 256, 32, 96, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 8, 96, 256, 32, 96, 128, 8, 8, 128, 4);
                TEST(8, 2, true, 8, 96, 256, 32, 96, 128, 8, 8, 128, 5);
                TEST(8, 2, true, 8, 96, 256, 32, 96, 128, 8, 8, 128, 6);
                // cta<8,112,256> warp<32,112,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 2, true, 8, 112, 256, 32, 112, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 112, 256, 32, 112, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 8, 112, 256, 32, 112, 128, 8, 8, 128, 4);
                TEST(8, 2, true, 8, 112, 256, 32, 112, 128, 8, 8, 128, 5);
                TEST(8, 2, true, 8, 112, 256, 32, 112, 128, 8, 8, 128, 6);
                // cta<8,128,256> warp<32,128,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 2, true, 8, 128, 256, 32, 128, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 128, 256, 32, 128, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 8, 128, 256, 32, 128, 128, 8, 8, 128, 4);
                TEST(8, 2, true, 8, 128, 256, 32, 128, 128, 8, 8, 128, 5);
                TEST(8, 2, true, 8, 128, 256, 32, 128, 128, 8, 8, 128, 6);
                // cta<4,32,256> warp<8,64,128> mma<8,8,128>   WARPS[4x1]
                TEST(8, 2, true, 4, 32, 256, 8, 64, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 32, 256, 8, 64, 128, 8, 8, 128, 3);
                // cta<4,48,256> warp<8,96,128> mma<8,8,128>   WARPS[4x1]
                TEST(8, 2, true, 4, 48, 256, 8, 96, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 48, 256, 8, 96, 128, 8, 8, 128, 3);
                // cta<4,64,256> warp<8,128,128> mma<8,8,128>   WARPS[4x1]
                TEST(8, 2, true, 4, 64, 256, 8, 128, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 64, 256, 8, 128, 128, 8, 8, 128, 3);
                // cta<8,32,256> warp<16,64,128> mma<8,8,128>   WARPS[4x1]
                TEST(8, 2, true, 8, 32, 256, 16, 64, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 32, 256, 16, 64, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 8, 32, 256, 16, 64, 128, 8, 8, 128, 4);
                // cta<8,48,256> warp<16,96,128> mma<8,8,128>   WARPS[4x1]
                TEST(8, 2, true, 8, 48, 256, 16, 96, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 48, 256, 16, 96, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 8, 48, 256, 16, 96, 128, 8, 8, 128, 4);
                TEST(8, 2, true, 8, 48, 256, 16, 96, 128, 8, 8, 128, 5);
                // cta<8,64,256> warp<16,128,128> mma<8,8,128>   WARPS[4x1]
                TEST(8, 2, true, 8, 64, 256, 16, 128, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 64, 256, 16, 128, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 8, 64, 256, 16, 128, 128, 8, 8, 128, 4);
                TEST(8, 2, true, 8, 64, 256, 16, 128, 128, 8, 8, 128, 5);
                // cta<1,32,384> warp<8,32,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 1, 32, 384, 8, 32, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 1, 32, 384, 8, 32, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 1, 32, 384, 8, 32, 128, 8, 8, 128, 4);
                // cta<1,48,384> warp<8,48,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 1, 48, 384, 8, 48, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 1, 48, 384, 8, 48, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 1, 48, 384, 8, 48, 128, 8, 8, 128, 4);
                // cta<1,64,384> warp<8,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 1, 64, 384, 8, 64, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 1, 64, 384, 8, 64, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 1, 64, 384, 8, 64, 128, 8, 8, 128, 4);
                // cta<1,80,384> warp<8,80,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 1, 80, 384, 8, 80, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 1, 80, 384, 8, 80, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 1, 80, 384, 8, 80, 128, 8, 8, 128, 4);
                // cta<1,96,384> warp<8,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 1, 96, 384, 8, 96, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 1, 96, 384, 8, 96, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 1, 96, 384, 8, 96, 128, 8, 8, 128, 4);
                // cta<1,112,384> warp<8,112,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 1, 112, 384, 8, 112, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 1, 112, 384, 8, 112, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 1, 112, 384, 8, 112, 128, 8, 8, 128, 4);
                // cta<1,128,384> warp<8,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 1, 128, 384, 8, 128, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 1, 128, 384, 8, 128, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 1, 128, 384, 8, 128, 128, 8, 8, 128, 4);
                // cta<4,32,384> warp<32,32,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 4, 32, 384, 32, 32, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 32, 384, 32, 32, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 4, 32, 384, 32, 32, 128, 8, 8, 128, 4);
                // cta<4,48,384> warp<32,48,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 4, 48, 384, 32, 48, 128, 8, 8, 128, 2);
                // cta<4,64,384> warp<32,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 4, 64, 384, 32, 64, 128, 8, 8, 128, 2);
                // cta<4,80,384> warp<32,80,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 4, 80, 384, 32, 80, 128, 8, 8, 128, 2);
                // cta<4,96,384> warp<32,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 4, 96, 384, 32, 96, 128, 8, 8, 128, 2);
                // cta<4,112,384> warp<32,112,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 4, 112, 384, 32, 112, 128, 8, 8, 128, 2);
                // cta<4,128,384> warp<32,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 4, 128, 384, 32, 128, 128, 8, 8, 128, 2);
                // cta<8,32,384> warp<64,32,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 8, 32, 384, 64, 32, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 32, 384, 64, 32, 128, 8, 8, 128, 3);
                // cta<8,48,384> warp<64,48,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 8, 48, 384, 64, 48, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 48, 384, 64, 48, 128, 8, 8, 128, 3);
                // cta<8,64,384> warp<64,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 8, 64, 384, 64, 64, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 64, 384, 64, 64, 128, 8, 8, 128, 3);
                // cta<8,80,384> warp<64,80,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 8, 80, 384, 64, 80, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 80, 384, 64, 80, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 8, 80, 384, 64, 80, 128, 8, 8, 128, 4);
                // cta<8,96,384> warp<64,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 8, 96, 384, 64, 96, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 96, 384, 64, 96, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 8, 96, 384, 64, 96, 128, 8, 8, 128, 4);
                // cta<8,112,384> warp<64,112,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 8, 112, 384, 64, 112, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 112, 384, 64, 112, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 8, 112, 384, 64, 112, 128, 8, 8, 128, 4);
                // cta<1,32,384> warp<8,16,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 1, 32, 384, 8, 16, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 1, 32, 384, 8, 16, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 1, 32, 384, 8, 16, 128, 8, 8, 128, 4);
                // cta<1,48,384> warp<8,24,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 1, 48, 384, 8, 24, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 1, 48, 384, 8, 24, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 1, 48, 384, 8, 24, 128, 8, 8, 128, 4);
                // cta<1,64,384> warp<8,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 1, 64, 384, 8, 32, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 1, 64, 384, 8, 32, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 1, 64, 384, 8, 32, 128, 8, 8, 128, 4);
                // cta<1,80,384> warp<8,40,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 1, 80, 384, 8, 40, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 1, 80, 384, 8, 40, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 1, 80, 384, 8, 40, 128, 8, 8, 128, 4);
                // cta<1,96,384> warp<8,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 1, 96, 384, 8, 48, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 1, 96, 384, 8, 48, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 1, 96, 384, 8, 48, 128, 8, 8, 128, 4);
                // cta<1,112,384> warp<8,56,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 1, 112, 384, 8, 56, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 1, 112, 384, 8, 56, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 1, 112, 384, 8, 56, 128, 8, 8, 128, 4);
                // cta<1,128,384> warp<8,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 1, 128, 384, 8, 64, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 1, 128, 384, 8, 64, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 1, 128, 384, 8, 64, 128, 8, 8, 128, 4);
                // cta<1,256,384> warp<8,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 1, 256, 384, 8, 128, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 1, 256, 384, 8, 128, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 1, 256, 384, 8, 128, 128, 8, 8, 128, 4);
                // cta<4,32,384> warp<32,16,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 4, 32, 384, 32, 16, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 32, 384, 32, 16, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 4, 32, 384, 32, 16, 128, 8, 8, 128, 4);
                // cta<4,48,384> warp<32,24,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 4, 48, 384, 32, 24, 128, 8, 8, 128, 2);
                // cta<4,64,384> warp<32,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 4, 64, 384, 32, 32, 128, 8, 8, 128, 2);
                // cta<4,80,384> warp<32,40,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 4, 80, 384, 32, 40, 128, 8, 8, 128, 2);
                // cta<4,96,384> warp<32,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 4, 96, 384, 32, 48, 128, 8, 8, 128, 2);
                // cta<4,112,384> warp<32,56,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 4, 112, 384, 32, 56, 128, 8, 8, 128, 2);
                // cta<4,128,384> warp<32,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 4, 128, 384, 32, 64, 128, 8, 8, 128, 2);
                // cta<4,256,384> warp<32,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 4, 256, 384, 32, 128, 128, 8, 8, 128, 2);
                // cta<8,32,384> warp<64,16,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 8, 32, 384, 64, 16, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 32, 384, 64, 16, 128, 8, 8, 128, 3);
                // cta<8,48,384> warp<64,24,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 8, 48, 384, 64, 24, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 48, 384, 64, 24, 128, 8, 8, 128, 3);
                // cta<8,64,384> warp<64,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 8, 64, 384, 64, 32, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 64, 384, 64, 32, 128, 8, 8, 128, 3);
                // cta<8,80,384> warp<64,40,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 8, 80, 384, 64, 40, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 80, 384, 64, 40, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 8, 80, 384, 64, 40, 128, 8, 8, 128, 4);
                // cta<8,96,384> warp<64,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 8, 96, 384, 64, 48, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 96, 384, 64, 48, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 8, 96, 384, 64, 48, 128, 8, 8, 128, 4);
                // cta<8,112,384> warp<64,56,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 8, 112, 384, 64, 56, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 112, 384, 64, 56, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 8, 112, 384, 64, 56, 128, 8, 8, 128, 4);
                // cta<8,128,384> warp<64,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 8, 128, 384, 64, 64, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 128, 384, 64, 64, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 8, 128, 384, 64, 64, 128, 8, 8, 128, 4);
                // cta<4,32,384> warp<16,64,128> mma<8,8,128>   WARPS[2x1]
                TEST(8, 2, true, 4, 32, 384, 16, 64, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 32, 384, 16, 64, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 4, 32, 384, 16, 64, 128, 8, 8, 128, 4);
                // cta<4,48,384> warp<16,96,128> mma<8,8,128>   WARPS[2x1]
                TEST(8, 2, true, 4, 48, 384, 16, 96, 128, 8, 8, 128, 2);
                // cta<4,64,384> warp<16,128,128> mma<8,8,128>   WARPS[2x1]
                TEST(8, 2, true, 4, 64, 384, 16, 128, 128, 8, 8, 128, 2);
                // cta<8,32,384> warp<32,64,128> mma<8,8,128>   WARPS[2x1]
                TEST(8, 2, true, 8, 32, 384, 32, 64, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 32, 384, 32, 64, 128, 8, 8, 128, 3);
                // cta<8,48,384> warp<32,96,128> mma<8,8,128>   WARPS[2x1]
                TEST(8, 2, true, 8, 48, 384, 32, 96, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 48, 384, 32, 96, 128, 8, 8, 128, 3);
                // cta<8,64,384> warp<32,128,128> mma<8,8,128>   WARPS[2x1]
                TEST(8, 2, true, 8, 64, 384, 32, 128, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 64, 384, 32, 128, 128, 8, 8, 128, 3);
                // cta<4,32,384> warp<16,32,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 2, true, 4, 32, 384, 16, 32, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 32, 384, 16, 32, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 4, 32, 384, 16, 32, 128, 8, 8, 128, 4);
                // cta<4,48,384> warp<16,48,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 2, true, 4, 48, 384, 16, 48, 128, 8, 8, 128, 2);
                // cta<4,64,384> warp<16,64,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 2, true, 4, 64, 384, 16, 64, 128, 8, 8, 128, 2);
                // cta<4,80,384> warp<16,80,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 2, true, 4, 80, 384, 16, 80, 128, 8, 8, 128, 2);
                // cta<4,96,384> warp<16,96,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 2, true, 4, 96, 384, 16, 96, 128, 8, 8, 128, 2);
                // cta<4,112,384> warp<16,112,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 2, true, 4, 112, 384, 16, 112, 128, 8, 8, 128, 2);
                // cta<4,128,384> warp<16,128,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 2, true, 4, 128, 384, 16, 128, 128, 8, 8, 128, 2);
                // cta<8,32,384> warp<32,32,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 2, true, 8, 32, 384, 32, 32, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 32, 384, 32, 32, 128, 8, 8, 128, 3);
                // cta<8,48,384> warp<32,48,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 2, true, 8, 48, 384, 32, 48, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 48, 384, 32, 48, 128, 8, 8, 128, 3);
                // cta<8,64,384> warp<32,64,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 2, true, 8, 64, 384, 32, 64, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 64, 384, 32, 64, 128, 8, 8, 128, 3);
                // cta<8,80,384> warp<32,80,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 2, true, 8, 80, 384, 32, 80, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 80, 384, 32, 80, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 8, 80, 384, 32, 80, 128, 8, 8, 128, 4);
                // cta<8,96,384> warp<32,96,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 2, true, 8, 96, 384, 32, 96, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 96, 384, 32, 96, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 8, 96, 384, 32, 96, 128, 8, 8, 128, 4);
                // cta<8,112,384> warp<32,112,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 2, true, 8, 112, 384, 32, 112, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 112, 384, 32, 112, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 8, 112, 384, 32, 112, 128, 8, 8, 128, 4);
                // cta<8,128,384> warp<32,128,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 2, true, 8, 128, 384, 32, 128, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 128, 384, 32, 128, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 8, 128, 384, 32, 128, 128, 8, 8, 128, 4);
                // cta<4,32,384> warp<8,64,128> mma<8,8,128>   WARPS[4x1]
                TEST(8, 2, true, 4, 32, 384, 8, 64, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 32, 384, 8, 64, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 4, 32, 384, 8, 64, 128, 8, 8, 128, 4);
                // cta<4,48,384> warp<8,96,128> mma<8,8,128>   WARPS[4x1]
                TEST(8, 2, true, 4, 48, 384, 8, 96, 128, 8, 8, 128, 2);
                // cta<4,64,384> warp<8,128,128> mma<8,8,128>   WARPS[4x1]
                TEST(8, 2, true, 4, 64, 384, 8, 128, 128, 8, 8, 128, 2);
                // cta<8,32,384> warp<16,64,128> mma<8,8,128>   WARPS[4x1]
                TEST(8, 2, true, 8, 32, 384, 16, 64, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 32, 384, 16, 64, 128, 8, 8, 128, 3);
                // cta<8,48,384> warp<16,96,128> mma<8,8,128>   WARPS[4x1]
                TEST(8, 2, true, 8, 48, 384, 16, 96, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 48, 384, 16, 96, 128, 8, 8, 128, 3);
                // cta<8,64,384> warp<16,128,128> mma<8,8,128>   WARPS[4x1]
                TEST(8, 2, true, 8, 64, 384, 16, 128, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 64, 384, 16, 128, 128, 8, 8, 128, 3);
                // cta<1,32,512> warp<8,32,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 1, 32, 512, 8, 32, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 1, 32, 512, 8, 32, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 1, 32, 512, 8, 32, 128, 8, 8, 128, 4);
                // cta<1,48,512> warp<8,48,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 1, 48, 512, 8, 48, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 1, 48, 512, 8, 48, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 1, 48, 512, 8, 48, 128, 8, 8, 128, 4);
                // cta<1,64,512> warp<8,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 1, 64, 512, 8, 64, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 1, 64, 512, 8, 64, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 1, 64, 512, 8, 64, 128, 8, 8, 128, 4);
                // cta<1,80,512> warp<8,80,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 1, 80, 512, 8, 80, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 1, 80, 512, 8, 80, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 1, 80, 512, 8, 80, 128, 8, 8, 128, 4);
                // cta<1,96,512> warp<8,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 1, 96, 512, 8, 96, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 1, 96, 512, 8, 96, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 1, 96, 512, 8, 96, 128, 8, 8, 128, 4);
                // cta<1,112,512> warp<8,112,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 1, 112, 512, 8, 112, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 1, 112, 512, 8, 112, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 1, 112, 512, 8, 112, 128, 8, 8, 128, 4);
                // cta<1,128,512> warp<8,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 1, 128, 512, 8, 128, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 1, 128, 512, 8, 128, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 1, 128, 512, 8, 128, 128, 8, 8, 128, 4);
                // cta<4,32,512> warp<32,32,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 4, 32, 512, 32, 32, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 32, 512, 32, 32, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 4, 32, 512, 32, 32, 128, 8, 8, 128, 4);
                // cta<4,48,512> warp<32,48,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 4, 48, 512, 32, 48, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 48, 512, 32, 48, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 4, 48, 512, 32, 48, 128, 8, 8, 128, 4);
                // cta<4,64,512> warp<32,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 4, 64, 512, 32, 64, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 64, 512, 32, 64, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 4, 64, 512, 32, 64, 128, 8, 8, 128, 4);
                // cta<4,80,512> warp<32,80,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 4, 80, 512, 32, 80, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 80, 512, 32, 80, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 4, 80, 512, 32, 80, 128, 8, 8, 128, 4);
                // cta<4,96,512> warp<32,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 4, 96, 512, 32, 96, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 96, 512, 32, 96, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 4, 96, 512, 32, 96, 128, 8, 8, 128, 4);
                // cta<4,112,512> warp<32,112,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 4, 112, 512, 32, 112, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 112, 512, 32, 112, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 4, 112, 512, 32, 112, 128, 8, 8, 128, 4);
                // cta<4,128,512> warp<32,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 4, 128, 512, 32, 128, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 128, 512, 32, 128, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 4, 128, 512, 32, 128, 128, 8, 8, 128, 4);
                // cta<8,32,512> warp<64,32,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 8, 32, 512, 64, 32, 128, 8, 8, 128, 2);
                // cta<8,48,512> warp<64,48,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 8, 48, 512, 64, 48, 128, 8, 8, 128, 2);
                // cta<8,64,512> warp<64,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 8, 64, 512, 64, 64, 128, 8, 8, 128, 2);
                // cta<8,80,512> warp<64,80,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 8, 80, 512, 64, 80, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 80, 512, 64, 80, 128, 8, 8, 128, 3);
                // cta<8,96,512> warp<64,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 8, 96, 512, 64, 96, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 96, 512, 64, 96, 128, 8, 8, 128, 3);
                // cta<8,112,512> warp<64,112,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 2, true, 8, 112, 512, 64, 112, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 112, 512, 64, 112, 128, 8, 8, 128, 3);
                // cta<1,32,512> warp<8,16,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 1, 32, 512, 8, 16, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 1, 32, 512, 8, 16, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 1, 32, 512, 8, 16, 128, 8, 8, 128, 4);
                // cta<1,48,512> warp<8,24,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 1, 48, 512, 8, 24, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 1, 48, 512, 8, 24, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 1, 48, 512, 8, 24, 128, 8, 8, 128, 4);
                // cta<1,64,512> warp<8,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 1, 64, 512, 8, 32, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 1, 64, 512, 8, 32, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 1, 64, 512, 8, 32, 128, 8, 8, 128, 4);
                // cta<1,80,512> warp<8,40,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 1, 80, 512, 8, 40, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 1, 80, 512, 8, 40, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 1, 80, 512, 8, 40, 128, 8, 8, 128, 4);
                // cta<1,96,512> warp<8,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 1, 96, 512, 8, 48, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 1, 96, 512, 8, 48, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 1, 96, 512, 8, 48, 128, 8, 8, 128, 4);
                // cta<1,112,512> warp<8,56,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 1, 112, 512, 8, 56, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 1, 112, 512, 8, 56, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 1, 112, 512, 8, 56, 128, 8, 8, 128, 4);
                // cta<1,128,512> warp<8,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 1, 128, 512, 8, 64, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 1, 128, 512, 8, 64, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 1, 128, 512, 8, 64, 128, 8, 8, 128, 4);
                // cta<1,256,512> warp<8,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 1, 256, 512, 8, 128, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 1, 256, 512, 8, 128, 128, 8, 8, 128, 3);
                // cta<4,32,512> warp<32,16,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 4, 32, 512, 32, 16, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 32, 512, 32, 16, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 4, 32, 512, 32, 16, 128, 8, 8, 128, 4);
                // cta<4,48,512> warp<32,24,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 4, 48, 512, 32, 24, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 48, 512, 32, 24, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 4, 48, 512, 32, 24, 128, 8, 8, 128, 4);
                // cta<4,64,512> warp<32,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 4, 64, 512, 32, 32, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 64, 512, 32, 32, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 4, 64, 512, 32, 32, 128, 8, 8, 128, 4);
                // cta<4,80,512> warp<32,40,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 4, 80, 512, 32, 40, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 80, 512, 32, 40, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 4, 80, 512, 32, 40, 128, 8, 8, 128, 4);
                // cta<4,96,512> warp<32,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 4, 96, 512, 32, 48, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 96, 512, 32, 48, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 4, 96, 512, 32, 48, 128, 8, 8, 128, 4);
                // cta<4,112,512> warp<32,56,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 4, 112, 512, 32, 56, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 112, 512, 32, 56, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 4, 112, 512, 32, 56, 128, 8, 8, 128, 4);
                // cta<4,128,512> warp<32,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 4, 128, 512, 32, 64, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 128, 512, 32, 64, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 4, 128, 512, 32, 64, 128, 8, 8, 128, 4);
                // cta<4,256,512> warp<32,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 4, 256, 512, 32, 128, 128, 8, 8, 128, 2);
                // cta<8,32,512> warp<64,16,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 8, 32, 512, 64, 16, 128, 8, 8, 128, 2);
                // cta<8,48,512> warp<64,24,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 8, 48, 512, 64, 24, 128, 8, 8, 128, 2);
                // cta<8,64,512> warp<64,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 8, 64, 512, 64, 32, 128, 8, 8, 128, 2);
                // cta<8,80,512> warp<64,40,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 8, 80, 512, 64, 40, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 80, 512, 64, 40, 128, 8, 8, 128, 3);
                // cta<8,96,512> warp<64,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 8, 96, 512, 64, 48, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 96, 512, 64, 48, 128, 8, 8, 128, 3);
                // cta<8,112,512> warp<64,56,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 8, 112, 512, 64, 56, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 112, 512, 64, 56, 128, 8, 8, 128, 3);
                // cta<8,128,512> warp<64,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 2, true, 8, 128, 512, 64, 64, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 128, 512, 64, 64, 128, 8, 8, 128, 3);
                // cta<4,32,512> warp<16,64,128> mma<8,8,128>   WARPS[2x1]
                TEST(8, 2, true, 4, 32, 512, 16, 64, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 32, 512, 16, 64, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 4, 32, 512, 16, 64, 128, 8, 8, 128, 4);
                // cta<4,48,512> warp<16,96,128> mma<8,8,128>   WARPS[2x1]
                TEST(8, 2, true, 4, 48, 512, 16, 96, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 48, 512, 16, 96, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 4, 48, 512, 16, 96, 128, 8, 8, 128, 4);
                // cta<4,64,512> warp<16,128,128> mma<8,8,128>   WARPS[2x1]
                TEST(8, 2, true, 4, 64, 512, 16, 128, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 64, 512, 16, 128, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 4, 64, 512, 16, 128, 128, 8, 8, 128, 4);
                // cta<8,32,512> warp<32,64,128> mma<8,8,128>   WARPS[2x1]
                TEST(8, 2, true, 8, 32, 512, 32, 64, 128, 8, 8, 128, 2);
                // cta<8,48,512> warp<32,96,128> mma<8,8,128>   WARPS[2x1]
                TEST(8, 2, true, 8, 48, 512, 32, 96, 128, 8, 8, 128, 2);
                // cta<8,64,512> warp<32,128,128> mma<8,8,128>   WARPS[2x1]
                TEST(8, 2, true, 8, 64, 512, 32, 128, 128, 8, 8, 128, 2);
                // cta<4,32,512> warp<16,32,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 2, true, 4, 32, 512, 16, 32, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 32, 512, 16, 32, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 4, 32, 512, 16, 32, 128, 8, 8, 128, 4);
                // cta<4,48,512> warp<16,48,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 2, true, 4, 48, 512, 16, 48, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 48, 512, 16, 48, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 4, 48, 512, 16, 48, 128, 8, 8, 128, 4);
                // cta<4,64,512> warp<16,64,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 2, true, 4, 64, 512, 16, 64, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 64, 512, 16, 64, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 4, 64, 512, 16, 64, 128, 8, 8, 128, 4);
                // cta<4,80,512> warp<16,80,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 2, true, 4, 80, 512, 16, 80, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 80, 512, 16, 80, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 4, 80, 512, 16, 80, 128, 8, 8, 128, 4);
                // cta<4,96,512> warp<16,96,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 2, true, 4, 96, 512, 16, 96, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 96, 512, 16, 96, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 4, 96, 512, 16, 96, 128, 8, 8, 128, 4);
                // cta<4,112,512> warp<16,112,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 2, true, 4, 112, 512, 16, 112, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 112, 512, 16, 112, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 4, 112, 512, 16, 112, 128, 8, 8, 128, 4);
                // cta<4,128,512> warp<16,128,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 2, true, 4, 128, 512, 16, 128, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 128, 512, 16, 128, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 4, 128, 512, 16, 128, 128, 8, 8, 128, 4);
                // cta<8,32,512> warp<32,32,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 2, true, 8, 32, 512, 32, 32, 128, 8, 8, 128, 2);
                // cta<8,48,512> warp<32,48,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 2, true, 8, 48, 512, 32, 48, 128, 8, 8, 128, 2);
                // cta<8,64,512> warp<32,64,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 2, true, 8, 64, 512, 32, 64, 128, 8, 8, 128, 2);
                // cta<8,80,512> warp<32,80,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 2, true, 8, 80, 512, 32, 80, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 80, 512, 32, 80, 128, 8, 8, 128, 3);
                // cta<8,96,512> warp<32,96,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 2, true, 8, 96, 512, 32, 96, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 96, 512, 32, 96, 128, 8, 8, 128, 3);
                // cta<8,112,512> warp<32,112,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 2, true, 8, 112, 512, 32, 112, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 112, 512, 32, 112, 128, 8, 8, 128, 3);
                // cta<8,128,512> warp<32,128,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 2, true, 8, 128, 512, 32, 128, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 8, 128, 512, 32, 128, 128, 8, 8, 128, 3);
                // cta<4,32,512> warp<8,64,128> mma<8,8,128>   WARPS[4x1]
                TEST(8, 2, true, 4, 32, 512, 8, 64, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 32, 512, 8, 64, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 4, 32, 512, 8, 64, 128, 8, 8, 128, 4);
                // cta<4,48,512> warp<8,96,128> mma<8,8,128>   WARPS[4x1]
                TEST(8, 2, true, 4, 48, 512, 8, 96, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 48, 512, 8, 96, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 4, 48, 512, 8, 96, 128, 8, 8, 128, 4);
                // cta<4,64,512> warp<8,128,128> mma<8,8,128>   WARPS[4x1]
                TEST(8, 2, true, 4, 64, 512, 8, 128, 128, 8, 8, 128, 2);
                TEST(8, 2, true, 4, 64, 512, 8, 128, 128, 8, 8, 128, 3);
                TEST(8, 2, true, 4, 64, 512, 8, 128, 128, 8, 8, 128, 4);
                // cta<8,32,512> warp<16,64,128> mma<8,8,128>   WARPS[4x1]
                TEST(8, 2, true, 8, 32, 512, 16, 64, 128, 8, 8, 128, 2);
                // cta<8,48,512> warp<16,96,128> mma<8,8,128>   WARPS[4x1]
                TEST(8, 2, true, 8, 48, 512, 16, 96, 128, 8, 8, 128, 2);
                // cta<8,64,512> warp<16,128,128> mma<8,8,128>   WARPS[4x1]
                TEST(8, 2, true, 8, 64, 512, 16, 128, 128, 8, 8, 128, 2);
            } else {
            }
            break;
        #endif
        #ifdef W3A8
        case 3:
            if(quant_sign){
                ////// W3A8 int
                // cta<1,32,256> warp<8,48,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 3, true, 1, 32, 256, 8, 48, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 1, 32, 256, 8, 48, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 1, 32, 256, 8, 48, 128, 8, 8, 128, 4);
                // cta<1,48,256> warp<8,72,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 3, true, 1, 48, 256, 8, 72, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 1, 48, 256, 8, 72, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 1, 48, 256, 8, 72, 128, 8, 8, 128, 4);
                // cta<1,64,256> warp<8,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 3, true, 1, 64, 256, 8, 96, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 1, 64, 256, 8, 96, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 1, 64, 256, 8, 96, 128, 8, 8, 128, 4);
                // cta<1,80,256> warp<8,120,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 3, true, 1, 80, 256, 8, 120, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 1, 80, 256, 8, 120, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 1, 80, 256, 8, 120, 128, 8, 8, 128, 4);
                // cta<4,32,256> warp<32,48,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 3, true, 4, 32, 256, 32, 48, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 4, 32, 256, 32, 48, 128, 8, 8, 128, 3);
                // cta<4,48,256> warp<32,72,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 3, true, 4, 48, 256, 32, 72, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 4, 48, 256, 32, 72, 128, 8, 8, 128, 3);
                // cta<4,64,256> warp<32,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 3, true, 4, 64, 256, 32, 96, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 4, 64, 256, 32, 96, 128, 8, 8, 128, 3);
                // cta<4,80,256> warp<32,120,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 3, true, 4, 80, 256, 32, 120, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 4, 80, 256, 32, 120, 128, 8, 8, 128, 3);
                // cta<8,32,256> warp<64,48,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 3, true, 8, 32, 256, 64, 48, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 8, 32, 256, 64, 48, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 8, 32, 256, 64, 48, 128, 8, 8, 128, 4);
                TEST(8, 3, true, 8, 32, 256, 64, 48, 128, 8, 8, 128, 5);
                // cta<8,48,256> warp<64,72,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 3, true, 8, 48, 256, 64, 72, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 8, 48, 256, 64, 72, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 8, 48, 256, 64, 72, 128, 8, 8, 128, 4);
                TEST(8, 3, true, 8, 48, 256, 64, 72, 128, 8, 8, 128, 5);
                // cta<8,64,256> warp<64,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 3, true, 8, 64, 256, 64, 96, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 8, 64, 256, 64, 96, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 8, 64, 256, 64, 96, 128, 8, 8, 128, 4);
                TEST(8, 3, true, 8, 64, 256, 64, 96, 128, 8, 8, 128, 5);
                TEST(8, 3, true, 8, 64, 256, 64, 96, 128, 8, 8, 128, 6);
                // cta<1,32,256> warp<8,24,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 3, true, 1, 32, 256, 8, 24, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 1, 32, 256, 8, 24, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 1, 32, 256, 8, 24, 128, 8, 8, 128, 4);
                // cta<1,64,256> warp<8,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 3, true, 1, 64, 256, 8, 48, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 1, 64, 256, 8, 48, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 1, 64, 256, 8, 48, 128, 8, 8, 128, 4);
                // cta<1,96,256> warp<8,72,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 3, true, 1, 96, 256, 8, 72, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 1, 96, 256, 8, 72, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 1, 96, 256, 8, 72, 128, 8, 8, 128, 4);
                // cta<1,128,256> warp<8,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 3, true, 1, 128, 256, 8, 96, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 1, 128, 256, 8, 96, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 1, 128, 256, 8, 96, 128, 8, 8, 128, 4);
                // cta<4,32,256> warp<32,24,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 3, true, 4, 32, 256, 32, 24, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 4, 32, 256, 32, 24, 128, 8, 8, 128, 3);
                // cta<4,64,256> warp<32,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 3, true, 4, 64, 256, 32, 48, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 4, 64, 256, 32, 48, 128, 8, 8, 128, 3);
                // cta<4,96,256> warp<32,72,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 3, true, 4, 96, 256, 32, 72, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 4, 96, 256, 32, 72, 128, 8, 8, 128, 3);
                // cta<4,128,256> warp<32,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 3, true, 4, 128, 256, 32, 96, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 4, 128, 256, 32, 96, 128, 8, 8, 128, 3);
                // cta<8,32,256> warp<64,24,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 3, true, 8, 32, 256, 64, 24, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 8, 32, 256, 64, 24, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 8, 32, 256, 64, 24, 128, 8, 8, 128, 4);
                TEST(8, 3, true, 8, 32, 256, 64, 24, 128, 8, 8, 128, 5);
                // cta<8,64,256> warp<64,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 3, true, 8, 64, 256, 64, 48, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 8, 64, 256, 64, 48, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 8, 64, 256, 64, 48, 128, 8, 8, 128, 4);
                TEST(8, 3, true, 8, 64, 256, 64, 48, 128, 8, 8, 128, 5);
                TEST(8, 3, true, 8, 64, 256, 64, 48, 128, 8, 8, 128, 6);
                // cta<8,96,256> warp<64,72,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 3, true, 8, 96, 256, 64, 72, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 8, 96, 256, 64, 72, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 8, 96, 256, 64, 72, 128, 8, 8, 128, 4);
                TEST(8, 3, true, 8, 96, 256, 64, 72, 128, 8, 8, 128, 5);
                TEST(8, 3, true, 8, 96, 256, 64, 72, 128, 8, 8, 128, 6);
                // cta<8,128,256> warp<64,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 3, true, 8, 128, 256, 64, 96, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 8, 128, 256, 64, 96, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 8, 128, 256, 64, 96, 128, 8, 8, 128, 4);
                TEST(8, 3, true, 8, 128, 256, 64, 96, 128, 8, 8, 128, 5);
                TEST(8, 3, true, 8, 128, 256, 64, 96, 128, 8, 8, 128, 6);
                TEST(8, 3, true, 8, 128, 256, 64, 96, 128, 8, 8, 128, 7);
                // cta<4,32,256> warp<16,96,128> mma<8,8,128>   WARPS[2x1]
                TEST(8, 3, true, 4, 32, 256, 16, 96, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 4, 32, 256, 16, 96, 128, 8, 8, 128, 3);
                // cta<8,32,256> warp<32,96,128> mma<8,8,128>   WARPS[2x1]
                TEST(8, 3, true, 8, 32, 256, 32, 96, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 8, 32, 256, 32, 96, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 8, 32, 256, 32, 96, 128, 8, 8, 128, 4);
                TEST(8, 3, true, 8, 32, 256, 32, 96, 128, 8, 8, 128, 5);
                // cta<4,32,256> warp<16,48,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 3, true, 4, 32, 256, 16, 48, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 4, 32, 256, 16, 48, 128, 8, 8, 128, 3);
                // cta<4,48,256> warp<16,72,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 3, true, 4, 48, 256, 16, 72, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 4, 48, 256, 16, 72, 128, 8, 8, 128, 3);
                // cta<4,64,256> warp<16,96,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 3, true, 4, 64, 256, 16, 96, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 4, 64, 256, 16, 96, 128, 8, 8, 128, 3);
                // cta<4,80,256> warp<16,120,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 3, true, 4, 80, 256, 16, 120, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 4, 80, 256, 16, 120, 128, 8, 8, 128, 3);
                // cta<8,32,256> warp<32,48,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 3, true, 8, 32, 256, 32, 48, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 8, 32, 256, 32, 48, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 8, 32, 256, 32, 48, 128, 8, 8, 128, 4);
                TEST(8, 3, true, 8, 32, 256, 32, 48, 128, 8, 8, 128, 5);
                // cta<8,48,256> warp<32,72,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 3, true, 8, 48, 256, 32, 72, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 8, 48, 256, 32, 72, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 8, 48, 256, 32, 72, 128, 8, 8, 128, 4);
                TEST(8, 3, true, 8, 48, 256, 32, 72, 128, 8, 8, 128, 5);
                // cta<8,64,256> warp<32,96,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 3, true, 8, 64, 256, 32, 96, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 8, 64, 256, 32, 96, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 8, 64, 256, 32, 96, 128, 8, 8, 128, 4);
                TEST(8, 3, true, 8, 64, 256, 32, 96, 128, 8, 8, 128, 5);
                TEST(8, 3, true, 8, 64, 256, 32, 96, 128, 8, 8, 128, 6);
                // cta<8,80,256> warp<32,120,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 3, true, 8, 80, 256, 32, 120, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 8, 80, 256, 32, 120, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 8, 80, 256, 32, 120, 128, 8, 8, 128, 4);
                TEST(8, 3, true, 8, 80, 256, 32, 120, 128, 8, 8, 128, 5);
                TEST(8, 3, true, 8, 80, 256, 32, 120, 128, 8, 8, 128, 6);
                // cta<4,32,256> warp<8,96,128> mma<8,8,128>   WARPS[4x1]
                TEST(8, 3, true, 4, 32, 256, 8, 96, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 4, 32, 256, 8, 96, 128, 8, 8, 128, 3);
                // cta<8,32,256> warp<16,96,128> mma<8,8,128>   WARPS[4x1]
                TEST(8, 3, true, 8, 32, 256, 16, 96, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 8, 32, 256, 16, 96, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 8, 32, 256, 16, 96, 128, 8, 8, 128, 4);
                TEST(8, 3, true, 8, 32, 256, 16, 96, 128, 8, 8, 128, 5);
                // cta<1,32,384> warp<8,48,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 3, true, 1, 32, 384, 8, 48, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 1, 32, 384, 8, 48, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 1, 32, 384, 8, 48, 128, 8, 8, 128, 4);
                // cta<1,48,384> warp<8,72,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 3, true, 1, 48, 384, 8, 72, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 1, 48, 384, 8, 72, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 1, 48, 384, 8, 72, 128, 8, 8, 128, 4);
                // cta<1,64,384> warp<8,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 3, true, 1, 64, 384, 8, 96, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 1, 64, 384, 8, 96, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 1, 64, 384, 8, 96, 128, 8, 8, 128, 4);
                // cta<1,80,384> warp<8,120,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 3, true, 1, 80, 384, 8, 120, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 1, 80, 384, 8, 120, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 1, 80, 384, 8, 120, 128, 8, 8, 128, 4);
                // cta<4,32,384> warp<32,48,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 3, true, 4, 32, 384, 32, 48, 128, 8, 8, 128, 2);
                // cta<4,48,384> warp<32,72,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 3, true, 4, 48, 384, 32, 72, 128, 8, 8, 128, 2);
                // cta<4,64,384> warp<32,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 3, true, 4, 64, 384, 32, 96, 128, 8, 8, 128, 2);
                // cta<4,80,384> warp<32,120,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 3, true, 4, 80, 384, 32, 120, 128, 8, 8, 128, 2);
                // cta<8,32,384> warp<64,48,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 3, true, 8, 32, 384, 64, 48, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 8, 32, 384, 64, 48, 128, 8, 8, 128, 3);
                // cta<8,48,384> warp<64,72,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 3, true, 8, 48, 384, 64, 72, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 8, 48, 384, 64, 72, 128, 8, 8, 128, 3);
                // cta<8,64,384> warp<64,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 3, true, 8, 64, 384, 64, 96, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 8, 64, 384, 64, 96, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 8, 64, 384, 64, 96, 128, 8, 8, 128, 4);
                // cta<1,32,384> warp<8,24,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 3, true, 1, 32, 384, 8, 24, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 1, 32, 384, 8, 24, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 1, 32, 384, 8, 24, 128, 8, 8, 128, 4);
                // cta<1,64,384> warp<8,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 3, true, 1, 64, 384, 8, 48, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 1, 64, 384, 8, 48, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 1, 64, 384, 8, 48, 128, 8, 8, 128, 4);
                // cta<1,96,384> warp<8,72,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 3, true, 1, 96, 384, 8, 72, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 1, 96, 384, 8, 72, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 1, 96, 384, 8, 72, 128, 8, 8, 128, 4);
                // cta<1,128,384> warp<8,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 3, true, 1, 128, 384, 8, 96, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 1, 128, 384, 8, 96, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 1, 128, 384, 8, 96, 128, 8, 8, 128, 4);
                // cta<4,32,384> warp<32,24,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 3, true, 4, 32, 384, 32, 24, 128, 8, 8, 128, 2);
                // cta<4,64,384> warp<32,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 3, true, 4, 64, 384, 32, 48, 128, 8, 8, 128, 2);
                // cta<4,96,384> warp<32,72,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 3, true, 4, 96, 384, 32, 72, 128, 8, 8, 128, 2);
                // cta<4,128,384> warp<32,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 3, true, 4, 128, 384, 32, 96, 128, 8, 8, 128, 2);
                // cta<8,32,384> warp<64,24,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 3, true, 8, 32, 384, 64, 24, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 8, 32, 384, 64, 24, 128, 8, 8, 128, 3);
                // cta<8,64,384> warp<64,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 3, true, 8, 64, 384, 64, 48, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 8, 64, 384, 64, 48, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 8, 64, 384, 64, 48, 128, 8, 8, 128, 4);
                // cta<8,96,384> warp<64,72,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 3, true, 8, 96, 384, 64, 72, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 8, 96, 384, 64, 72, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 8, 96, 384, 64, 72, 128, 8, 8, 128, 4);
                // cta<8,128,384> warp<64,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 3, true, 8, 128, 384, 64, 96, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 8, 128, 384, 64, 96, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 8, 128, 384, 64, 96, 128, 8, 8, 128, 4);
                // cta<4,32,384> warp<16,96,128> mma<8,8,128>   WARPS[2x1]
                TEST(8, 3, true, 4, 32, 384, 16, 96, 128, 8, 8, 128, 2);
                // cta<8,32,384> warp<32,96,128> mma<8,8,128>   WARPS[2x1]
                TEST(8, 3, true, 8, 32, 384, 32, 96, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 8, 32, 384, 32, 96, 128, 8, 8, 128, 3);
                // cta<4,32,384> warp<16,48,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 3, true, 4, 32, 384, 16, 48, 128, 8, 8, 128, 2);
                // cta<4,48,384> warp<16,72,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 3, true, 4, 48, 384, 16, 72, 128, 8, 8, 128, 2);
                // cta<4,64,384> warp<16,96,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 3, true, 4, 64, 384, 16, 96, 128, 8, 8, 128, 2);
                // cta<4,80,384> warp<16,120,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 3, true, 4, 80, 384, 16, 120, 128, 8, 8, 128, 2);
                // cta<8,32,384> warp<32,48,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 3, true, 8, 32, 384, 32, 48, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 8, 32, 384, 32, 48, 128, 8, 8, 128, 3);
                // cta<8,48,384> warp<32,72,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 3, true, 8, 48, 384, 32, 72, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 8, 48, 384, 32, 72, 128, 8, 8, 128, 3);
                // cta<8,64,384> warp<32,96,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 3, true, 8, 64, 384, 32, 96, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 8, 64, 384, 32, 96, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 8, 64, 384, 32, 96, 128, 8, 8, 128, 4);
                // cta<8,80,384> warp<32,120,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 3, true, 8, 80, 384, 32, 120, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 8, 80, 384, 32, 120, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 8, 80, 384, 32, 120, 128, 8, 8, 128, 4);
                // cta<4,32,384> warp<8,96,128> mma<8,8,128>   WARPS[4x1]
                TEST(8, 3, true, 4, 32, 384, 8, 96, 128, 8, 8, 128, 2);
                // cta<8,32,384> warp<16,96,128> mma<8,8,128>   WARPS[4x1]
                TEST(8, 3, true, 8, 32, 384, 16, 96, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 8, 32, 384, 16, 96, 128, 8, 8, 128, 3);
                // cta<1,32,512> warp<8,48,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 3, true, 1, 32, 512, 8, 48, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 1, 32, 512, 8, 48, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 1, 32, 512, 8, 48, 128, 8, 8, 128, 4);
                // cta<1,48,512> warp<8,72,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 3, true, 1, 48, 512, 8, 72, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 1, 48, 512, 8, 72, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 1, 48, 512, 8, 72, 128, 8, 8, 128, 4);
                // cta<1,64,512> warp<8,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 3, true, 1, 64, 512, 8, 96, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 1, 64, 512, 8, 96, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 1, 64, 512, 8, 96, 128, 8, 8, 128, 4);
                // cta<1,80,512> warp<8,120,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 3, true, 1, 80, 512, 8, 120, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 1, 80, 512, 8, 120, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 1, 80, 512, 8, 120, 128, 8, 8, 128, 4);
                // cta<4,32,512> warp<32,48,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 3, true, 4, 32, 512, 32, 48, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 4, 32, 512, 32, 48, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 4, 32, 512, 32, 48, 128, 8, 8, 128, 4);
                // cta<4,48,512> warp<32,72,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 3, true, 4, 48, 512, 32, 72, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 4, 48, 512, 32, 72, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 4, 48, 512, 32, 72, 128, 8, 8, 128, 4);
                // cta<4,64,512> warp<32,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 3, true, 4, 64, 512, 32, 96, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 4, 64, 512, 32, 96, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 4, 64, 512, 32, 96, 128, 8, 8, 128, 4);
                // cta<4,80,512> warp<32,120,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 3, true, 4, 80, 512, 32, 120, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 4, 80, 512, 32, 120, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 4, 80, 512, 32, 120, 128, 8, 8, 128, 4);
                // cta<8,32,512> warp<64,48,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 3, true, 8, 32, 512, 64, 48, 128, 8, 8, 128, 2);
                // cta<8,48,512> warp<64,72,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 3, true, 8, 48, 512, 64, 72, 128, 8, 8, 128, 2);
                // cta<8,64,512> warp<64,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 3, true, 8, 64, 512, 64, 96, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 8, 64, 512, 64, 96, 128, 8, 8, 128, 3);
                // cta<1,32,512> warp<8,24,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 3, true, 1, 32, 512, 8, 24, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 1, 32, 512, 8, 24, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 1, 32, 512, 8, 24, 128, 8, 8, 128, 4);
                // cta<1,64,512> warp<8,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 3, true, 1, 64, 512, 8, 48, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 1, 64, 512, 8, 48, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 1, 64, 512, 8, 48, 128, 8, 8, 128, 4);
                // cta<1,96,512> warp<8,72,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 3, true, 1, 96, 512, 8, 72, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 1, 96, 512, 8, 72, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 1, 96, 512, 8, 72, 128, 8, 8, 128, 4);
                // cta<1,128,512> warp<8,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 3, true, 1, 128, 512, 8, 96, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 1, 128, 512, 8, 96, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 1, 128, 512, 8, 96, 128, 8, 8, 128, 4);
                // cta<4,32,512> warp<32,24,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 3, true, 4, 32, 512, 32, 24, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 4, 32, 512, 32, 24, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 4, 32, 512, 32, 24, 128, 8, 8, 128, 4);
                // cta<4,64,512> warp<32,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 3, true, 4, 64, 512, 32, 48, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 4, 64, 512, 32, 48, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 4, 64, 512, 32, 48, 128, 8, 8, 128, 4);
                // cta<4,96,512> warp<32,72,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 3, true, 4, 96, 512, 32, 72, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 4, 96, 512, 32, 72, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 4, 96, 512, 32, 72, 128, 8, 8, 128, 4);
                // cta<4,128,512> warp<32,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 3, true, 4, 128, 512, 32, 96, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 4, 128, 512, 32, 96, 128, 8, 8, 128, 3);
                // cta<8,32,512> warp<64,24,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 3, true, 8, 32, 512, 64, 24, 128, 8, 8, 128, 2);
                // cta<8,64,512> warp<64,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 3, true, 8, 64, 512, 64, 48, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 8, 64, 512, 64, 48, 128, 8, 8, 128, 3);
                // cta<8,96,512> warp<64,72,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 3, true, 8, 96, 512, 64, 72, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 8, 96, 512, 64, 72, 128, 8, 8, 128, 3);
                // cta<8,128,512> warp<64,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 3, true, 8, 128, 512, 64, 96, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 8, 128, 512, 64, 96, 128, 8, 8, 128, 3);
                // cta<4,32,512> warp<16,96,128> mma<8,8,128>   WARPS[2x1]
                TEST(8, 3, true, 4, 32, 512, 16, 96, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 4, 32, 512, 16, 96, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 4, 32, 512, 16, 96, 128, 8, 8, 128, 4);
                // cta<8,32,512> warp<32,96,128> mma<8,8,128>   WARPS[2x1]
                TEST(8, 3, true, 8, 32, 512, 32, 96, 128, 8, 8, 128, 2);
                // cta<4,32,512> warp<16,48,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 3, true, 4, 32, 512, 16, 48, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 4, 32, 512, 16, 48, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 4, 32, 512, 16, 48, 128, 8, 8, 128, 4);
                // cta<4,48,512> warp<16,72,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 3, true, 4, 48, 512, 16, 72, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 4, 48, 512, 16, 72, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 4, 48, 512, 16, 72, 128, 8, 8, 128, 4);
                // cta<4,64,512> warp<16,96,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 3, true, 4, 64, 512, 16, 96, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 4, 64, 512, 16, 96, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 4, 64, 512, 16, 96, 128, 8, 8, 128, 4);
                // cta<4,80,512> warp<16,120,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 3, true, 4, 80, 512, 16, 120, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 4, 80, 512, 16, 120, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 4, 80, 512, 16, 120, 128, 8, 8, 128, 4);
                // cta<8,32,512> warp<32,48,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 3, true, 8, 32, 512, 32, 48, 128, 8, 8, 128, 2);
                // cta<8,48,512> warp<32,72,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 3, true, 8, 48, 512, 32, 72, 128, 8, 8, 128, 2);
                // cta<8,64,512> warp<32,96,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 3, true, 8, 64, 512, 32, 96, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 8, 64, 512, 32, 96, 128, 8, 8, 128, 3);
                // cta<8,80,512> warp<32,120,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 3, true, 8, 80, 512, 32, 120, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 8, 80, 512, 32, 120, 128, 8, 8, 128, 3);
                // cta<4,32,512> warp<8,96,128> mma<8,8,128>   WARPS[4x1]
                TEST(8, 3, true, 4, 32, 512, 8, 96, 128, 8, 8, 128, 2);
                TEST(8, 3, true, 4, 32, 512, 8, 96, 128, 8, 8, 128, 3);
                TEST(8, 3, true, 4, 32, 512, 8, 96, 128, 8, 8, 128, 4);
                // cta<8,32,512> warp<16,96,128> mma<8,8,128>   WARPS[4x1]
                TEST(8, 3, true, 8, 32, 512, 16, 96, 128, 8, 8, 128, 2);
            }
            break;
        #endif
        #ifdef W4A8
        case 4:
            if (quant_sign) {
                ////// W4A8 int
                // cta<1,32,256> warp<8,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 4, true, 1, 32, 256, 8, 64, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 1, 32, 256, 8, 64, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 1, 32, 256, 8, 64, 128, 8, 8, 128, 4);
                // cta<1,48,256> warp<8,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 4, true, 1, 48, 256, 8, 96, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 1, 48, 256, 8, 96, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 1, 48, 256, 8, 96, 128, 8, 8, 128, 4);
                // cta<1,64,256> warp<8,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 4, true, 1, 64, 256, 8, 128, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 1, 64, 256, 8, 128, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 1, 64, 256, 8, 128, 128, 8, 8, 128, 4);
                // cta<4,32,256> warp<32,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 4, true, 4, 32, 256, 32, 64, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 4, 32, 256, 32, 64, 128, 8, 8, 128, 3);
                // cta<4,48,256> warp<32,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 4, true, 4, 48, 256, 32, 96, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 4, 48, 256, 32, 96, 128, 8, 8, 128, 3);
                // cta<4,64,256> warp<32,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 4, true, 4, 64, 256, 32, 128, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 4, 64, 256, 32, 128, 128, 8, 8, 128, 3);
                // cta<8,32,256> warp<64,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 4, true, 8, 32, 256, 64, 64, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 8, 32, 256, 64, 64, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 8, 32, 256, 64, 64, 128, 8, 8, 128, 4);
                TEST(8, 4, true, 8, 32, 256, 64, 64, 128, 8, 8, 128, 5);
                // cta<8,48,256> warp<64,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 4, true, 8, 48, 256, 64, 96, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 8, 48, 256, 64, 96, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 8, 48, 256, 64, 96, 128, 8, 8, 128, 4);
                TEST(8, 4, true, 8, 48, 256, 64, 96, 128, 8, 8, 128, 5);
                TEST(8, 4, true, 8, 48, 256, 64, 96, 128, 8, 8, 128, 6);
                // cta<1,32,256> warp<8,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 1, 32, 256, 8, 32, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 1, 32, 256, 8, 32, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 1, 32, 256, 8, 32, 128, 8, 8, 128, 4);
                // cta<1,48,256> warp<8,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 1, 48, 256, 8, 48, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 1, 48, 256, 8, 48, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 1, 48, 256, 8, 48, 128, 8, 8, 128, 4);
                // cta<1,64,256> warp<8,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 1, 64, 256, 8, 64, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 1, 64, 256, 8, 64, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 1, 64, 256, 8, 64, 128, 8, 8, 128, 4);
                // cta<1,80,256> warp<8,80,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 1, 80, 256, 8, 80, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 1, 80, 256, 8, 80, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 1, 80, 256, 8, 80, 128, 8, 8, 128, 4);
                // cta<1,96,256> warp<8,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 1, 96, 256, 8, 96, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 1, 96, 256, 8, 96, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 1, 96, 256, 8, 96, 128, 8, 8, 128, 4);
                // cta<1,112,256> warp<8,112,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 1, 112, 256, 8, 112, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 1, 112, 256, 8, 112, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 1, 112, 256, 8, 112, 128, 8, 8, 128, 4);
                // cta<1,128,256> warp<8,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 1, 128, 256, 8, 128, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 1, 128, 256, 8, 128, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 1, 128, 256, 8, 128, 128, 8, 8, 128, 4);
                // cta<4,32,256> warp<32,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 4, 32, 256, 32, 32, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 4, 32, 256, 32, 32, 128, 8, 8, 128, 3);
                // cta<4,48,256> warp<32,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 4, 48, 256, 32, 48, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 4, 48, 256, 32, 48, 128, 8, 8, 128, 3);
                // cta<4,64,256> warp<32,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 4, 64, 256, 32, 64, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 4, 64, 256, 32, 64, 128, 8, 8, 128, 3);
                // cta<4,80,256> warp<32,80,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 4, 80, 256, 32, 80, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 4, 80, 256, 32, 80, 128, 8, 8, 128, 3);
                // cta<4,96,256> warp<32,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 4, 96, 256, 32, 96, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 4, 96, 256, 32, 96, 128, 8, 8, 128, 3);
                // cta<4,112,256> warp<32,112,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 4, 112, 256, 32, 112, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 4, 112, 256, 32, 112, 128, 8, 8, 128, 3);
                // cta<4,128,256> warp<32,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 4, 128, 256, 32, 128, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 4, 128, 256, 32, 128, 128, 8, 8, 128, 3);
                // cta<8,32,256> warp<64,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 8, 32, 256, 64, 32, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 8, 32, 256, 64, 32, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 8, 32, 256, 64, 32, 128, 8, 8, 128, 4);
                TEST(8, 4, true, 8, 32, 256, 64, 32, 128, 8, 8, 128, 5);
                // cta<8,48,256> warp<64,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 8, 48, 256, 64, 48, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 8, 48, 256, 64, 48, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 8, 48, 256, 64, 48, 128, 8, 8, 128, 4);
                TEST(8, 4, true, 8, 48, 256, 64, 48, 128, 8, 8, 128, 5);
                TEST(8, 4, true, 8, 48, 256, 64, 48, 128, 8, 8, 128, 6);
                // cta<8,64,256> warp<64,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 8, 64, 256, 64, 64, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 8, 64, 256, 64, 64, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 8, 64, 256, 64, 64, 128, 8, 8, 128, 4);
                TEST(8, 4, true, 8, 64, 256, 64, 64, 128, 8, 8, 128, 5);
                TEST(8, 4, true, 8, 64, 256, 64, 64, 128, 8, 8, 128, 6);
                // cta<8,80,256> warp<64,80,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 8, 80, 256, 64, 80, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 8, 80, 256, 64, 80, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 8, 80, 256, 64, 80, 128, 8, 8, 128, 4);
                TEST(8, 4, true, 8, 80, 256, 64, 80, 128, 8, 8, 128, 5);
                TEST(8, 4, true, 8, 80, 256, 64, 80, 128, 8, 8, 128, 6);
                // cta<8,96,256> warp<64,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 8, 96, 256, 64, 96, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 8, 96, 256, 64, 96, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 8, 96, 256, 64, 96, 128, 8, 8, 128, 4);
                TEST(8, 4, true, 8, 96, 256, 64, 96, 128, 8, 8, 128, 5);
                TEST(8, 4, true, 8, 96, 256, 64, 96, 128, 8, 8, 128, 6);
                TEST(8, 4, true, 8, 96, 256, 64, 96, 128, 8, 8, 128, 7);
                // cta<4,32,256> warp<16,128,128> mma<8,8,128>   WARPS[2x1]
                TEST(8, 4, true, 4, 32, 256, 16, 128, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 4, 32, 256, 16, 128, 128, 8, 8, 128, 3);
                // cta<8,32,256> warp<32,128,128> mma<8,8,128>   WARPS[2x1]
                TEST(8, 4, true, 8, 32, 256, 32, 128, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 8, 32, 256, 32, 128, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 8, 32, 256, 32, 128, 128, 8, 8, 128, 4);
                TEST(8, 4, true, 8, 32, 256, 32, 128, 128, 8, 8, 128, 5);
                // cta<4,32,256> warp<16,64,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 4, true, 4, 32, 256, 16, 64, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 4, 32, 256, 16, 64, 128, 8, 8, 128, 3);
                // cta<4,48,256> warp<16,96,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 4, true, 4, 48, 256, 16, 96, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 4, 48, 256, 16, 96, 128, 8, 8, 128, 3);
                // cta<4,64,256> warp<16,128,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 4, true, 4, 64, 256, 16, 128, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 4, 64, 256, 16, 128, 128, 8, 8, 128, 3);
                // cta<8,32,256> warp<32,64,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 4, true, 8, 32, 256, 32, 64, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 8, 32, 256, 32, 64, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 8, 32, 256, 32, 64, 128, 8, 8, 128, 4);
                TEST(8, 4, true, 8, 32, 256, 32, 64, 128, 8, 8, 128, 5);
                // cta<8,48,256> warp<32,96,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 4, true, 8, 48, 256, 32, 96, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 8, 48, 256, 32, 96, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 8, 48, 256, 32, 96, 128, 8, 8, 128, 4);
                TEST(8, 4, true, 8, 48, 256, 32, 96, 128, 8, 8, 128, 5);
                TEST(8, 4, true, 8, 48, 256, 32, 96, 128, 8, 8, 128, 6);
                // cta<8,64,256> warp<32,128,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 4, true, 8, 64, 256, 32, 128, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 8, 64, 256, 32, 128, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 8, 64, 256, 32, 128, 128, 8, 8, 128, 4);
                TEST(8, 4, true, 8, 64, 256, 32, 128, 128, 8, 8, 128, 5);
                TEST(8, 4, true, 8, 64, 256, 32, 128, 128, 8, 8, 128, 6);
                // cta<4,32,256> warp<8,128,128> mma<8,8,128>   WARPS[4x1]
                TEST(8, 4, true, 4, 32, 256, 8, 128, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 4, 32, 256, 8, 128, 128, 8, 8, 128, 3);
                // cta<8,32,256> warp<16,128,128> mma<8,8,128>   WARPS[4x1]
                TEST(8, 4, true, 8, 32, 256, 16, 128, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 8, 32, 256, 16, 128, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 8, 32, 256, 16, 128, 128, 8, 8, 128, 4);
                TEST(8, 4, true, 8, 32, 256, 16, 128, 128, 8, 8, 128, 5);
                // cta<1,32,384> warp<8,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 4, true, 1, 32, 384, 8, 64, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 1, 32, 384, 8, 64, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 1, 32, 384, 8, 64, 128, 8, 8, 128, 4);
                // cta<1,48,384> warp<8,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 4, true, 1, 48, 384, 8, 96, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 1, 48, 384, 8, 96, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 1, 48, 384, 8, 96, 128, 8, 8, 128, 4);
                // cta<1,64,384> warp<8,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 4, true, 1, 64, 384, 8, 128, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 1, 64, 384, 8, 128, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 1, 64, 384, 8, 128, 128, 8, 8, 128, 4);
                // cta<4,32,384> warp<32,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 4, true, 4, 32, 384, 32, 64, 128, 8, 8, 128, 2);
                // cta<4,48,384> warp<32,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 4, true, 4, 48, 384, 32, 96, 128, 8, 8, 128, 2);
                // cta<4,64,384> warp<32,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 4, true, 4, 64, 384, 32, 128, 128, 8, 8, 128, 2);
                // cta<8,32,384> warp<64,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 4, true, 8, 32, 384, 64, 64, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 8, 32, 384, 64, 64, 128, 8, 8, 128, 3);
                // cta<8,48,384> warp<64,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 4, true, 8, 48, 384, 64, 96, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 8, 48, 384, 64, 96, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 8, 48, 384, 64, 96, 128, 8, 8, 128, 4);
                // cta<1,32,384> warp<8,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 1, 32, 384, 8, 32, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 1, 32, 384, 8, 32, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 1, 32, 384, 8, 32, 128, 8, 8, 128, 4);
                // cta<1,48,384> warp<8,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 1, 48, 384, 8, 48, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 1, 48, 384, 8, 48, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 1, 48, 384, 8, 48, 128, 8, 8, 128, 4);
                // cta<1,64,384> warp<8,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 1, 64, 384, 8, 64, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 1, 64, 384, 8, 64, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 1, 64, 384, 8, 64, 128, 8, 8, 128, 4);
                // cta<1,80,384> warp<8,80,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 1, 80, 384, 8, 80, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 1, 80, 384, 8, 80, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 1, 80, 384, 8, 80, 128, 8, 8, 128, 4);
                // cta<1,96,384> warp<8,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 1, 96, 384, 8, 96, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 1, 96, 384, 8, 96, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 1, 96, 384, 8, 96, 128, 8, 8, 128, 4);
                // cta<1,112,384> warp<8,112,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 1, 112, 384, 8, 112, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 1, 112, 384, 8, 112, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 1, 112, 384, 8, 112, 128, 8, 8, 128, 4);
                // cta<1,128,384> warp<8,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 1, 128, 384, 8, 128, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 1, 128, 384, 8, 128, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 1, 128, 384, 8, 128, 128, 8, 8, 128, 4);
                // cta<4,32,384> warp<32,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 4, 32, 384, 32, 32, 128, 8, 8, 128, 2);
                // cta<4,48,384> warp<32,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 4, 48, 384, 32, 48, 128, 8, 8, 128, 2);
                // cta<4,64,384> warp<32,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 4, 64, 384, 32, 64, 128, 8, 8, 128, 2);
                // cta<4,80,384> warp<32,80,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 4, 80, 384, 32, 80, 128, 8, 8, 128, 2);
                // cta<4,96,384> warp<32,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 4, 96, 384, 32, 96, 128, 8, 8, 128, 2);
                // cta<4,112,384> warp<32,112,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 4, 112, 384, 32, 112, 128, 8, 8, 128, 2);
                // cta<4,128,384> warp<32,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 4, 128, 384, 32, 128, 128, 8, 8, 128, 2);
                // cta<8,32,384> warp<64,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 8, 32, 384, 64, 32, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 8, 32, 384, 64, 32, 128, 8, 8, 128, 3);
                // cta<8,48,384> warp<64,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 8, 48, 384, 64, 48, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 8, 48, 384, 64, 48, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 8, 48, 384, 64, 48, 128, 8, 8, 128, 4);
                // cta<8,64,384> warp<64,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 8, 64, 384, 64, 64, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 8, 64, 384, 64, 64, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 8, 64, 384, 64, 64, 128, 8, 8, 128, 4);
                // cta<8,80,384> warp<64,80,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 8, 80, 384, 64, 80, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 8, 80, 384, 64, 80, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 8, 80, 384, 64, 80, 128, 8, 8, 128, 4);
                // cta<8,96,384> warp<64,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 8, 96, 384, 64, 96, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 8, 96, 384, 64, 96, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 8, 96, 384, 64, 96, 128, 8, 8, 128, 4);
                // cta<4,32,384> warp<16,128,128> mma<8,8,128>   WARPS[2x1]
                TEST(8, 4, true, 4, 32, 384, 16, 128, 128, 8, 8, 128, 2);
                // cta<8,32,384> warp<32,128,128> mma<8,8,128>   WARPS[2x1]
                TEST(8, 4, true, 8, 32, 384, 32, 128, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 8, 32, 384, 32, 128, 128, 8, 8, 128, 3);
                // cta<4,32,384> warp<16,64,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 4, true, 4, 32, 384, 16, 64, 128, 8, 8, 128, 2);
                // cta<4,48,384> warp<16,96,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 4, true, 4, 48, 384, 16, 96, 128, 8, 8, 128, 2);
                // cta<4,64,384> warp<16,128,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 4, true, 4, 64, 384, 16, 128, 128, 8, 8, 128, 2);
                // cta<8,32,384> warp<32,64,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 4, true, 8, 32, 384, 32, 64, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 8, 32, 384, 32, 64, 128, 8, 8, 128, 3);
                // cta<8,48,384> warp<32,96,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 4, true, 8, 48, 384, 32, 96, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 8, 48, 384, 32, 96, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 8, 48, 384, 32, 96, 128, 8, 8, 128, 4);
                // cta<8,64,384> warp<32,128,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 4, true, 8, 64, 384, 32, 128, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 8, 64, 384, 32, 128, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 8, 64, 384, 32, 128, 128, 8, 8, 128, 4);
                // cta<4,32,384> warp<8,128,128> mma<8,8,128>   WARPS[4x1]
                TEST(8, 4, true, 4, 32, 384, 8, 128, 128, 8, 8, 128, 2);
                // cta<8,32,384> warp<16,128,128> mma<8,8,128>   WARPS[4x1]
                TEST(8, 4, true, 8, 32, 384, 16, 128, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 8, 32, 384, 16, 128, 128, 8, 8, 128, 3);
                // cta<1,32,512> warp<8,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 4, true, 1, 32, 512, 8, 64, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 1, 32, 512, 8, 64, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 1, 32, 512, 8, 64, 128, 8, 8, 128, 4);
                // cta<1,48,512> warp<8,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 4, true, 1, 48, 512, 8, 96, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 1, 48, 512, 8, 96, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 1, 48, 512, 8, 96, 128, 8, 8, 128, 4);
                // cta<1,64,512> warp<8,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 4, true, 1, 64, 512, 8, 128, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 1, 64, 512, 8, 128, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 1, 64, 512, 8, 128, 128, 8, 8, 128, 4);
                // cta<4,32,512> warp<32,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 4, true, 4, 32, 512, 32, 64, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 4, 32, 512, 32, 64, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 4, 32, 512, 32, 64, 128, 8, 8, 128, 4);
                // cta<4,48,512> warp<32,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 4, true, 4, 48, 512, 32, 96, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 4, 48, 512, 32, 96, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 4, 48, 512, 32, 96, 128, 8, 8, 128, 4);
                // cta<4,64,512> warp<32,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 4, true, 4, 64, 512, 32, 128, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 4, 64, 512, 32, 128, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 4, 64, 512, 32, 128, 128, 8, 8, 128, 4);
                // cta<8,32,512> warp<64,64,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 4, true, 8, 32, 512, 64, 64, 128, 8, 8, 128, 2);
                // cta<8,48,512> warp<64,96,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 4, true, 8, 48, 512, 64, 96, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 8, 48, 512, 64, 96, 128, 8, 8, 128, 3);
                // cta<1,32,512> warp<8,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 1, 32, 512, 8, 32, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 1, 32, 512, 8, 32, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 1, 32, 512, 8, 32, 128, 8, 8, 128, 4);
                // cta<1,48,512> warp<8,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 1, 48, 512, 8, 48, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 1, 48, 512, 8, 48, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 1, 48, 512, 8, 48, 128, 8, 8, 128, 4);
                // cta<1,64,512> warp<8,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 1, 64, 512, 8, 64, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 1, 64, 512, 8, 64, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 1, 64, 512, 8, 64, 128, 8, 8, 128, 4);
                // cta<1,80,512> warp<8,80,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 1, 80, 512, 8, 80, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 1, 80, 512, 8, 80, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 1, 80, 512, 8, 80, 128, 8, 8, 128, 4);
                // cta<1,96,512> warp<8,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 1, 96, 512, 8, 96, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 1, 96, 512, 8, 96, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 1, 96, 512, 8, 96, 128, 8, 8, 128, 4);
                // cta<1,112,512> warp<8,112,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 1, 112, 512, 8, 112, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 1, 112, 512, 8, 112, 128, 8, 8, 128, 3);
                // cta<1,128,512> warp<8,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 1, 128, 512, 8, 128, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 1, 128, 512, 8, 128, 128, 8, 8, 128, 3);
                // cta<4,32,512> warp<32,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 4, 32, 512, 32, 32, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 4, 32, 512, 32, 32, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 4, 32, 512, 32, 32, 128, 8, 8, 128, 4);
                // cta<4,48,512> warp<32,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 4, 48, 512, 32, 48, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 4, 48, 512, 32, 48, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 4, 48, 512, 32, 48, 128, 8, 8, 128, 4);
                // cta<4,64,512> warp<32,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 4, 64, 512, 32, 64, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 4, 64, 512, 32, 64, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 4, 64, 512, 32, 64, 128, 8, 8, 128, 4);
                // cta<4,80,512> warp<32,80,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 4, 80, 512, 32, 80, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 4, 80, 512, 32, 80, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 4, 80, 512, 32, 80, 128, 8, 8, 128, 4);
                // cta<4,96,512> warp<32,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 4, 96, 512, 32, 96, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 4, 96, 512, 32, 96, 128, 8, 8, 128, 3);
                // cta<4,112,512> warp<32,112,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 4, 112, 512, 32, 112, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 4, 112, 512, 32, 112, 128, 8, 8, 128, 3);
                // cta<4,128,512> warp<32,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 4, 128, 512, 32, 128, 128, 8, 8, 128, 2);
                // cta<8,32,512> warp<64,32,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 8, 32, 512, 64, 32, 128, 8, 8, 128, 2);
                // cta<8,48,512> warp<64,48,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 8, 48, 512, 64, 48, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 8, 48, 512, 64, 48, 128, 8, 8, 128, 3);
                // cta<8,64,512> warp<64,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 8, 64, 512, 64, 64, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 8, 64, 512, 64, 64, 128, 8, 8, 128, 3);
                // cta<8,80,512> warp<64,80,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 8, 80, 512, 64, 80, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 8, 80, 512, 64, 80, 128, 8, 8, 128, 3);
                // cta<8,96,512> warp<64,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 4, true, 8, 96, 512, 64, 96, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 8, 96, 512, 64, 96, 128, 8, 8, 128, 3);
                // cta<4,32,512> warp<16,128,128> mma<8,8,128>   WARPS[2x1]
                TEST(8, 4, true, 4, 32, 512, 16, 128, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 4, 32, 512, 16, 128, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 4, 32, 512, 16, 128, 128, 8, 8, 128, 4);
                // cta<8,32,512> warp<32,128,128> mma<8,8,128>   WARPS[2x1]
                TEST(8, 4, true, 8, 32, 512, 32, 128, 128, 8, 8, 128, 2);
                // cta<4,32,512> warp<16,64,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 4, true, 4, 32, 512, 16, 64, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 4, 32, 512, 16, 64, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 4, 32, 512, 16, 64, 128, 8, 8, 128, 4);
                // cta<4,48,512> warp<16,96,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 4, true, 4, 48, 512, 16, 96, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 4, 48, 512, 16, 96, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 4, 48, 512, 16, 96, 128, 8, 8, 128, 4);
                // cta<4,64,512> warp<16,128,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 4, true, 4, 64, 512, 16, 128, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 4, 64, 512, 16, 128, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 4, 64, 512, 16, 128, 128, 8, 8, 128, 4);
                // cta<8,32,512> warp<32,64,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 4, true, 8, 32, 512, 32, 64, 128, 8, 8, 128, 2);
                // cta<8,48,512> warp<32,96,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 4, true, 8, 48, 512, 32, 96, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 8, 48, 512, 32, 96, 128, 8, 8, 128, 3);
                // cta<8,64,512> warp<32,128,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 4, true, 8, 64, 512, 32, 128, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 8, 64, 512, 32, 128, 128, 8, 8, 128, 3);
                // cta<4,32,512> warp<8,128,128> mma<8,8,128>   WARPS[4x1]
                TEST(8, 4, true, 4, 32, 512, 8, 128, 128, 8, 8, 128, 2);
                TEST(8, 4, true, 4, 32, 512, 8, 128, 128, 8, 8, 128, 3);
                TEST(8, 4, true, 4, 32, 512, 8, 128, 128, 8, 8, 128, 4);
                // cta<8,32,512> warp<16,128,128> mma<8,8,128>   WARPS[4x1]
                TEST(8, 4, true, 8, 32, 512, 16, 128, 128, 8, 8, 128, 2);
            } else {
            }
            break;
        #endif
        #ifdef W8A8
        case 8:
            if (quant_sign) {
                ////// W8A8 int
                // cta<1,32,256> warp<8,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 8, true, 1, 32, 256, 8, 128, 128, 8, 8, 128, 2);
                TEST(8, 8, true, 1, 32, 256, 8, 128, 128, 8, 8, 128, 3);
                TEST(8, 8, true, 1, 32, 256, 8, 128, 128, 8, 8, 128, 4);
                // cta<4,32,256> warp<32,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 8, true, 4, 32, 256, 32, 128, 128, 8, 8, 128, 2);
                TEST(8, 8, true, 4, 32, 256, 32, 128, 128, 8, 8, 128, 3);
                // cta<1,32,256> warp<8,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 8, true, 1, 32, 256, 8, 64, 128, 8, 8, 128, 2);
                TEST(8, 8, true, 1, 32, 256, 8, 64, 128, 8, 8, 128, 3);
                TEST(8, 8, true, 1, 32, 256, 8, 64, 128, 8, 8, 128, 4);
                // cta<1,48,256> warp<8,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 8, true, 1, 48, 256, 8, 96, 128, 8, 8, 128, 2);
                TEST(8, 8, true, 1, 48, 256, 8, 96, 128, 8, 8, 128, 3);
                TEST(8, 8, true, 1, 48, 256, 8, 96, 128, 8, 8, 128, 4);
                // cta<1,64,256> warp<8,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 8, true, 1, 64, 256, 8, 128, 128, 8, 8, 128, 2);
                TEST(8, 8, true, 1, 64, 256, 8, 128, 128, 8, 8, 128, 3);
                TEST(8, 8, true, 1, 64, 256, 8, 128, 128, 8, 8, 128, 4);
                // cta<4,32,256> warp<32,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 8, true, 4, 32, 256, 32, 64, 128, 8, 8, 128, 2);
                TEST(8, 8, true, 4, 32, 256, 32, 64, 128, 8, 8, 128, 3);
                // cta<4,48,256> warp<32,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 8, true, 4, 48, 256, 32, 96, 128, 8, 8, 128, 2);
                TEST(8, 8, true, 4, 48, 256, 32, 96, 128, 8, 8, 128, 3);
                // cta<4,64,256> warp<32,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 8, true, 4, 64, 256, 32, 128, 128, 8, 8, 128, 2);
                TEST(8, 8, true, 4, 64, 256, 32, 128, 128, 8, 8, 128, 3);
                // cta<8,32,256> warp<64,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 8, true, 8, 32, 256, 64, 64, 128, 8, 8, 128, 2);
                TEST(8, 8, true, 8, 32, 256, 64, 64, 128, 8, 8, 128, 3);
                TEST(8, 8, true, 8, 32, 256, 64, 64, 128, 8, 8, 128, 4);
                TEST(8, 8, true, 8, 32, 256, 64, 64, 128, 8, 8, 128, 5);
                TEST(8, 8, true, 8, 32, 256, 64, 64, 128, 8, 8, 128, 6);
                // cta<8,48,256> warp<64,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 8, true, 8, 48, 256, 64, 96, 128, 8, 8, 128, 2);
                TEST(8, 8, true, 8, 48, 256, 64, 96, 128, 8, 8, 128, 3);
                TEST(8, 8, true, 8, 48, 256, 64, 96, 128, 8, 8, 128, 4);
                TEST(8, 8, true, 8, 48, 256, 64, 96, 128, 8, 8, 128, 5);
                TEST(8, 8, true, 8, 48, 256, 64, 96, 128, 8, 8, 128, 6);
                TEST(8, 8, true, 8, 48, 256, 64, 96, 128, 8, 8, 128, 7);
                // cta<4,32,256> warp<16,128,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 8, true, 4, 32, 256, 16, 128, 128, 8, 8, 128, 2);
                TEST(8, 8, true, 4, 32, 256, 16, 128, 128, 8, 8, 128, 3);
                // cta<8,32,256> warp<32,128,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 8, true, 8, 32, 256, 32, 128, 128, 8, 8, 128, 2);
                TEST(8, 8, true, 8, 32, 256, 32, 128, 128, 8, 8, 128, 3);
                TEST(8, 8, true, 8, 32, 256, 32, 128, 128, 8, 8, 128, 4);
                TEST(8, 8, true, 8, 32, 256, 32, 128, 128, 8, 8, 128, 5);
                TEST(8, 8, true, 8, 32, 256, 32, 128, 128, 8, 8, 128, 6);
                // cta<1,32,384> warp<8,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 8, true, 1, 32, 384, 8, 128, 128, 8, 8, 128, 2);
                TEST(8, 8, true, 1, 32, 384, 8, 128, 128, 8, 8, 128, 3);
                TEST(8, 8, true, 1, 32, 384, 8, 128, 128, 8, 8, 128, 4);
                // cta<4,32,384> warp<32,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 8, true, 4, 32, 384, 32, 128, 128, 8, 8, 128, 2);
                // cta<1,32,384> warp<8,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 8, true, 1, 32, 384, 8, 64, 128, 8, 8, 128, 2);
                TEST(8, 8, true, 1, 32, 384, 8, 64, 128, 8, 8, 128, 3);
                TEST(8, 8, true, 1, 32, 384, 8, 64, 128, 8, 8, 128, 4);
                // cta<1,48,384> warp<8,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 8, true, 1, 48, 384, 8, 96, 128, 8, 8, 128, 2);
                TEST(8, 8, true, 1, 48, 384, 8, 96, 128, 8, 8, 128, 3);
                TEST(8, 8, true, 1, 48, 384, 8, 96, 128, 8, 8, 128, 4);
                // cta<1,64,384> warp<8,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 8, true, 1, 64, 384, 8, 128, 128, 8, 8, 128, 2);
                TEST(8, 8, true, 1, 64, 384, 8, 128, 128, 8, 8, 128, 3);
                TEST(8, 8, true, 1, 64, 384, 8, 128, 128, 8, 8, 128, 4);
                // cta<4,32,384> warp<32,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 8, true, 4, 32, 384, 32, 64, 128, 8, 8, 128, 2);
                // cta<4,48,384> warp<32,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 8, true, 4, 48, 384, 32, 96, 128, 8, 8, 128, 2);
                // cta<4,64,384> warp<32,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 8, true, 4, 64, 384, 32, 128, 128, 8, 8, 128, 2);
                // cta<8,32,384> warp<64,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 8, true, 8, 32, 384, 64, 64, 128, 8, 8, 128, 2);
                TEST(8, 8, true, 8, 32, 384, 64, 64, 128, 8, 8, 128, 3);
                TEST(8, 8, true, 8, 32, 384, 64, 64, 128, 8, 8, 128, 4);
                // cta<8,48,384> warp<64,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 8, true, 8, 48, 384, 64, 96, 128, 8, 8, 128, 2);
                TEST(8, 8, true, 8, 48, 384, 64, 96, 128, 8, 8, 128, 3);
                TEST(8, 8, true, 8, 48, 384, 64, 96, 128, 8, 8, 128, 4);
                // cta<4,32,384> warp<16,128,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 8, true, 4, 32, 384, 16, 128, 128, 8, 8, 128, 2);
                // cta<8,32,384> warp<32,128,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 8, true, 8, 32, 384, 32, 128, 128, 8, 8, 128, 2);
                TEST(8, 8, true, 8, 32, 384, 32, 128, 128, 8, 8, 128, 3);
                TEST(8, 8, true, 8, 32, 384, 32, 128, 128, 8, 8, 128, 4);
                // cta<1,32,512> warp<8,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 8, true, 1, 32, 512, 8, 128, 128, 8, 8, 128, 2);
                TEST(8, 8, true, 1, 32, 512, 8, 128, 128, 8, 8, 128, 3);
                TEST(8, 8, true, 1, 32, 512, 8, 128, 128, 8, 8, 128, 4);
                // cta<4,32,512> warp<32,128,128> mma<8,8,128>   WARPS[1x2]
                TEST(8, 8, true, 4, 32, 512, 32, 128, 128, 8, 8, 128, 2);
                TEST(8, 8, true, 4, 32, 512, 32, 128, 128, 8, 8, 128, 3);
                TEST(8, 8, true, 4, 32, 512, 32, 128, 128, 8, 8, 128, 4);
                // cta<1,32,512> warp<8,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 8, true, 1, 32, 512, 8, 64, 128, 8, 8, 128, 2);
                TEST(8, 8, true, 1, 32, 512, 8, 64, 128, 8, 8, 128, 3);
                TEST(8, 8, true, 1, 32, 512, 8, 64, 128, 8, 8, 128, 4);
                // cta<1,48,512> warp<8,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 8, true, 1, 48, 512, 8, 96, 128, 8, 8, 128, 2);
                TEST(8, 8, true, 1, 48, 512, 8, 96, 128, 8, 8, 128, 3);
                TEST(8, 8, true, 1, 48, 512, 8, 96, 128, 8, 8, 128, 4);
                // cta<1,64,512> warp<8,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 8, true, 1, 64, 512, 8, 128, 128, 8, 8, 128, 2);
                TEST(8, 8, true, 1, 64, 512, 8, 128, 128, 8, 8, 128, 3);
                // cta<4,32,512> warp<32,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 8, true, 4, 32, 512, 32, 64, 128, 8, 8, 128, 2);
                TEST(8, 8, true, 4, 32, 512, 32, 64, 128, 8, 8, 128, 3);
                TEST(8, 8, true, 4, 32, 512, 32, 64, 128, 8, 8, 128, 4);
                // cta<4,48,512> warp<32,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 8, true, 4, 48, 512, 32, 96, 128, 8, 8, 128, 2);
                TEST(8, 8, true, 4, 48, 512, 32, 96, 128, 8, 8, 128, 3);
                // cta<4,64,512> warp<32,128,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 8, true, 4, 64, 512, 32, 128, 128, 8, 8, 128, 2);
                // cta<8,32,512> warp<64,64,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 8, true, 8, 32, 512, 64, 64, 128, 8, 8, 128, 2);
                TEST(8, 8, true, 8, 32, 512, 64, 64, 128, 8, 8, 128, 3);
                // cta<8,48,512> warp<64,96,128> mma<8,8,128>   WARPS[1x4]
                TEST(8, 8, true, 8, 48, 512, 64, 96, 128, 8, 8, 128, 2);
                TEST(8, 8, true, 8, 48, 512, 64, 96, 128, 8, 8, 128, 3);
                // cta<4,32,512> warp<16,128,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 8, true, 4, 32, 512, 16, 128, 128, 8, 8, 128, 2);
                TEST(8, 8, true, 4, 32, 512, 16, 128, 128, 8, 8, 128, 3);
                TEST(8, 8, true, 4, 32, 512, 16, 128, 128, 8, 8, 128, 4);
                // cta<8,32,512> warp<32,128,128> mma<8,8,128>   WARPS[2x2]
                TEST(8, 8, true, 8, 32, 512, 32, 128, 128, 8, 8, 128, 2);
                TEST(8, 8, true, 8, 32, 512, 32, 128, 128, 8, 8, 128, 3);
            } else {
            }
            break;
        #endif
        default:
            printf("unsupport w%da%d\n", w_bits, x_bits);
        }
        break;
    default:
        printf("unsupport w%da%d\n", w_bits, x_bits);
    }
    printf("The best kernel config is %s with %f TOPS\n", best_config.str().c_str(), max_gflop);
    free(h_x);
    free(h_w);
    free(h_x_pack);
    free(h_w_pack);
    free(h_out);
    free(h_ref_out);
    hipFree(d_x);
    hipFree(d_w);
    hipFree(d_x_pack);
    hipFree(d_w_pack);
    hipFree(d_out);

    hipStreamDestroy(stream);
    return 0;
}