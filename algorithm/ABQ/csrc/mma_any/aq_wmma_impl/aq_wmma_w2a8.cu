// Copyright (C) ABQ.2024 (liusongwei.zju@bytedance.com)
// 
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
// 
//          http://www.apache.org/licenses/LICENSE-2.0
// 
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "common/base.h"
#include "mma_any/aq_wmma_op.h"

// cta<2,64,256> warp<16,64,128> mma<8,8,128>   WARPS[1x2]
AQ_INSTANTIATE_FUN(AqBWMMA, 8, 2, true, 2, 64, 256, 16, 64, 128, 8, 8, 128, 4);
AQ_INSTANTIATE_FUN(AqBWMMA, 8, 2, false, 2, 64, 256, 16, 64, 128, 8, 8, 128, 4);